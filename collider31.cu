#include "hip/hip_runtime.h"
/*
nvcc collider31.cu -o collider31 -lglut -lm -lGLU -lGL --use_fast_math  -O3  -Xptxas  "-warn-lmem-usage -warn-spills" -arch=sm_52
nvcc collider31.cu -o collider31 -lglut -lm -lGLU -lGL -prec-div=false -prec-sqrt=false -ftz=true -O3 
nvcc collider31.cu -o collider31nofast -lglut -lm -lGLU -lGL -O3 
*/

#include <GL/glut.h>
#include <GL/glu.h>
#include <GL/gl.h>
#include <math.h>
#include <stdio.h>
#include "stdio.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <dirent.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <signal.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <time.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

#define BLOCKSIZE 256

#define NUMBEROFEARTHRADIFORMOONMATERIAL 20.0

//Global to hold the time of the collision
double RunTime = 0.0;

//Continue and branch run globals
int TypeOfRun = 0;
char RootFolderName[256] = "";
double AddedRunTime = 0;

//Globals for files
FILE *RunStatsFile;
FILE *PosAndVelFile;
FILE *StartPosAndVelFile;
FILE *ContinueRunStatsFile;
FILE *ContinueRunPosAndVelFile;

//Globals to hold positions, velocities, and forces on both the GPU and CPU
float4 *PlaceHolder; //needs to be hard defined for cuda
float4 *Pos, *Vel, *Force;
float4 *Pos_DEV0, *Vel_DEV0, *Force_DEV0;

float4 *PosFstHalf_0, *VelFstHalf_0, *ForceFstHalf_0;
float4 *PosSndHalf_0, *VelSndHalf_0;
float4 *PosFstHalf_1, *VelFstHalf_1;
float4 *PosSndHalf_1, *VelSndHalf_1, *ForceSndHalf_1;

//Globals to setup the kernals
dim3 BlockConfig, GridConfig;
int NumberOfGpus, Gpu0Access, Gpu1Access;

//Globals to be set by the setRunParameters function
double UnitLength = -1.0;
double Diameter = -1.0;
double UnitMass = -1.0;
double MassSi = -1.0;
double MassFe = -1.0;
double MassOfBody1 = -1.0;
double MassOfBody2 = -1.0;
double UnitTime = -1.0;
double Gravity = -1.0;

int NSi = -1; 
int NSi1 = -1; 
int NSi2 = -1;
int NFe = -1; 
int NFe1 = -1; 
int NFe2 = -1;

//Globals to be set by the findEarthAndMoon function
int NumberOfEarthElements = -1;
int NumberOfMoonElements = -1;
int *EarthIndex;
int *MoonIndex;

//Global to trigger printing collision stats to the screen
int PrintCollisionStats = 0; 

//Global to trigger printing continue stats to the screen
int PrintContinueStats = 0;

//Globals for the run to be read in from the runSetup file
float3 InitialPosition1;
float3 InitialPosition2;
float3 InitialVelocity1;
float3 InitialVelocity2;
float4 InitialSpin1;
float4 InitialSpin2;
float3 BranchPosition1;
float3 BranchPosition2;
float3 BranchVelocity1;
float3 BranchVelocity2;
float4 BranchSpin1;
float4 BranchSpin2;
double FractionEarthMassOfBody1;	//Mass of body 1 as a proportion of the Earth's mass
double FractionEarthMassOfBody2;	//Mass of body 2 as a proportion of the Earth's mass
double FractionFeBody1;			//Percent by mass of iron in body 1
double FractionSiBody1;			//Percent by mass of silicate in body 1
double FractionFeBody2;			//Percent by mass of iron in body 2
double FractionSiBody2;			//Percent by mass of silicate in body 2
float DampRateBody1;
float DampRateBody2;
float EnergyTargetBody1;
float EnergyTargetBody2;
int N;
float TotalRunTime;
float BranchRunTime;
float DampTime;
float DampRestTime;
float EnergyAdjustmentTime;
float EnergyAdjustmentRestTime;
float SpinRestTime;
float BranchSpinRestTime;
float SetupTime;
float Dt;
int WriteToFile;
int RecordRate;
double DensityFe;			//Density of iron in kilograms meterE-3 (Canup science 2012)
double DensitySi;			//Density of silcate in kilograms meterE-3 (Canup science 2012)
double KFe;
double KSi;
double KRFe;
double KRSi;
double SDFe;
double SDSi;
int DrawRate;
int DrawQuality;
int UseMultipleGPU;
double UniversalGravity;	//Universal gravitational constant in kilometersE3 kilogramsE-1 and secondsE-2 (??? source)
double MassOfEarth;
double MassOfMoon;
double AngularMomentumEarthMoonSystem;
double EarthAxialTilt;
double MoonAxialTilt;
double Pi;

void readRunParameters()
{
	ifstream data;
	string name;
	if(TypeOfRun == 0)
	{
		data.open("RunSetup");
	}
	else if(TypeOfRun == 1)
	{
		data.open("RootSetup");
	}
	else if(TypeOfRun == 2)
	{
		data.open("RunSetup");
		if(data.is_open() != 1) data.open("RootSetup");
	}
	else 
	{
		printf("\nTSU Error bad TypeOfRun selected\n");
		exit(0);
	}
	
	if(data.is_open() == 1)
	{
		getline(data,name,'=');
		data >> InitialPosition1.x;
		getline(data,name,'=');
		data >> InitialPosition1.y;
		getline(data,name,'=');
		data >> InitialPosition1.z;
		getline(data,name,'=');
		data >> InitialPosition2.x;
		getline(data,name,'=');
		data >> InitialPosition2.y;
		getline(data,name,'=');
		data >> InitialPosition2.z;
		
		getline(data,name,'=');
		data >> InitialVelocity1.x;
		getline(data,name,'=');
		data >> InitialVelocity1.y;
		getline(data,name,'=');
		data >> InitialVelocity1.z;
		getline(data,name,'=');
		data >> InitialVelocity2.x;
		getline(data,name,'=');
		data >> InitialVelocity2.y;
		getline(data,name,'=');
		data >> InitialVelocity2.z;
		
		getline(data,name,'=');
		data >> InitialSpin1.x;
		getline(data,name,'=');
		data >> InitialSpin1.y;
		getline(data,name,'=');
		data >> InitialSpin1.z;
		getline(data,name,'=');
		data >> InitialSpin1.w;
		
		getline(data,name,'=');
		data >> InitialSpin2.x;
		getline(data,name,'=');
		data >> InitialSpin2.y;
		getline(data,name,'=');
		data >> InitialSpin2.z;
		getline(data,name,'=');
		data >> InitialSpin2.w;
		
		getline(data,name,'=');
		data >> FractionEarthMassOfBody1;
		getline(data,name,'=');
		data >> FractionEarthMassOfBody2;
		
		getline(data,name,'=');
		data >> FractionFeBody1;
		getline(data,name,'=');
		data >> FractionSiBody1;
		getline(data,name,'=');
		data >> FractionFeBody2;
		getline(data,name,'=');
		data >> FractionSiBody2;
		
		getline(data,name,'=');
		data >> DampRateBody1;
		getline(data,name,'=');
		data >> DampRateBody2;
		
		getline(data,name,'=');
		data >> EnergyTargetBody1;
		getline(data,name,'=');
		data >> EnergyTargetBody2;
		
		getline(data,name,'=');
		data >> N;
		
		getline(data,name,'=');
		data >> TotalRunTime;
		getline(data,name,'=');
		data >> DampTime;
		getline(data,name,'=');
		data >> DampRestTime;
		getline(data,name,'=');
		data >> EnergyAdjustmentTime;
		getline(data,name,'=');
		data >> EnergyAdjustmentRestTime;
		getline(data,name,'=');
		data >> SpinRestTime;
		
		getline(data,name,'=');
		data >> Dt;
		
		getline(data,name,'=');
		data >> WriteToFile;
		
		getline(data,name,'=');
		data >> RecordRate;
		
		getline(data,name,'=');
		data >> DensityFe;
		getline(data,name,'=');
		data >> DensitySi;
		
		getline(data,name,'=');
		data >> KFe;
		getline(data,name,'=');
		data >> KSi;
		getline(data,name,'=');
		data >> KRFe;
		getline(data,name,'=');
		data >> KRSi;
		getline(data,name,'=');
		data >> SDFe;
		getline(data,name,'=');
		data >> SDSi;
		
		getline(data,name,'=');
		data >> DrawRate;
		getline(data,name,'=');
		data >> DrawQuality;
		
		getline(data,name,'=');
		data >> UseMultipleGPU;
		
		getline(data,name,'=');
		data >> UniversalGravity;
		getline(data,name,'=');
		data >> MassOfEarth;
		getline(data,name,'=');
		data >> MassOfMoon;
		getline(data,name,'=');
		data >> AngularMomentumEarthMoonSystem;
		getline(data,name,'=');
		data >> EarthAxialTilt;
		getline(data,name,'=');
		data >> MoonAxialTilt;
		getline(data,name,'=');
		data >> Pi;
	}
	else
	{
		printf("\nTSU Error could not open run or root Setup file\n");
		exit(0);
	}
	data.close();
}

void readBranchParameters()
{
	ifstream data;
	string name;
	data.open("BranchSetup");
	if(data.is_open() == 1)
	{
		getline(data,name,'=');
		data >> BranchPosition1.x;
		getline(data,name,'=');
		data >> BranchPosition1.y;
		getline(data,name,'=');
		data >> BranchPosition1.z;
		getline(data,name,'=');
		data >> BranchPosition2.x;
		getline(data,name,'=');
		data >> BranchPosition2.y;
		getline(data,name,'=');
		data >> BranchPosition2.z;
		
		getline(data,name,'=');
		data >> BranchVelocity1.x;
		getline(data,name,'=');
		data >> BranchVelocity1.y;
		getline(data,name,'=');
		data >> BranchVelocity1.z;
		getline(data,name,'=');
		data >> BranchVelocity2.x;
		getline(data,name,'=');
		data >> BranchVelocity2.y;
		getline(data,name,'=');
		data >> BranchVelocity2.z;
		
		getline(data,name,'=');
		data >> BranchSpin1.x;
		getline(data,name,'=');
		data >> BranchSpin1.y;
		getline(data,name,'=');
		data >> BranchSpin1.z;
		getline(data,name,'=');
		data >> BranchSpin1.w;
		
		getline(data,name,'=');
		data >> BranchSpin2.x;
		getline(data,name,'=');
		data >> BranchSpin2.y;
		getline(data,name,'=');
		data >> BranchSpin2.z;
		getline(data,name,'=');
		data >> BranchSpin2.w;
		
		getline(data,name,'=');
		data >> BranchSpinRestTime;
		getline(data,name,'=');
		data >> BranchRunTime;
	}
	else
	{
		printf("\nTSU Error could not open Branch Setup file\n");
		exit(0);
	}
	data.close();
}

void setRunParameters()
{
	double massBody1 = MassOfEarth*FractionEarthMassOfBody1;
	double massBody2 = MassOfEarth*FractionEarthMassOfBody2;
	if(FractionFeBody1 + FractionSiBody1 != 1.0) 
	{
		printf("\nTSU Error: body1 fraction don't add to 1\n");
		exit(0);
	}
	if(FractionFeBody2 + FractionSiBody2 != 1.0) 
	{
		printf("\nTSU Error: body2 fraction don't add to 1\n");
		exit(0);
	}
	double totalMassOfFeBody1 = FractionFeBody1*massBody1;
	double totalMassOfSiBody1 = FractionSiBody1*massBody1;
	double totalMassOfFeBody2 = FractionFeBody2*massBody2;
	double totalMassOfSiBody2 = FractionSiBody2*massBody2;
	double totalMassOfFe = totalMassOfFeBody1 + totalMassOfFeBody2;
	double totalMassOfSi = totalMassOfSiBody1 + totalMassOfSiBody2;
	double massFe;
	double massSi;
	double diameterOfElement;
	
	if(totalMassOfFe != 0.0) NFe = (double)N*(DensitySi/DensityFe)/(totalMassOfSi/totalMassOfFe + DensitySi/DensityFe);
	else NFe = 0;
	NSi = N - NFe;
	
	if(totalMassOfFe != 0.0) NFe1 = NFe*totalMassOfFeBody1/totalMassOfFe; 
	else NFe1 = 0;
	
	NFe2 = NFe - NFe1;
	
	if(totalMassOfSi != 0.0) NSi1 = NSi*totalMassOfSiBody1/totalMassOfSi; 
	else NSi1 = 0;
	
	NSi2 = NSi - NSi1;
	
	if(NFe != 0) massFe = totalMassOfFe/NFe;
	else massFe = 0.0;
	if(NSi != 0) massSi = totalMassOfSi/NSi;
	else massSi = 0.0;
	
	if(NSi != 0) diameterOfElement = pow((6.0*massSi)/(Pi*DensitySi), (1.0/3.0));
	else diameterOfElement = pow((6.0*massFe)/(Pi*DensityFe), (1.0/3.0));
	
	UnitLength = diameterOfElement;
	
	if(NSi != 0) UnitMass = massSi;
	else UnitMass = massFe;
	
	if(NSi != 0) UnitTime = sqrt((6.0*massSi*(double)NSi)/(UniversalGravity*Pi*DensitySi*totalMassOfSi));
	else if(NFe != 0) UnitTime = sqrt((6.0*massFe*(double)NFe)/(UniversalGravity*Pi*DensityFe*totalMassOfFe));
	else 
	{
		printf("TSU Error: No mass, function setRunParameters\n");
		exit(0);
	}
	
	//In this system this is what sets the length unit, the time unit, and the mass unit. 
	Diameter = 1.0;
	Gravity = 1.0;

	if(NSi != 0)
	{
		MassSi = 1.0;
		MassFe = DensityFe/DensitySi;
	}
	else if(NFe != 0)
	{
		MassFe = 1.0;
	}
	else 
	{
		printf("TSU Error: No mass, function setRunParameters\n");
		exit(0);
	}
	
	//Setting mass of bodies in our units
	MassOfBody1 = massBody1/UnitMass;
	MassOfBody2 = massBody2/UnitMass;
	
	//Putting Initial positions into our units
	InitialPosition1.x /= UnitLength;
	InitialPosition1.y /= UnitLength;
	InitialPosition1.z /= UnitLength;

	InitialPosition2.x /= UnitLength;
	InitialPosition2.y /= UnitLength;
	InitialPosition2.z /= UnitLength;

	//Putting Initial Velocities into our units
	InitialVelocity1.x *= UnitTime/UnitLength;
	InitialVelocity1.y *= UnitTime/UnitLength;
	InitialVelocity1.z *= UnitTime/UnitLength;

	InitialVelocity2.x *= UnitTime/UnitLength;
	InitialVelocity2.y *= UnitTime/UnitLength;
	InitialVelocity2.z *= UnitTime/UnitLength;

	//Putting Initial Angule Velocities into our units
	InitialSpin1.w *= UnitTime/3600.0;

	InitialSpin2.w *= UnitTime/3600.0;
	
	//Putting Run times into our units
	TotalRunTime *= 3600.0/UnitTime;
	DampTime *= 3600.0/UnitTime;
	DampRestTime *= 3600.0/UnitTime;
	EnergyAdjustmentTime *= 3600.0/UnitTime;
	EnergyAdjustmentRestTime *= 3600.0/UnitTime;
	SpinRestTime *= 3600.0/UnitTime;
	SetupTime = (DampTime + DampRestTime + EnergyAdjustmentTime + EnergyAdjustmentRestTime + SpinRestTime); 
	
	KFe *= UnitTime*UnitTime*UnitLength/UnitMass;
	KSi *= UnitTime*UnitTime*UnitLength/UnitMass;
}

void setBranchParameters()
{
	//Putting Branch positions into our units
	BranchPosition1.x /= UnitLength;
	BranchPosition1.y /= UnitLength;
	BranchPosition1.z /= UnitLength;

	BranchPosition2.x /= UnitLength;
	BranchPosition2.y /= UnitLength;
	BranchPosition2.z /= UnitLength;

	//Putting Branch Velocities into our units
	BranchVelocity1.x *= UnitTime/UnitLength;
	BranchVelocity1.y *= UnitTime/UnitLength;
	BranchVelocity1.z *= UnitTime/UnitLength;

	BranchVelocity2.x *= UnitTime/UnitLength;
	BranchVelocity2.y *= UnitTime/UnitLength;
	BranchVelocity2.z *= UnitTime/UnitLength;

	//Putting Branch Angule Velocities into our units
	BranchSpin1.w *= UnitTime/3600.0;

	BranchSpin2.w *= UnitTime/3600.0;
	
	//Putting Branch Run times into our units
	BranchSpinRestTime *= 3600.0/UnitTime;
	BranchRunTime *= 3600.0/UnitTime;
}

//Globals for setting up the viewing window 
int XWindowSize = 2500;
int YWindowSize = 2500; 
double Near = 0.2;
double Far = 600.0;

double ViewBoxSize = 300.0;

GLdouble Left = -ViewBoxSize;
GLdouble Right = ViewBoxSize;
GLdouble Bottom = -ViewBoxSize;
GLdouble Top = ViewBoxSize;
GLdouble Front = ViewBoxSize;
GLdouble Back = -ViewBoxSize;

//Direction here your eye is located location

double EyeX = 100.0;
double EyeY = 100.0;
double EyeZ = 100.0;

//Where you are looking

double CenterX = 0.0;
double CenterY = 0.0;
double CenterZ = 0.0;

//Up vector for viewing

double UpX = 0.0;
double UpY = 1.0;
double UpZ = 0.0;

void createFolderForNewRun()
{   	
	//Create output folder to store run parameters and run positions and velocities
	time_t t = time(0); 
	struct tm * now = localtime( & t );
	int month = now->tm_mon + 1, day = now->tm_mday, curTimeHour = now->tm_hour, curTimeMin = now->tm_min;
	stringstream smonth, sday, stimeHour, stimeMin;
	smonth << month;
	sday << day;
	stimeHour << curTimeHour;
	stimeMin << curTimeMin;
	string monthday;
	if (curTimeMin <= 9)	monthday = smonth.str() + "-" + sday.str() + "-" + stimeHour.str() + ":0" + stimeMin.str();
	else			monthday = smonth.str() + "-" + sday.str() + "-" + stimeHour.str() + ":" + stimeMin.str();
	string foldernametemp = "Run:" + monthday;
	const char *foldername = foldernametemp.c_str();
	mkdir(foldername , S_IRWXU|S_IRWXG|S_IRWXO);
	chdir(foldername);
	
	//Copying the RunSetup file into the run folder
	FILE *runSetupIn;
	FILE *runSetupOut;
	long sizeOfFile;
  	char * buffer;
    	
    	runSetupIn = fopen("../RunSetup", "rb");
    	fseek (runSetupIn , 0 , SEEK_END);
  	sizeOfFile = ftell (runSetupIn);
  	rewind (runSetupIn);
  	buffer = (char*) malloc (sizeof(char)*sizeOfFile);
  	fread (buffer, 1, sizeOfFile, runSetupIn);
  	
    	runSetupOut = fopen("RunSetup", "wb");
    	fwrite (buffer, 1, sizeOfFile, runSetupOut);

	fclose(runSetupIn);
	fclose(runSetupOut);
	free (buffer);
}

void createFolderForBranchRun(const char* rootFolder)
{   	
	//Create output folder to store run parameters and run positions and velocities
	time_t t = time(0); 
	struct tm * now = localtime( & t );
	int month = now->tm_mon + 1, day = now->tm_mday, curTimeHour = now->tm_hour, curTimeMin = now->tm_min;
	stringstream smonth, sday, stimeHour, stimeMin;
	smonth << month;
	sday << day;
	stimeHour << curTimeHour;
	stimeMin << curTimeMin;
	string monthday;
	if (curTimeMin <= 9)	monthday = smonth.str() + "-" + sday.str() + "-" + stimeHour.str() + ":0" + stimeMin.str();
	else			monthday = smonth.str() + "-" + sday.str() + "-" + stimeHour.str() + ":" + stimeMin.str();
	string foldernametemp = "BranchRun:" + monthday;
	const char *foldername = foldernametemp.c_str();
	mkdir(foldername , S_IRWXU|S_IRWXG|S_IRWXO);
	chdir(foldername);
	
	FILE *fileIn;
	FILE *fileOut;
	long sizeOfFile;
  	char * buffer;
  	char path[256];
  	
  	//Copying the RunSetup file into the branch run folder
  	strcpy(path,  "../");
  	strcat(path, rootFolder);
  	strcat(path,"/RunSetup");
    	
    	fileIn = fopen(path, "rb");
    	if(fileIn == NULL)
    	{
    		printf("\n\n The RunSetup file does not exist\n\n");
    		exit(0);
    	}
    	
    	fseek (fileIn , 0 , SEEK_END);
  	sizeOfFile = ftell (fileIn);
  	rewind (fileIn);
  	buffer = (char*) malloc (sizeof(char)*sizeOfFile);
  	fread (buffer, 1, sizeOfFile, fileIn);
  	
    	fileOut = fopen("RootSetup", "wb");
    	fwrite (buffer, 1, sizeOfFile, fileOut);
    	fclose(fileOut);
    	
    	fileOut = fopen("RunSetup", "wb");
    	fwrite (buffer, 1, sizeOfFile, fileOut);

	fclose(fileIn);
	fclose(fileOut);
	free (buffer);
	
	//Copying the RunStatsFile file into the branch run folder
  	strcpy(path,  "../");
  	strcat(path, rootFolder);
  	strcat(path,"/RunStats");
    	
    	fileIn = fopen(path, "rb");
    	if(fileIn == NULL)
    	{
    		printf("\n\n The RunStats file does not exist\n\n");
    		exit(0);
    	}
    	fseek (fileIn , 0 , SEEK_END);
  	sizeOfFile = ftell (fileIn);
  	rewind (fileIn);
  	buffer = (char*) malloc (sizeof(char)*sizeOfFile);
  	fread (buffer, 1, sizeOfFile, fileIn);
  	
    	fileOut = fopen("RootRunStats", "wb");
    	fwrite (buffer, 1, sizeOfFile, fileOut);

	fclose(fileIn);
	fclose(fileOut);
	free (buffer);
	
	//Copying the Branch Positions and Velocities file into the branch run folder
  	strcpy(path,  "../");
  	strcat(path, rootFolder);
  	strcat(path,"/StartPosAndVel");
    	
    	fileIn = fopen(path, "rb");
    	if(fileIn == NULL)
    	{
    		printf("\n\n The StartPosAndVel file does not exist\n\n");
    		exit(0);
    	}
    	fseek (fileIn , 0 , SEEK_END);
  	sizeOfFile = ftell (fileIn);
  	rewind (fileIn);
  	buffer = (char*) malloc (sizeof(char)*sizeOfFile);
  	fread (buffer, 1, sizeOfFile, fileIn);
  	
    	fileOut = fopen("RootStartPosAndVel", "wb");
    	fwrite (buffer, 1, sizeOfFile, fileOut);

	fclose(fileIn);
	fclose(fileOut);
	free (buffer);
	
	//Copying the Branch setup file into the branch run folder
  	strcpy(path,  "../");
  	strcat(path,"BranchSetup");
    	
    	fileIn = fopen(path, "rb");
    	if(fileIn == NULL)
    	{
    		printf("\n\n The BranchSetup file does not exist\n\n");
    		exit(0);
    	}
    	fseek (fileIn , 0 , SEEK_END);
  	sizeOfFile = ftell (fileIn);
  	rewind (fileIn);
  	buffer = (char*) malloc (sizeof(char)*sizeOfFile);
  	fread (buffer, 1, sizeOfFile, fileIn);
  	
    	fileOut = fopen("BranchSetup", "wb");
    	fwrite (buffer, 1, sizeOfFile, fileOut);

	fclose(fileIn);
	fclose(fileOut);
	free (buffer);
}

void openNewRunFiles()
{
	RunStatsFile = fopen("RunStats", "wb");	
	PosAndVelFile = fopen("PosAndVel", "wb");
	StartPosAndVelFile = fopen("StartPosAndVel", "wb");
	ContinueRunStatsFile = fopen("ContinueRunStats", "wb");
	ContinueRunPosAndVelFile = fopen("ContinueRunPosAndVel", "wb");
}

void openBranchRunFiles()
{
	RunStatsFile = fopen("RunStats", "wb");	
	PosAndVelFile = fopen("PosAndVel", "wb");
	StartPosAndVelFile = fopen("StartPosAndVel", "wb");
	ContinueRunStatsFile = fopen("ContinueRunStats", "wb");
	ContinueRunPosAndVelFile = fopen("ContinueRunPosAndVel", "wb");
}

void openContinueRunFiles()
{
	RunStatsFile = fopen("RunStats", "wb");	
	PosAndVelFile = fopen("PosAndVel", "ab");
	//fseek(PosAndVelFile,0,SEEK_END);
	ContinueRunStatsFile = fopen("ContinueRunStats", "wb");
	ContinueRunPosAndVelFile = fopen("ContinueRunPosAndVel", "wb");
}

void recordSetupStats()
{
	float mag;
	fprintf(RunStatsFile, "The conversion parameters to take you to and from our units to the real world units follow\n");
	
	fprintf(RunStatsFile, "\nOur length unit is this many kilometers: 	UnitLength = %f", UnitLength);
	fprintf(RunStatsFile, "\nOur mass unit is this many kilograms: 		UnitMass = %e", UnitMass);
	fprintf(RunStatsFile, "\nOur time unit is this many seconds: 		UnitTime = %f\n", UnitTime);
	
	fprintf(RunStatsFile,   "\nThe initail statistics for this run in our units follow\n");
	fprintf(RunStatsFile, "\nDiameter of an element: 		Diameter = %f", Diameter);
	fprintf(RunStatsFile, "\nGravity in our units: 			Gravity = %f", Gravity);
	fprintf(RunStatsFile, "\nThe mass of a silicate element: 	MassSi = %f", MassSi);
	fprintf(RunStatsFile, "\nThe mass of an iron element: 		MassFe = %f\n", MassFe);
	
	fprintf(RunStatsFile, "\nThe push back strength of iron: 	KFe = %f", KFe);
	fprintf(RunStatsFile, "\nThe push back strength of silicate: 	KSi = %f\n", KSi);
	
	fprintf(RunStatsFile, "\nThe mass of body one: 	MassOfBody1 = %f", MassOfBody1);
	fprintf(RunStatsFile, "\nThe mass of body two: 	MassOfBody2 = %f\n", MassOfBody2);
	
	fprintf(RunStatsFile, "\nThe initial position of body one: (%f, %f, %f)", InitialPosition1.x, InitialPosition1.y, InitialPosition1.z);
	fprintf(RunStatsFile, "\nThe initial position of body two: (%f, %f, %f)\n", InitialPosition2.x, InitialPosition2.y, InitialPosition2.z);
	
	fprintf(RunStatsFile, "\nThe initial velocity of body one: (%f, %f, %f)", InitialVelocity1.x, InitialVelocity1.y, InitialVelocity1.z);
	fprintf(RunStatsFile, "\nThe initial velocity of body two: (%f, %f, %f)\n", InitialVelocity2.x, InitialVelocity2.y, InitialVelocity2.z);
	
	mag = sqrt(InitialSpin1.x*InitialSpin1.x + InitialSpin1.y*InitialSpin1.y + InitialSpin1.z*InitialSpin1.z);
	fprintf(RunStatsFile, "\nThe initial spin in revolutions per time unit of body one: (%f, %f, %f, %f)", InitialSpin1.x/mag, InitialSpin1.y/mag, InitialSpin1.z/mag, InitialSpin1.w);
	mag = sqrt(InitialSpin2.x*InitialSpin2.x + InitialSpin2.y*InitialSpin2.y + InitialSpin2.z*InitialSpin2.z);
	fprintf(RunStatsFile, "\nThe initial spin in revolutions per time unit of body two: (%f, %f, %f, %f)\n", InitialSpin2.x/mag, InitialSpin2.y/mag, InitialSpin2.z/mag, InitialSpin2.w);
	
	
	fprintf(RunStatsFile, "\nTotal number of elements: 				N = %d", N);
	fprintf(RunStatsFile, "\nTotal number of iron elements: 				NFe = %d", NFe);
	fprintf(RunStatsFile, "\nTotal number of silicate elements: 			NSi = %d", NSi);
	fprintf(RunStatsFile, "\nTotal number of iron elements in body1: 		NFe1 = %d", NFe1);
	fprintf(RunStatsFile, "\nTotal number of silicate elements in body1: 		NSi1 = %d", NSi1);
	fprintf(RunStatsFile, "\nTotal number of iron elements in body2 			NFe2: = %d", NFe2);
	fprintf(RunStatsFile, "\nTotal number of silicate elements in body2: 		NSi2 = %d\n", NSi2);
	
	fprintf(RunStatsFile, "\nTime step in our units: 	Dt = %f", Dt);
	fprintf(RunStatsFile, "\nRecord rate: 			RecordRate = %d", RecordRate);
	fprintf(RunStatsFile, "\nTotal run time in our units: 	TotalRunTime = %f\n", TotalRunTime);
	
	fprintf(RunStatsFile, "\nDamp time in our units: 			DampTime = %f", DampTime);
	fprintf(RunStatsFile, "\nDamp rest time in our units: 			DampRestTime = %f", DampRestTime);
	fprintf(RunStatsFile, "\nEnergy adjustment time in our units: 		EnergyAdjustmentTime = %f", EnergyAdjustmentTime);
	fprintf(RunStatsFile, "\nEnergy adjustment rest time in our units: 	EnergyAdjustmentRestTime = %f", EnergyAdjustmentRestTime);
	fprintf(RunStatsFile, "\nSpin rest time in our units: 			SpinRestTime = %f", SpinRestTime);
	fprintf(RunStatsFile, "\nTotal setup time in our units: 			SetupTime = %f\n", SetupTime);
}

//Creating structures to hold constants needed in the kernals
struct forceSeperateKernalConstantsStruct
{
	float GMassFeFe;
	float GMassFeSi;    
	float KFeFe;
	float KSiSi;
	float KFeSi;
	float KRFeFe;
	float KRSiSi;
	float KRFeSi;
	float KRMix;
	float ShellBreakFe;
	float ShellBreakSi;
	float ShellBreakFeSi1;
	float ShellBreakFeSi2; 
	int boarder1; 
	int boarder2;
	int boarder3;  
};

struct forceCollisionKernalConstantsStruct
{
	float GMassFeFe;
	float GMassFeSi;    
	float KFeFe;
	float KSiSi;
	float KFeSi;
	float KRFeFe;
	float KRSiSi;
	float KRFeSi;
	float KRMix;
	float ShellBreakFe;
	float ShellBreakSi;
	float ShellBreakFeSi1;
	float ShellBreakFeSi2; 
	int NFe;   
};

struct moveSeperateKernalConstantsStruct
{
	float Dt;
	float DtOverMassFe;
	float DtOverMassSi;
	int boarder1; 
	int boarder2;
	int boarder3;
};

struct moveCollisionKernalConstantsStruct
{
	float Dt;
	float DtOverMassFe;
	float DtOverMassSi;
	int NFe;
};

//Globals to hold kernal constants
forceSeperateKernalConstantsStruct ForceSeperateConstant;
forceCollisionKernalConstantsStruct ForceCollisionConstant;
moveSeperateKernalConstantsStruct MoveSeperateConstant;
moveCollisionKernalConstantsStruct MoveCollisionConstant; 

void loadKernalConstantStructures()
{
	//Force kernal seperate
	ForceSeperateConstant.GMassFeFe = Gravity*MassFe*MassFe;
	ForceSeperateConstant.GMassFeSi = Gravity*MassFe*MassSi;
	
	ForceSeperateConstant.KFeFe = 2.0*KFe;
	ForceSeperateConstant.KSiSi = 2.0*KSi;
	ForceSeperateConstant.KFeSi = KFe + KSi;
	
	ForceSeperateConstant.KRFeFe = 2.0*KFe*KRFe;
	ForceSeperateConstant.KRSiSi = 2.0*KSi*KRSi;
	ForceSeperateConstant.KRFeSi = KFe*KRFe + KSi*KRSi;
	
	if(SDFe >= SDSi) 	ForceSeperateConstant.KRMix = KFe + KSi*KRSi; 
	else 			ForceSeperateConstant.KRMix = KFe*KRFe + KSi;
	
	ForceSeperateConstant.ShellBreakFe = Diameter - Diameter*SDFe;
	ForceSeperateConstant.ShellBreakSi = Diameter - Diameter*SDSi;
	if(SDFe >= SDSi)
	{
		ForceSeperateConstant.ShellBreakFeSi1 = Diameter - Diameter*SDSi;
		ForceSeperateConstant.ShellBreakFeSi2 = Diameter - Diameter*SDFe;
	} 
	else 
	{
		ForceSeperateConstant.ShellBreakFeSi1 = Diameter - Diameter*SDFe;
		ForceSeperateConstant.ShellBreakFeSi2 = Diameter - Diameter*SDSi;
	}
	
	ForceSeperateConstant.boarder1 = NFe1;
	ForceSeperateConstant.boarder2 = NFe1 + NSi1;
	ForceSeperateConstant.boarder3 = NFe1 + NSi1 + NFe2;
	
	//Force kernal Earth Moon System
	ForceCollisionConstant.GMassFeFe = Gravity*MassFe*MassFe;
	ForceCollisionConstant.GMassFeSi = Gravity*MassFe*MassSi;
	
	ForceCollisionConstant.KFeFe = 2.0*KFe;
	ForceCollisionConstant.KSiSi = 2.0*KSi;
	ForceCollisionConstant.KFeSi = KFe + KSi;
	
	ForceCollisionConstant.KRFeFe = 2.0*KFe*KRFe;
	ForceCollisionConstant.KRSiSi = 2.0*KSi*KRSi;
	ForceCollisionConstant.KRFeSi = KFe*KRFe + KSi*KRSi;
	
	if(SDFe >= SDSi) 	ForceCollisionConstant.KRMix = KFe + KSi*KRSi; 
	else 			ForceCollisionConstant.KRMix = KFe*KRFe + KSi;
	
	ForceCollisionConstant.ShellBreakFe = Diameter - Diameter*SDFe;
	ForceCollisionConstant.ShellBreakSi = Diameter - Diameter*SDSi;
	if(SDFe >= SDSi)
	{
		ForceCollisionConstant.ShellBreakFeSi1 = Diameter - Diameter*SDSi;
		ForceCollisionConstant.ShellBreakFeSi2 = Diameter - Diameter*SDFe;
	} 
	else 
	{
		ForceCollisionConstant.ShellBreakFeSi1 = Diameter - Diameter*SDFe;
		ForceCollisionConstant.ShellBreakFeSi2 = Diameter - Diameter*SDSi;
	}
	
	ForceCollisionConstant.NFe = NFe;
	
	//Move kernal seperate	
	MoveSeperateConstant.Dt = Dt;
	MoveSeperateConstant.DtOverMassFe = Dt/MassFe;
	MoveSeperateConstant.DtOverMassSi = Dt/MassSi;
	MoveSeperateConstant.boarder1 = NFe1;
	MoveSeperateConstant.boarder2 = NSi1 + NFe1;
	MoveSeperateConstant.boarder3 = NFe1 + NSi1 + NFe2;
	
	//Move kernal Earth Moon System
	MoveCollisionConstant.Dt = Dt;
	MoveCollisionConstant.DtOverMassSi = Dt/MassSi;
	MoveCollisionConstant.DtOverMassFe = Dt/MassFe;
	MoveCollisionConstant.NFe = NFe;
}

void errorCheck(const char *message)
{
  hipError_t  error;
  error = hipGetLastError();

  if(error != hipSuccess)
  {
    printf("\n CUDA ERROR: %s = %s\n", message, hipGetErrorString(error));
    exit(0);
  }
}

void allocateCPUMemory()
{
	PlaceHolder = (float4*)malloc(N*sizeof(float4));
	Pos = (float4*)malloc(N*sizeof(float4));
	Vel = (float4*)malloc(N*sizeof(float4));
	Force = (float4*)malloc(N*sizeof(float4));
}

void checkSetupForErrors()
{
	if(N%BLOCKSIZE != 0)
	{
		printf("\nTSU Error: Number of Particles is not a multiple of the block size \n\n");
		exit(0);
	}
}

void deviceSetupSeperate()
{
	BlockConfig.x = BLOCKSIZE;
	BlockConfig.y = 1;
	BlockConfig.z = 1;
	
	GridConfig.x = (N-1)/BlockConfig.x + 1;
	GridConfig.y = 1;
	GridConfig.z = 1;
	
	hipMalloc((void**)&Pos_DEV0, N *sizeof(float4));
	errorCheck("hipMalloc Pos");
	hipMalloc((void**)&Vel_DEV0, N *sizeof(float4));
	errorCheck("hipMalloc Vel");
	hipMalloc((void**)&Force_DEV0, N *sizeof(float4));
	errorCheck("hipMalloc Force");
}

void deviceSetupCollision()
{
	hipGetDeviceCount(&NumberOfGpus);
	printf("\n***** You have %d GPUs available\n", NumberOfGpus);
	errorCheck("hipGetDeviceCount");
	hipDeviceCanAccessPeer(&Gpu0Access,0,1);
	errorCheck("cudaDeviceCanAccessPeer0");
	hipDeviceCanAccessPeer(&Gpu1Access,1,0);
	errorCheck("cudaDeviceCanAccessPeer1");
	if(1 < NumberOfGpus && UseMultipleGPU == 1)
	{
		printf("\n***** You will be using %d GPUs\n", NumberOfGpus);
		if(Gpu0Access == 0)
		{
			printf("\nTSU Error: Device0 can not do peer to peer\n");
		}
	
		if(Gpu1Access == 0)
		{
			printf("\nTSU Error: Device1 can not do peer to peer\n");
		}
		hipDeviceEnablePeerAccess(1,0);
		errorCheck("hipDeviceEnablePeerAccess");
		
		BlockConfig.x = BLOCKSIZE;
		BlockConfig.y = 1;
		BlockConfig.z = 1;
		
		GridConfig.x = ((N/2)-1)/BlockConfig.x + 1;
		GridConfig.y = 1;
		GridConfig.z = 1;
		
		hipSetDevice(0);
		errorCheck("cudaSetDevice0");
		hipMalloc( (void**)&PosFstHalf_0, (N/2)*sizeof(float4) );
		errorCheck("hipMalloc PFH0");
		hipMalloc( (void**)&PosSndHalf_0, (N/2)*sizeof(float4) );
		errorCheck("hipMalloc PSH0");
		hipMalloc( (void**)&VelFstHalf_0, (N/2)*sizeof(float4) );
		errorCheck("hipMalloc VFH0");
		hipMalloc( (void**)&VelSndHalf_0, (N/2)*sizeof(float4) );
		errorCheck("hipMalloc VSH0");
		hipMalloc( (void**)&ForceFstHalf_0, (N/2)*sizeof(float4) );
		errorCheck("hipMalloc FFH0");

		hipSetDevice(1);
		errorCheck("cudaSetDevice1");
		hipMalloc( (void**)&PosFstHalf_1, (N/2)*sizeof(float4) );
		errorCheck("hipMalloc PFH1");
		hipMalloc( (void**)&PosSndHalf_1, (N/2)*sizeof(float4) );
		errorCheck("hipMalloc PSH1");
		hipMalloc( (void**)&VelFstHalf_1, (N/2)*sizeof(float4) );
		errorCheck("hipMalloc VFH1");
		hipMalloc( (void**)&VelSndHalf_1, (N/2)*sizeof(float4) );
		errorCheck("hipMalloc VSH1");
		hipMalloc( (void**)&ForceSndHalf_1, (N/2)*sizeof(float4) );
		errorCheck("hipMalloc FSH1");
	}
	else
	{
		BlockConfig.x = BLOCKSIZE;
		BlockConfig.y = 1;
		BlockConfig.z = 1;
	
		GridConfig.x = (N-1)/BlockConfig.x + 1;
		GridConfig.y = 1;
		GridConfig.z = 1;
	
		hipMalloc((void**)&Pos_DEV0, N *sizeof(float4));
		errorCheck("hipMalloc P0");
		hipMalloc((void**)&Vel_DEV0, N *sizeof(float4));
		errorCheck("hipMalloc V0");
		hipMalloc((void**)&Force_DEV0, N *sizeof(float4));
		errorCheck("hipMalloc F0");
	}
}

void cleanUpSeperate()
{
	hipFree(Pos_DEV0);
	hipFree(Vel_DEV0);
	hipFree(Force_DEV0);
	fclose(StartPosAndVelFile);
}

void cleanUpCollision()
{
	fclose(RunStatsFile);
	fclose(PosAndVelFile);
	fclose(ContinueRunStatsFile);
	fclose(ContinueRunPosAndVelFile);
	
	if(1 < NumberOfGpus && UseMultipleGPU == 1)
	{
		hipSetDevice(0);
		errorCheck("hipSetDevice 0");
		hipFree(PosFstHalf_0);
		hipFree(VelFstHalf_0);
		hipFree(ForceFstHalf_0);
		hipFree(PosSndHalf_0);
		hipFree(VelSndHalf_0);
		
		hipSetDevice(1);
		errorCheck("hipSetDevice 0");
		hipFree(PosFstHalf_1);
		hipFree(VelFstHalf_1);
		hipFree(ForceSndHalf_1);
		hipFree(PosSndHalf_1);
		hipFree(VelSndHalf_1);
	}
	else
	{
		hipFree(Pos_DEV0);
		hipFree(Vel_DEV0);
		hipFree(Force_DEV0);
	}
}

void createBodies()
{
	float radius1, radius2, stretch;
	float volume, mag, radius, seperation;
	int test, repeatCount;
	time_t t;
	
	printf("\nCreating the raw bodies\n");
	//Creating body one
	//This assumes a 68% packing ratio of a shpere with shperes and then stretches it by strecth 
	//to safely fit all the balls in.
	stretch = 2.0;
	volume = ((4.0/3.0)*Pi*pow(Diameter,3)*(float)NFe1/0.68)*stretch;
	radius1 = pow(volume/((4.0/3.0)*Pi),(1.0/3.0));
	volume = ((4.0/3.0)*Pi*pow(Diameter,3)*(float)(NFe1 + NSi1)/0.68)*stretch;
	radius2 = pow(volume/((4.0/3.0)*Pi),(1.0/3.0));
	srand((unsigned) time(&t));
	
	repeatCount = 0;
	for(int i=0; i<NFe1; i++)
	{
		test = 0;
		while(test == 0)
		{
			Pos[i].x = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			Pos[i].y = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			Pos[i].z = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			mag = sqrt(Pos[i].x*Pos[i].x + Pos[i].y*Pos[i].y + Pos[i].z*Pos[i].z);
			radius = ((float)rand()/(float)RAND_MAX)*radius1;
			Pos[i].x *= radius/mag;
			Pos[i].y *= radius/mag;
			Pos[i].z *= radius/mag;
			test = 1;
			for(int j = 0; j < i; j++)
			{
				seperation = mag = sqrt((Pos[i].x-Pos[j].x)*(Pos[i].x-Pos[j].x) + (Pos[i].y-Pos[j].y)*(Pos[i].y-Pos[j].y) + (Pos[i].z-Pos[j].z)*(Pos[i].z-Pos[j].z));
				if(seperation < Diameter)
				{
					test = 0;
					repeatCount++;
					break;
				}
			}
		}
		Pos[i].w = 0.0;
		
		Vel[i].x = 0.0;
		Vel[i].y = 0.0;
		Vel[i].z = 0.0;
		Vel[i].w = MassFe;
	}
	
	for(int i = NFe1; i < (NFe1 + NSi1); i++)
	{
		test = 0;
		while(test == 0)
		{
			Pos[i].x = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			Pos[i].y = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			Pos[i].z = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			mag = sqrt(Pos[i].x*Pos[i].x + Pos[i].y*Pos[i].y + Pos[i].z*Pos[i].z);
			radius = ((float)rand()/(float)RAND_MAX)*(radius2-radius1) + radius1 + Diameter;
			Pos[i].x *= radius/mag;
			Pos[i].y *= radius/mag;
			Pos[i].z *= radius/mag;
			test = 1;
			for(int j = NFe1; j < i; j++)
			{
				seperation = mag = sqrt((Pos[i].x-Pos[j].x)*(Pos[i].x-Pos[j].x) + (Pos[i].y-Pos[j].y)*(Pos[i].y-Pos[j].y) + (Pos[i].z-Pos[j].z)*(Pos[i].z-Pos[j].z));
				if(seperation < Diameter)
				{
					test = 0;
					repeatCount++;
					break;
				}
			}
		}
		Pos[i].w = 1.0;
		
		Vel[i].x = 0.0;
		Vel[i].y = 0.0;
		Vel[i].z = 0.0;
		Vel[i].w = MassSi;
	}
	printf("\nrepeat count body one= %d", repeatCount);
	
	//Setting the body one's center of mass location
	for(int i=0; i<(NFe1 + NSi1); i++)
	{
		Pos[i].x += InitialPosition1.x;
		Pos[i].y += InitialPosition1.y;
		Pos[i].z += InitialPosition1.z;
	}
	
	//Creating body two
	//This assumes a 68% packing ratio of a shpere with shperes and then stretches it by strecth 
 	//to safely fit all the balls in.
	stretch = 2.0;
	volume = ((4.0/3.0)*Pi*pow(Diameter,3)*(float)NFe2/0.68)*stretch;
	radius1 = pow(volume/((4.0/3.0)*Pi),(1.0/3.0));
	volume = ((4.0/3.0)*Pi*pow(Diameter,3)*(float)(NFe2 + NSi2)/0.68)*stretch;
	radius2 = pow(volume/((4.0/3.0)*Pi),(1.0/3.0));
	srand((unsigned) time(&t));
	
	repeatCount = 0;
	for(int i = (NFe1 + NSi1); i < (NFe1 + NSi1 + NFe2); i++)
	{
		test = 0;
		while(test == 0)
		{
			Pos[i].x = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			Pos[i].y = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			Pos[i].z = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			mag = sqrt(Pos[i].x*Pos[i].x + Pos[i].y*Pos[i].y + Pos[i].z*Pos[i].z);
			radius = ((float)rand()/(float)RAND_MAX)*radius1;
			Pos[i].x *= radius/mag;
			Pos[i].y *= radius/mag;
			Pos[i].z *= radius/mag;
			test = 1;
			for(int j = (NFe1 + NSi1); j < i; j++)
			{
				seperation = mag = sqrt((Pos[i].x-Pos[j].x)*(Pos[i].x-Pos[j].x) + (Pos[i].y-Pos[j].y)*(Pos[i].y-Pos[j].y) + (Pos[i].z-Pos[j].z)*(Pos[i].z-Pos[j].z));
				if(seperation < Diameter)
				{
					test = 0;
					repeatCount++;
					break;
				}
			}
		}
		Pos[i].w = 2.0;
		
		Vel[i].x = 0.0;
		Vel[i].y = 0.0;
		Vel[i].z = 0.0;
		Vel[i].w = MassFe;
	}
	for(int i = (NFe1 + NSi1 + NFe2); i < N; i++)
	{
		test = 0;
		while(test == 0)
		{
			Pos[i].x = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			Pos[i].y = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			Pos[i].z = ((float)rand()/(float)RAND_MAX)*2.0 - 1.0;
			mag = sqrt(Pos[i].x*Pos[i].x + Pos[i].y*Pos[i].y + Pos[i].z*Pos[i].z);
			radius = ((float)rand()/(float)RAND_MAX)*(radius2-radius1) + radius1 + Diameter;
			Pos[i].x *= radius/mag;
			Pos[i].y *= radius/mag;
			Pos[i].z *= radius/mag;
			test = 1;
			for(int j = (NFe1 + NSi1 + NFe2); j < i; j++)
			{
				seperation = mag = sqrt((Pos[i].x-Pos[j].x)*(Pos[i].x-Pos[j].x) + (Pos[i].y-Pos[j].y)*(Pos[i].y-Pos[j].y) + (Pos[i].z-Pos[j].z)*(Pos[i].z-Pos[j].z));
				if(seperation < Diameter)
				{
					test = 0;
					repeatCount++;
					break;
				}
			}
		}
		Pos[i].w = 3.0;
		
		Vel[i].x = 0.0;
		Vel[i].y = 0.0;
		Vel[i].z = 0.0;
		Vel[i].w = MassSi;
	}
	printf("\nrepeat count body two = %d", repeatCount);
	
	//Setting the body one's center of mass location
	for(int i = (NFe1 + NSi1); i < N; i++)
	{
		Pos[i].x += InitialPosition2.x;
		Pos[i].y += InitialPosition2.y;
		Pos[i].z += InitialPosition2.z;
	}
	printf("\n************************************************** Initial bodies have been formed\n");
}

__global__ void getForcesSeperate(float4 *pos, float4 *vel, float4 *force, forceSeperateKernalConstantsStruct constant)
{
	int id, ids;
	int i,j;
	int inout;
    	float4 forceSum;
    	float4 posMe;
    	float4 velMe;
    	int test;
    	int materialSwitch;
    	float force_mag;
    	float4 dp;
    	float4 dv;
    	float r2;
    	float r;
    	float invr;
    
    	__shared__ float4 shPos[BLOCKSIZE];
    	__shared__ float4 shVel[BLOCKSIZE];
    
    	id = threadIdx.x + blockDim.x*blockIdx.x;
		
	forceSum.x = 0.0f;
	forceSum.y = 0.0f;
	forceSum.z = 0.0f;
		
	posMe.x = pos[id].x;
	posMe.y = pos[id].y;
	posMe.z = pos[id].z;
	
	velMe.x = vel[id].x;
	velMe.y = vel[id].y;
	velMe.z = vel[id].z;
		
	for(j = 0; j < gridDim.x; j++)
	{
		shPos[threadIdx.x] = pos[threadIdx.x + blockDim.x*j];
		shVel[threadIdx.x] = vel[threadIdx.x + blockDim.x*j];
		__syncthreads();
	   
		for(i = 0; i < blockDim.x; i++)	
		{
			ids = i + blockDim.x*j;
    			if((id < constant.boarder2 && ids < constant.boarder2) || (constant.boarder2 <= id && constant.boarder2 <= ids))
    			{
	    			if((id < constant.boarder2) && (ids < constant.boarder2)) materialSwitch = constant.boarder1;
	    			if((constant.boarder2 <= id) && (constant.boarder2 <= ids)) materialSwitch = constant.boarder3;
	    			
				dp.x = shPos[i].x - posMe.x;
				dp.y = shPos[i].y - posMe.y;
				dp.z = shPos[i].z - posMe.z;
				r2 = dp.x*dp.x + dp.y*dp.y + dp.z*dp.z;
				r = sqrt(r2);
				if(id == ids) invr = 0;
				else invr = 1.0f/r;

				test = 0;
				if(id < materialSwitch) test = 1;
				if(ids < materialSwitch) test++;
		
				if(test == 0) //silicate silicate force
				{
					if(1.0 <= r)
					{
						force_mag = 1.0/r2;  // G = 1 and mass of silicate elemnet =1
					}
					else if(constant.ShellBreakSi <= r)
					{
						force_mag = 1.0 - constant.KSiSi*(1.0 - r2);
					}
					else
					{
						dv.x = shVel[i].x - velMe.x;
						dv.y = shVel[i].y - velMe.y;
	 					dv.z = shVel[i].z - velMe.z;
						inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
						if(inout <= 0) 	force_mag  = 1.0 - constant.KSiSi*(1.0 - r2);
						else 		force_mag  = 1.0 - constant.KRSiSi*(1.0 - r2);
					}
				}
   	 			else if(test == 1) //Silicate iron force
				{
					if(1.0 <= r)
					{
						force_mag  = constant.GMassFeSi/r2;
					}
					else if(constant.ShellBreakFeSi1 <= r)
					{
						force_mag  = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
					}
					else if(constant.ShellBreakFeSi2 <= r)
					{
						dv.x = shVel[i].x - velMe.x;
						dv.y = shVel[i].y - velMe.y;
						dv.z = shVel[i].z - velMe.z;
						inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
						if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
		 				else 		force_mag = constant.GMassFeSi - constant.KRMix*(1.0 - r2);
					}
					else
					{
						dv.x = shVel[i].x - velMe.x;
						dv.y = shVel[i].y - velMe.y;
						dv.z = shVel[i].z - velMe.z;
						inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
						if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
						else 		force_mag = constant.GMassFeSi - constant.KRFeSi*(1.0 - r2);
		 			}
				}
				else //Iron iron force
				{
					if(1.0 <= r)
					{
						force_mag = constant.GMassFeFe/r2;
					}
					else if(constant.ShellBreakFe <= r)
					{
		    				force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
					}
					else
					{
						dv.x = shVel[i].x - velMe.x;
						dv.y = shVel[i].y - velMe.y;
						dv.z = shVel[i].z - velMe.z;
						inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
		   				if(inout <= 0) 	force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
		  				else 		force_mag = constant.GMassFeFe - constant.KRFeFe*(1.0 - r2);
					}
				}

				forceSum.x += force_mag*dp.x*invr;
				forceSum.y += force_mag*dp.y*invr;
				forceSum.z += force_mag*dp.z*invr;
			}
		}
		force[id].x = forceSum.x;
		force[id].y = forceSum.y;
		force[id].z = forceSum.z;
		__syncthreads();
	}
}

__global__ void moveBodiesSeperate(float4 *pos, float4 *vel, float4 * force, moveSeperateKernalConstantsStruct constant)
{
	float temp;
	int id;
	
    	id = threadIdx.x + blockDim.x*blockIdx.x;
    
	if(constant.boarder3 <= id) temp = constant.DtOverMassSi;
	else if(constant.boarder2 <= id) temp = constant.DtOverMassFe;
	else if(constant.boarder1 <= id) temp = constant.DtOverMassSi;
	else temp = constant.DtOverMassFe;

	vel[id].x += (force[id].x)*temp;
	vel[id].y += (force[id].y)*temp;
	vel[id].z += (force[id].z)*temp;

	pos[id].x += vel[id].x*constant.Dt;
	pos[id].y += vel[id].y*constant.Dt;
	pos[id].z += vel[id].z*constant.Dt;
}

__global__ void moveBodiesDampedSeperate(float4 *pos, float4 *vel, float4 * force, moveSeperateKernalConstantsStruct constant, float DampRateBody1, float DampRateBody2)
{
	float temp;
	float damp;
	int id;
	
    	id = threadIdx.x + blockDim.x*blockIdx.x;
 
	if(constant.boarder3 <= id) 
	{
		temp = constant.DtOverMassSi;
		damp = DampRateBody2;
	}
	else if(constant.boarder2 <= id) 
	{
		temp = constant.DtOverMassFe;
		damp = DampRateBody2;
	}
	else if(constant.boarder1 <= id) 
	{
		temp = constant.DtOverMassSi;
		damp = DampRateBody1;
	}
	else 
	{
		temp = constant.DtOverMassFe;
		damp = DampRateBody1;
	}
	
	vel[id].x += (force[id].x-damp*vel[id].x)*temp;
	vel[id].y += (force[id].y-damp*vel[id].y)*temp;
	vel[id].z += (force[id].z-damp*vel[id].z)*temp;

	pos[id].x += vel[id].x*constant.Dt;
	pos[id].y += vel[id].y*constant.Dt;
	pos[id].z += vel[id].z*constant.Dt;
}

__global__ void getForcesCollisionSingleGPU(float4 *pos, float4 *vel, float4 *force, forceCollisionKernalConstantsStruct constant)
{
	int id, ids;
	int inout;
	float4 forceSum;
	float4 posMe;
	float4 velMe;
	int test;
	float force_mag;
	float4 dp;
	float4 dv;
	float r2;
	float r;
	float invr;
	
	__shared__ float4 shPos[BLOCKSIZE];
	__shared__ float4 shVel[BLOCKSIZE];
	    
	id = threadIdx.x + blockDim.x*blockIdx.x;
	    
	forceSum.x = 0.0f;
	forceSum.y = 0.0f;
	forceSum.z = 0.0f;
		
	posMe.x = pos[id].x;
	posMe.y = pos[id].y;
	posMe.z = pos[id].z;
	
	velMe.x = vel[id].x;
	velMe.y = vel[id].y;
	velMe.z = vel[id].z;
		    
	for(int j=0; j < gridDim.x; j++)
	{
    		shPos[threadIdx.x] = pos[threadIdx.x + blockDim.x*j];
    		shVel[threadIdx.x] = vel[threadIdx.x + blockDim.x*j];
    		__syncthreads();
   
		for(int i=0; i < blockDim.x; i++)	
		{
			ids = i + blockDim.x*j;
		    	dp.x = shPos[i].x - posMe.x;
			dp.y = shPos[i].y - posMe.y;
			dp.z = shPos[i].z - posMe.z;
			r2 = dp.x*dp.x + dp.y*dp.y + dp.z*dp.z;
			r = sqrt(r2);
			if(id == ids) invr = 0;
			else invr = 1.0f/r;

		    	test = 0;
		    	if(id < constant.NFe) test = 1;
		    	if(ids < constant.NFe) test++;
	    
			if(test == 0) //Silicate silicate force
			{
				if(1.0 <= r)
				{
	    				force_mag = 1.0/r2; // G = 1 and mass of silicate elemnet =1
				}
				else if(constant.ShellBreakSi <= r)
				{
					force_mag = 1.0 - constant.KSiSi*(1.0 - r2); // because D = 1 G = 1 and mass of silicate = 1
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag  = 1.0 - constant.KSiSi*(1.0 - r2);
					else 		force_mag  = 1.0 - constant.KRSiSi*(1.0 - r2);
				}
	    		}
			else if(test == 1) //Silicate iron force
			{
				if(1.0 <= r)
				{
					force_mag  = constant.GMassFeSi/r2;
				}
				else if(constant.ShellBreakFeSi1 <= r)
				{
					force_mag  = constant.GMassFeSi -constant.KFeSi*(1.0 - r2);
				}
				else if(constant.ShellBreakFeSi2 <= r)
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
	 				else 		force_mag = constant.GMassFeSi - constant.KRMix*(1.0 - r2);
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
					else 		force_mag = constant.GMassFeSi - constant.KRFeSi*(1.0 - r2);
	 			}
			}
			else //Iron iron force
			{
				if(1.0 <= r)
				{
					force_mag = constant.GMassFeFe/r2;
				}
				else if(constant.ShellBreakFe <= r)
				{
	    			force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
	   				if(inout <= 0) 	force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
	  				else 		force_mag = constant.GMassFeFe - constant.KRFeFe*(1.0 - r2);
				}
			}

			forceSum.x += force_mag*dp.x*invr;
			forceSum.y += force_mag*dp.y*invr;
			forceSum.z += force_mag*dp.z*invr;
		}
		__syncthreads();
	}
	force[id].x = forceSum.x;
	force[id].y = forceSum.y;
	force[id].z = forceSum.z;
}

__global__ void moveBodiesCollisionSingleGPU(float4 *pos, float4 *vel, float4 * force, moveCollisionKernalConstantsStruct MoveCollisionConstant)
{
	float temp;
	int id;
    	id = threadIdx.x + blockDim.x*blockIdx.x;
    	if(id < MoveCollisionConstant.NFe) temp = MoveCollisionConstant.DtOverMassFe;
    	else temp = MoveCollisionConstant.DtOverMassSi;
	
	vel[id].x += (force[id].x)*temp;
	vel[id].y += (force[id].y)*temp;
	vel[id].z += (force[id].z)*temp;
	
	pos[id].x += vel[id].x*MoveCollisionConstant.Dt;
	pos[id].y += vel[id].y*MoveCollisionConstant.Dt;
	pos[id].z += vel[id].z*MoveCollisionConstant.Dt;
}

__global__ void getForcesCollisionDoubleGPU0(float4 *posFstHalf, float4 *posSndHalf, float4 *velFstHalf, float4 *velSndHalf,  float4 *forceFstHalf,  int N, forceCollisionKernalConstantsStruct constant)
{
	int id, ids;
	int i,j;
	int inout;
	float4 forceSum;
	float4 posMe;
	float4 velMe;
	int test;
	float force_mag;
	float4 dp;
	float4 dv;
	float r2;
	float r;
	float invr;

	__shared__ float4 shPos[BLOCKSIZE];
	__shared__ float4 shVel[BLOCKSIZE];

	id = threadIdx.x + blockDim.x*blockIdx.x;
	
	forceSum.x = 0.0f;
	forceSum.y = 0.0f;
	forceSum.z = 0.0f;

	posMe.x = posFstHalf[id].x;
	posMe.y = posFstHalf[id].y;
	posMe.z = posFstHalf[id].z;

	velMe.x = velFstHalf[id].x;
	velMe.y = velFstHalf[id].y;
	velMe.z = velFstHalf[id].z;
	    
	for(j=0;  j < gridDim.x;  j++)
	{
		shPos[threadIdx.x] = posFstHalf[threadIdx.x + blockDim.x*j];
		shVel[threadIdx.x]  = velFstHalf[threadIdx.x + blockDim.x*j];
		__syncthreads();

		for(i=0; i < blockDim.x; i++)	
		{
			ids = i + blockDim.x*j;
		    	dp.x = shPos[i].x - posMe.x;
			dp.y = shPos[i].y - posMe.y;
			dp.z = shPos[i].z - posMe.z;
			r2 = dp.x*dp.x + dp.y*dp.y + dp.z*dp.z;
			r = sqrt(r2);
			if(id == ids) invr = 0;
			else invr = 1.0f/r;

		    	test = 0;
		    	if(id < constant.NFe) test = 1;
		    	if(ids < constant.NFe) test++;   	
			    	
			if(test == 0) //Silicate silicate force
			{
				if(1.0 <= r)
				{
	    				force_mag = 1.0/r2; // G = 1 and mass of silicate elemnet =1
				}
				else if(constant.ShellBreakSi <= r)
				{
					force_mag = 1.0 - constant.KSiSi*(1.0 - r2); // because D = 1 G = 1 and mass of silicate = 1
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag  = 1.0 - constant.KSiSi*(1.0 - r2);
					else 		force_mag  = 1.0 - constant.KRSiSi*(1.0 - r2);
				}
	    		}
			else if(test == 1) //Silicate iron force
			{
				if(1.0 <= r)
				{
					force_mag  = constant.GMassFeSi/r2;
				}
				else if(constant.ShellBreakFeSi1 <= r)
				{
					force_mag  = constant.GMassFeSi -constant.KFeSi*(1.0 - r2);
				}
				else if(constant.ShellBreakFeSi2 <= r)
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
	 				else 		force_mag = constant.GMassFeSi - constant.KRMix*(1.0 - r2);
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
					else 		force_mag = constant.GMassFeSi - constant.KRFeSi*(1.0 - r2);
	 			}
			}
			else //Iron iron force
			{
				if(1.0 <= r)
				{
					force_mag = constant.GMassFeFe/r2;
				}
				else if(constant.ShellBreakFe <= r)
				{
	    			force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
	   				if(inout <= 0) 	force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
	  				else 		force_mag = constant.GMassFeFe - constant.KRFeFe*(1.0 - r2);
				}
			}

			forceSum.x += force_mag*dp.x*invr;
			forceSum.y += force_mag*dp.y*invr;
			forceSum.z += force_mag*dp.z*invr;
		}
		__syncthreads();
	}
	
	for(j=0; j < gridDim.x; j++)
	{
		shPos[threadIdx.x] = posSndHalf[threadIdx.x + blockDim.x*j];
		shVel[threadIdx.x] = velSndHalf[threadIdx.x + blockDim.x*j];
		__syncthreads();

		for(i=0; i < blockDim.x; i++)	
		{
			ids = i + blockDim.x*j;
		    	dp.x = shPos[i].x - posMe.x;
			dp.y = shPos[i].y - posMe.y;
			dp.z = shPos[i].z - posMe.z;
			r2 = dp.x*dp.x + dp.y*dp.y + dp.z*dp.z;
			r = sqrt(r2);
		 	invr = 1.0f/r;

		    	test = 0;
		    	if(id  < constant.NFe) test = 1;
		    	if(ids+(N/2) < constant.NFe) test++;   	
			    	
			if(test == 0) //Silicate silicate force
			{
				if(1.0 <= r)
				{
	    				force_mag = 1.0/r2; // G = 1 and mass of silicate elemnet =1
				}
				else if(constant.ShellBreakSi <= r)
				{
					force_mag = 1.0 - constant.KSiSi*(1.0 - r2); // because D = 1 G = 1 and mass of silicate = 1
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag  = 1.0 - constant.KSiSi*(1.0 - r2);
					else 		force_mag  = 1.0 - constant.KRSiSi*(1.0 - r2);
				}
	    		}
			else if(test == 1) //Silicate iron force
			{
				if(1.0 <= r)
				{
					force_mag  = constant.GMassFeSi/r2;
				}
				else if(constant.ShellBreakFeSi1 <= r)
				{
					force_mag  = constant.GMassFeSi -constant.KFeSi*(1.0 - r2);
				}
				else if(constant.ShellBreakFeSi2 <= r)
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
	 				else 		force_mag = constant.GMassFeSi - constant.KRMix*(1.0 - r2);
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
					else 		force_mag = constant.GMassFeSi - constant.KRFeSi*(1.0 - r2);
	 			}
			}
			else //Iron iron force
			{
				if(1.0 <= r)
				{
					force_mag = constant.GMassFeFe/r2;
				}
				else if(constant.ShellBreakFe <= r)
				{
	    			force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
	   				if(inout <= 0) 	force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
	  				else 		force_mag = constant.GMassFeFe - constant.KRFeFe*(1.0 - r2);
				}
			}

			forceSum.x += force_mag*dp.x*invr;
			forceSum.y += force_mag*dp.y*invr;
			forceSum.z += force_mag*dp.z*invr;
		}
		__syncthreads();
	}

	forceFstHalf[id].x = forceSum.x;
	forceFstHalf[id].y = forceSum.y;
	forceFstHalf[id].z = forceSum.z;
}

__global__ void getForcesCollisionDoubleGPU1(float4 *posFstHalf, float4 *posSndHalf, float4 *velFstHalf, float4 *velSndHalf,  float4 *forceSndHalf,  int N, forceCollisionKernalConstantsStruct constant)
{
	int id, ids;
	int i,j;
	int inout;
	float4 forceSum;
	float4 posMe;
	float4 velMe;
	int test;
	float force_mag;
	float4 dp;
	float4 dv;
	float r2;
	float r;
	float invr;

	__shared__ float4 shPos[BLOCKSIZE];
	__shared__ float4 shVel[BLOCKSIZE];

	id = threadIdx.x + blockDim.x*blockIdx.x;
	
	forceSum.x = 0.0f;
	forceSum.y = 0.0f;
	forceSum.z = 0.0f;

	posMe.x = posSndHalf[id].x;
	posMe.y = posSndHalf[id].y;
	posMe.z = posSndHalf[id].z;

	velMe.x = velSndHalf[id].x;
	velMe.y = velSndHalf[id].y;
	velMe.z = velSndHalf[id].z;
	    
	for(j=0;  j < gridDim.x;  j++)
	{
		shPos[threadIdx.x] = posFstHalf[threadIdx.x + blockDim.x*j];
		shVel[threadIdx.x]  = velFstHalf[threadIdx.x + blockDim.x*j];
		__syncthreads();

		for(i=0; i < blockDim.x; i++)	
		{
			ids = i + blockDim.x*j;
		    	dp.x = shPos[i].x - posMe.x;
			dp.y = shPos[i].y - posMe.y;
			dp.z = shPos[i].z - posMe.z;
			r2 = dp.x*dp.x + dp.y*dp.y + dp.z*dp.z;
			r = sqrt(r2);
			invr = 1.0f/r;

		    	test = 0;
		    	if(id + (N/2) < constant.NFe) test = 1;
		    	if(ids < constant.NFe) test++;   	
			    	
			if(test == 0) //Silicate silicate force
			{
				if(1.0 <= r)
				{
	    				force_mag = 1.0/r2; // G = 1 and mass of silicate elemnet =1
				}
				else if(constant.ShellBreakSi <= r)
				{
					force_mag = 1.0 - constant.KSiSi*(1.0 - r2); // because D = 1 G = 1 and mass of silicate = 1
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag  = 1.0 - constant.KSiSi*(1.0 - r2);
					else 		force_mag  = 1.0 - constant.KRSiSi*(1.0 - r2);
				}
	    		}
			else if(test == 1) //Silicate iron force
			{
				if(1.0 <= r)
				{
					force_mag  = constant.GMassFeSi/r2;
				}
				else if(constant.ShellBreakFeSi1 <= r)
				{
					force_mag  = constant.GMassFeSi -constant.KFeSi*(1.0 - r2);
				}
				else if(constant.ShellBreakFeSi2 <= r)
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
	 				else 		force_mag = constant.GMassFeSi - constant.KRMix*(1.0 - r2);
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
					else 		force_mag = constant.GMassFeSi - constant.KRFeSi*(1.0 - r2);
	 			}
			}
			else //Iron iron force
			{
				if(1.0 <= r)
				{
					force_mag = constant.GMassFeFe/r2;
				}
				else if(constant.ShellBreakFe <= r)
				{
	    			force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
	   				if(inout <= 0) 	force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
	  				else 		force_mag = constant.GMassFeFe - constant.KRFeFe*(1.0 - r2);
				}
			}

			forceSum.x += force_mag*dp.x*invr;
			forceSum.y += force_mag*dp.y*invr;
			forceSum.z += force_mag*dp.z*invr;
		}
		__syncthreads();
	}
	
	for(j=0; j < gridDim.x; j++)
	{
		shPos[threadIdx.x] = posSndHalf[threadIdx.x + blockDim.x*j];
		shVel[threadIdx.x] = velSndHalf[threadIdx.x + blockDim.x*j];
		__syncthreads();

		for(i=0; i < blockDim.x; i++)	
		{
			ids = i + blockDim.x*j ;
		    	dp.x = shPos[i].x - posMe.x;
			dp.y = shPos[i].y - posMe.y;
			dp.z = shPos[i].z - posMe.z;
			r2 = dp.x*dp.x + dp.y*dp.y + dp.z*dp.z;
			r = sqrt(r2);
			if(id == ids) invr = 0;
			else invr = 1.0f/r;

		    	test = 0;
		    	if(id + (N/2) < constant.NFe) test = 1;
		    	if(ids+(N/2) < constant.NFe) test++;   	
			    	
			if(test == 0) //Silicate silicate force
			{
				if(1.0 <= r)
				{
	    				force_mag = 1.0/r2; // G = 1 and mass of silicate elemnet =1
				}
				else if(constant.ShellBreakSi <= r)
				{
					force_mag = 1.0 - constant.KSiSi*(1.0 - r2); // because D = 1 G = 1 and mass of silicate = 1
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag  = 1.0 - constant.KSiSi*(1.0 - r2);
					else 		force_mag  = 1.0 - constant.KRSiSi*(1.0 - r2);
				}
	    		}
			else if(test == 1) //Silicate iron force
			{
				if(1.0 <= r)
				{
					force_mag  = constant.GMassFeSi/r2;
				}
				else if(constant.ShellBreakFeSi1 <= r)
				{
					force_mag  = constant.GMassFeSi -constant.KFeSi*(1.0 - r2);
				}
				else if(constant.ShellBreakFeSi2 <= r)
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
	 				else 		force_mag = constant.GMassFeSi - constant.KRMix*(1.0 - r2);
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
					if(inout <= 0) 	force_mag = constant.GMassFeSi - constant.KFeSi*(1.0 - r2);
					else 		force_mag = constant.GMassFeSi - constant.KRFeSi*(1.0 - r2);
	 			}
			}
			else //Iron iron force
			{
				if(1.0 <= r)
				{
					force_mag = constant.GMassFeFe/r2;
				}
				else if(constant.ShellBreakFe <= r)
				{
	    			force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
				}
				else
				{
					dv.x = shVel[i].x - velMe.x;
					dv.y = shVel[i].y - velMe.y;
					dv.z = shVel[i].z - velMe.z;
					inout = dp.x*dv.x + dp.y*dv.y + dp.z*dv.z;
	   				if(inout <= 0) 	force_mag = constant.GMassFeFe - constant.KFeFe*(1.0 - r2);
	  				else 		force_mag = constant.GMassFeFe - constant.KRFeFe*(1.0 - r2);
				}
			}

			forceSum.x += force_mag*dp.x*invr;
			forceSum.y += force_mag*dp.y*invr;
			forceSum.z += force_mag*dp.z*invr;
		}
		__syncthreads();
	}

	forceSndHalf[id].x = forceSum.x;
	forceSndHalf[id].y = forceSum.y;
	forceSndHalf[id].z = forceSum.z;
}
		
__global__ void moveBodiesCollisionDoubleGPU0(float4 *posFstHalf,   float4 *velFstHalf,  float4 * forceFstHalf,  int N, moveCollisionKernalConstantsStruct constant)
{
	float temp;
	int id;
	id = threadIdx.x + blockDim.x*blockIdx.x;
	if(id < constant.NFe) temp = constant.DtOverMassFe;
    	else temp = constant.DtOverMassSi;
	
	velFstHalf[id].x += (forceFstHalf[id].x)*temp;
	velFstHalf[id].y += (forceFstHalf[id].y)*temp;
	velFstHalf[id].z += (forceFstHalf[id].z)*temp;

	posFstHalf[id].x += velFstHalf[id].x*constant.Dt;
	posFstHalf[id].y += velFstHalf[id].y*constant.Dt;
	posFstHalf[id].z += velFstHalf[id].z*constant.Dt;
}

__global__ void moveBodiesCollisionDoubleGPU1(float4 *posSndHalf,  float4 *velSndHalf,  float4 * forceSndHalf,  int N, moveCollisionKernalConstantsStruct constant)
{
	float temp;
	int id;
	id = threadIdx.x + blockDim.x*blockIdx.x;
	if(id + (N/2) < constant.NFe) temp = constant.DtOverMassFe;
    	else temp = constant.DtOverMassSi;
	
	velSndHalf[id].x += (forceSndHalf[id].x)*temp;
	velSndHalf[id].y += (forceSndHalf[id].y)*temp;
	velSndHalf[id].z += (forceSndHalf[id].z)*temp;

	posSndHalf[id].x += velSndHalf[id].x*constant.Dt;
	posSndHalf[id].y += velSndHalf[id].y*constant.Dt;
	posSndHalf[id].z += velSndHalf[id].z*constant.Dt;
}

float3 getCenterOfMassSeperate(int scope)
{
	float totalMass;
	float assumeZero = 0.0000001;
	float3 centerOfMass;
	
	centerOfMass.x = 0.0f;
	centerOfMass.y = 0.0f;
	centerOfMass.z = 0.0f;
	
	if(scope == 0) //entire system
	{
		totalMass = MassOfBody1 + MassOfBody2;
		if(totalMass < assumeZero) return(centerOfMass);
		
		for(int i = 0; i < NFe1; i++)
		{
	    		centerOfMass.x += Pos[i].x*MassFe;
			centerOfMass.y += Pos[i].y*MassFe;
			centerOfMass.z += Pos[i].z*MassFe;
		}
		for(int i = NFe1; i < NFe1 + NSi1; i++)
		{
	    		centerOfMass.x += Pos[i].x*MassSi;
			centerOfMass.y += Pos[i].y*MassSi;
			centerOfMass.z += Pos[i].z*MassSi;
		}
		for(int i = NFe1 + NSi1; i < NFe1 + NSi1 + NFe2; i++)
		{
	    		centerOfMass.x += Pos[i].x*MassFe;
			centerOfMass.y += Pos[i].y*MassFe;
			centerOfMass.z += Pos[i].z*MassFe;
		}
		for(int i = NFe1 + NSi1 + NFe2; i < N; i++)
		{
	    		centerOfMass.x += Pos[i].x*MassSi;
			centerOfMass.y += Pos[i].y*MassSi;
			centerOfMass.z += Pos[i].z*MassSi;
		}
	}
	else if(scope == 1) //body1
	{
		totalMass = MassOfBody1;
		if(totalMass < assumeZero) return(centerOfMass);
		
		for(int i = 0; i < NFe1; i++)
		{
	    		centerOfMass.x += Pos[i].x*MassFe;
			centerOfMass.y += Pos[i].y*MassFe;
			centerOfMass.z += Pos[i].z*MassFe;
		}
		for(int i = NFe1; i < NFe1 + NSi1; i++)
		{
	    		centerOfMass.x += Pos[i].x*MassSi;
			centerOfMass.y += Pos[i].y*MassSi;
			centerOfMass.z += Pos[i].z*MassSi;
		}
	}
	else if(scope == 2) //body2
	{
		totalMass = MassOfBody2;
		if(totalMass < assumeZero) return(centerOfMass);
		
		for(int i = NFe1 + NSi1; i < NFe1 + NSi1 + NFe2; i++)
		{
	    		centerOfMass.x += Pos[i].x*MassFe;
			centerOfMass.y += Pos[i].y*MassFe;
			centerOfMass.z += Pos[i].z*MassFe;
		}
		for(int i = NFe1 + NSi1 + NFe2; i < N; i++)
		{
	    		centerOfMass.x += Pos[i].x*MassSi;
			centerOfMass.y += Pos[i].y*MassSi;
			centerOfMass.z += Pos[i].z*MassSi;
		}
	}
	else
	{
		printf("\nTSU Error: In getCenterOfMassSeperate function scope invalid\n");
		exit(0);
	}
	
	centerOfMass.x /= totalMass;
	centerOfMass.y /= totalMass;
	centerOfMass.z /= totalMass;
	return(centerOfMass);
}

float3 getLinearVelocitySeperate(int scope)
{
	double totalMass;
	float assumeZero = 0.0000001;
	float3 linearVelocity;
	
	linearVelocity.x = 0.0f;
	linearVelocity.y = 0.0f;
	linearVelocity.z = 0.0f;
	
	if(scope == 0) //Entire system
	{
		totalMass = MassOfBody1 + MassOfBody2;
		if(totalMass < assumeZero) return(linearVelocity);
		
		for(int i = 0; i < NFe1; i++)
		{
	    		linearVelocity.x += Vel[i].x*MassFe;
			linearVelocity.y += Vel[i].y*MassFe;
			linearVelocity.z += Vel[i].z*MassFe;
		}
		for(int i = NFe1; i < NFe1 + NSi1; i++)
		{
	    		linearVelocity.x += Vel[i].x*MassSi;
			linearVelocity.y += Vel[i].y*MassSi;
			linearVelocity.z += Vel[i].z*MassSi;
		}

		for(int i = NFe1 + NSi1; i < NFe1 + NSi1 + NFe2; i++)
		{
	    		linearVelocity.x += Vel[i].x*MassFe;
			linearVelocity.y += Vel[i].y*MassFe;
			linearVelocity.z += Vel[i].z*MassFe;
		}
		for(int i = NFe1 + NSi1 + NFe2; i < N; i++)
		{
	    		linearVelocity.x += Vel[i].x*MassSi;
			linearVelocity.y += Vel[i].y*MassSi;
			linearVelocity.z += Vel[i].z*MassSi;
		}
	}
	else if(scope == 1) //body1
	{
		totalMass = MassOfBody1;
		if(totalMass < assumeZero) return(linearVelocity);
		
		for(int i = 0; i < NFe1; i++)
		{
	    		linearVelocity.x += Vel[i].x*MassFe;
			linearVelocity.y += Vel[i].y*MassFe;
			linearVelocity.z += Vel[i].z*MassFe;
		}
		for(int i = NFe1; i < NFe1 + NSi1; i++)
		{
	    		linearVelocity.x += Vel[i].x*MassSi;
			linearVelocity.y += Vel[i].y*MassSi;
			linearVelocity.z += Vel[i].z*MassSi;
		}
	}
	else if (scope == 2) //body2
	{
		totalMass = MassOfBody2;
		if(totalMass < assumeZero) return(linearVelocity);
		
		for(int i = NFe1 + NSi1; i < NFe1 + NSi1 + NFe2; i++)
		{
	    		linearVelocity.x += Vel[i].x*MassFe;
			linearVelocity.y += Vel[i].y*MassFe;
			linearVelocity.z += Vel[i].z*MassFe;
		}
		for(int i = NFe1 + NSi1 + NFe2; i < N; i++)
		{
	    		linearVelocity.x += Vel[i].x*MassSi;
			linearVelocity.y += Vel[i].y*MassSi;
			linearVelocity.z += Vel[i].z*MassSi;
		}
	}
	else
	{
		printf("\nTSU Error: In getLinearVelocitySeperate function scope invalid\n");
		exit(0);
	}
	
	linearVelocity.x /= totalMass;
	linearVelocity.y /= totalMass;
	linearVelocity.z /= totalMass;
	return(linearVelocity);
}

float3 getAngularMomentumSeperate(int scope, float3 center, float3 velocity)
{
	float3 angularMomentum;
	float3 r;
	float3 v;
	
	angularMomentum.x = 0.0f;
	angularMomentum.y = 0.0f;
	angularMomentum.z = 0.0f;
	
	if(scope == 0) //entire system
	{	
		for(int i = 0; i < NFe1; i++)
		{
			r.x = Pos[i].x - center.x;
			r.y = Pos[i].y - center.y;
			r.z = Pos[i].z - center.z;
		
			v.x = Vel[i].x - velocity.x;
			v.y = Vel[i].y - velocity.y;
			v.z = Vel[i].z - velocity.z;
		
			angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassFe;
			angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassFe;
			angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassFe;
		}
		for(int i = NFe1; i < NFe1 + NSi1; i++)
		{
			r.x = Pos[i].x - center.x;
			r.y = Pos[i].y - center.y;
			r.z = Pos[i].z - center.z;
		
			v.x = Vel[i].x - velocity.x;
			v.y = Vel[i].y - velocity.y;
			v.z = Vel[i].z - velocity.z;
		
			angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassSi;
			angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassSi;
			angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassSi;
		}
		for(int i = NFe1 + NSi1; i < NFe1 + NSi1 + NFe2; i++)
		{
			r.x = Pos[i].x - center.x;
			r.y = Pos[i].y - center.y;
			r.z = Pos[i].z - center.z;
		
			v.x = Vel[i].x - velocity.x;
			v.y = Vel[i].y - velocity.y;
			v.z = Vel[i].z - velocity.z;
		
			angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassFe;
			angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassFe;
			angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassFe;
		}
		for(int i = NFe1 + NSi1 + NFe2; i < N; i++)
		{
			r.x = Pos[i].x - center.x;
			r.y = Pos[i].y - center.y;
			r.z = Pos[i].z - center.z;
		
			v.x = Vel[i].x - velocity.x;
			v.y = Vel[i].y - velocity.y;
			v.z = Vel[i].z - velocity.z;
		
			angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassSi;
			angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassSi;
			angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassSi;
		}
	}
	else if(scope == 1) //body1
	{	
		for(int i = 0; i < NFe1; i++)
		{
			r.x = Pos[i].x - center.x;
			r.y = Pos[i].y - center.y;
			r.z = Pos[i].z - center.z;
		
			v.x = Vel[i].x - velocity.x;
			v.y = Vel[i].y - velocity.y;
			v.z = Vel[i].z - velocity.z;
		
			angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassFe;
			angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassFe;
			angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassFe;
		}
		for(int i = NFe1; i < NFe1 + NSi1; i++)
		{
			r.x = Pos[i].x - center.x;
			r.y = Pos[i].y - center.y;
			r.z = Pos[i].z - center.z;
		
			v.x = Vel[i].x - velocity.x;
			v.y = Vel[i].y - velocity.y;
			v.z = Vel[i].z - velocity.z;
		
			angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassSi;
			angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassSi;
			angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassSi;
		}
	}
	else if(scope == 2) //body2
	{
		for(int i = NFe1 + NSi1; i < NFe1 + NSi1 + NFe2; i++)
		{
			r.x = Pos[i].x - center.x;
			r.y = Pos[i].y - center.y;
			r.z = Pos[i].z - center.z;
		
			v.x = Vel[i].x - velocity.x;
			v.y = Vel[i].y - velocity.y;
			v.z = Vel[i].z - velocity.z;
		
			angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassFe;
			angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassFe;
			angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassFe;
		}
		for(int i = NFe1 + NSi1 + NFe2; i < N; i++)
		{
			r.x = Pos[i].x - center.x;
			r.y = Pos[i].y - center.y;
			r.z = Pos[i].z - center.z;
		
			v.x = Vel[i].x - velocity.x;
			v.y = Vel[i].y - velocity.y;
			v.z = Vel[i].z - velocity.z;
		
			angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassSi;
			angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassSi;
			angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassSi;
		}
	}
	else
	{
		printf("\nTSU Error: In getAngularMomentumSeperate function scope invalid\n");
		exit(0);
	}

	return(angularMomentum);
}

void setBodyPositionSeperate(int bodyId, float x, float y, float z)
{
	int	start, stop;
	
	if(bodyId == 1)
	{
		start = 0;
		stop = NFe1 + NSi1;
	}
	else if(bodyId == 2)
	{
		start = NFe1 + NSi1;
		stop = N;
	}
	else 
	{
		printf("\nTSU Error: in setBodyPositionSeperate function bodyId invalid\n");
		exit(0);
	}
	
	float3 centerOfMass = getCenterOfMassSeperate(bodyId); 
	
	for(int i = start; i < stop; i++)
	{
		Pos[i].x += x - centerOfMass.x;
		Pos[i].y += y - centerOfMass.y;
		Pos[i].z += z - centerOfMass.z;
	}	
}

void setBodyVelocitySeperate(int bodyId, float vx, float vy, float vz)
{
	int	start, stop;
	
	if(bodyId == 1)
	{
		start = 0;
		stop = NFe1 + NSi1;
	}
	else if(bodyId == 2)
	{
		start = NFe1 + NSi1;
		stop = N;
	}
	else 
	{
		printf("\nTSU Error: in setBodyVelocitySeperate invalid bodyId\n");
		exit(0);
	}
	
	float3 RandomlinearVelocity = getLinearVelocitySeperate(bodyId); 
	
	for(int i = start; i < stop; i++)
	{
		Vel[i].x += vx - RandomlinearVelocity.x;
		Vel[i].y += vy - RandomlinearVelocity.y;
		Vel[i].z += vz - RandomlinearVelocity.z;
	}	
}

void spinBodySeperate(int bodyId, float4 spinVector)
{
	float3 	r;  			//vector from center of mass to the position vector
	float3 	centerOfMass;
	float3	n;			//Unit vector perpendicular to the plane of spin
	float 	mag;
	float 	assumeZero = 0.0000001;
	int	start, stop;
	
	if(bodyId == 1)
	{
		start = 0;
		stop = NFe1 + NSi1;
	}
	else
	{
		start = NFe1 + NSi1;
		stop = N;
	}
	
	//Making sure the spin vector is a unit vector
	mag = sqrt(spinVector.x*spinVector.x + spinVector.y*spinVector.y + spinVector.z*spinVector.z);
	if(assumeZero < mag)
	{
		spinVector.x /= mag;
		spinVector.y /= mag;
		spinVector.z /= mag;
	}
	else 
	{
		printf("\nTSU Error: In spinBodySeperate. The spin direction vector is zero.\n");
		exit(0);
	}
	
	centerOfMass = getCenterOfMassSeperate(bodyId);
	for(int i = start; i < stop; i++)
	{
		//Creating a vector from the center of mass to the point
		r.x = Pos[i].x - centerOfMass.x;
		r.y = Pos[i].y - centerOfMass.y;
		r.z = Pos[i].z - centerOfMass.z;
		float magsquared = r.x*r.x + r.y*r.y + r.z*r.z;
		float spinDota = spinVector.x*r.x + spinVector.y*r.y + spinVector.z*r.z;
		float perpendicularDistance = sqrt(magsquared - spinDota*spinDota);
		float perpendicularVelocity = spinVector.w*2.0*Pi*perpendicularDistance;
		
		//finding unit vector perpendicular to both the position vector and the spin vector
		n.x =  (spinVector.y*r.z - spinVector.z*r.y);
		n.y = -(spinVector.x*r.z - spinVector.z*r.x);
		n.z =  (spinVector.x*r.y - spinVector.y*r.x);
		mag = sqrt(n.x*n.x + n.y*n.y + n.z*n.z);
		if(mag != 0.0)
		{
			n.x /= mag;
			n.y /= mag;
			n.z /= mag;
				
			//Spining the element
			Vel[i].x += perpendicularVelocity*n.x;
			Vel[i].y += perpendicularVelocity*n.y;
			Vel[i].z += perpendicularVelocity*n.z;
		}
	}		
}

double vectorMagnitude(float3 v)
{
	return(sqrt(v.x*v.x + v.y*v.y + v.z*v.z));
}

void recordStatsOfCreatedBodies()
{
	float radiusOfBody;
	float massOfBody;
	float3 r;
	double mag, d;
	
	float3 centerOfMass;
	float3 linearVelocity;
	float3 angularMomentum;
	
	double lengthConvertion = UnitLength;
	double massConvertion = UnitMass;
	double velocityConvertion = UnitLength/UnitTime;
	double AngularMomentumConvertion = (UnitMass*UnitLength*UnitLength)/(UnitTime);
	
	fprintf(RunStatsFile, "\n\n\n*****************************************************************************************************\n");
	fprintf(RunStatsFile, "\nThe follow are the statistics of the system right before they are released to collide in real world units\n");

	fprintf(RunStatsFile, "\n\n***** Stats for the univeral system *****\n");
	centerOfMass = getCenterOfMassSeperate(0);
	fprintf(RunStatsFile, "\nThe center of mass = (%f, %f, %f) Kilometers from (0, 0, 0)\n", centerOfMass.x*lengthConvertion, centerOfMass.y*lengthConvertion, centerOfMass.z*lengthConvertion);
	
	linearVelocity = getLinearVelocitySeperate(0);
	fprintf(RunStatsFile, "\nThe average linear velocity = (%f, %f, %f)", linearVelocity.x*velocityConvertion, linearVelocity.y*velocityConvertion, linearVelocity.z*velocityConvertion);
	mag = vectorMagnitude(linearVelocity);
	fprintf(RunStatsFile, "\nThe magitude of the avergae linear velocity = %f Kilometers/second\n", mag*velocityConvertion);
	
	angularMomentum = getAngularMomentumSeperate(0, getCenterOfMassSeperate(0), getLinearVelocitySeperate(0));
	fprintf(RunStatsFile, "\nThe angular momentum = (%e, %e, %e)", angularMomentum.x*AngularMomentumConvertion, angularMomentum.y*AngularMomentumConvertion, angularMomentum.z*AngularMomentumConvertion);
	mag = vectorMagnitude(angularMomentum);
	fprintf(RunStatsFile, "\nThe magitude of the angular momentum = %e Kilograms*kilometers*kilometers/second\n", mag*AngularMomentumConvertion);
	
	fprintf(RunStatsFile, "\n\n***** Stats for Body1 *****\n");
	centerOfMass = getCenterOfMassSeperate(1);
	
	radiusOfBody = 0.0;
	massOfBody = 0.0;
	for(int i = 0; i < NFe1; i++)
	{
		r.x = Pos[i].x - centerOfMass.x;
		r.y = Pos[i].y - centerOfMass.y;
		r.z = Pos[i].z - centerOfMass.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(d > radiusOfBody) radiusOfBody = d;
		
		massOfBody += MassFe;
	}
	
	for(int i = NFe1; i < NSi1; i++)
	{
		r.x = Pos[i].x - centerOfMass.x;
		r.y = Pos[i].y - centerOfMass.y;
		r.z = Pos[i].z - centerOfMass.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(d > radiusOfBody) radiusOfBody = d;
		
		massOfBody += MassSi;
	}
	
	fprintf(RunStatsFile, "\nMass =  %e Kilograms\n", massOfBody*massConvertion);
	fprintf(RunStatsFile, "\nRadius =  %f Kilometers\n", radiusOfBody*lengthConvertion);
	
	fprintf(RunStatsFile, "\nThe center of mass = (%f, %f, %f) Kilometers from (0, 0, 0)\n", centerOfMass.x*lengthConvertion, centerOfMass.y*lengthConvertion, centerOfMass.z*lengthConvertion);
	
	linearVelocity = getLinearVelocitySeperate(1);
	fprintf(RunStatsFile, "\nThe average linear velocity = (%f, %f, %f)", linearVelocity.x*velocityConvertion, linearVelocity.y*velocityConvertion, linearVelocity.z*velocityConvertion);
	mag = vectorMagnitude(linearVelocity);
	fprintf(RunStatsFile, "\nThe magitude of the avergae linear velocity = %f Kilometers/second\n", mag*velocityConvertion);
	
	angularMomentum = getAngularMomentumSeperate(1, getCenterOfMassSeperate(1), getLinearVelocitySeperate(1));
	fprintf(RunStatsFile, "\nThe angular momentum = (%e, %e, %e)", angularMomentum.x*AngularMomentumConvertion, angularMomentum.y*AngularMomentumConvertion, angularMomentum.z*AngularMomentumConvertion);
	mag = vectorMagnitude(angularMomentum);
	fprintf(RunStatsFile, "\nThe magitude of the angular momentum = %e Kilograms*kilometers*kilometers/second\n", mag*AngularMomentumConvertion);
	
	fprintf(RunStatsFile, "\n\n***** Stats for Body2 *****\n");
	centerOfMass = getCenterOfMassSeperate(2);
	
	radiusOfBody = 0.0;
	massOfBody = 0.0;
	for(int i = NFe1 + NSi1; i < NFe1 + NSi1 + NFe2; i++)
	{
		r.x = Pos[i].x - centerOfMass.x;
		r.y = Pos[i].y - centerOfMass.y;
		r.z = Pos[i].z - centerOfMass.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(d > radiusOfBody) radiusOfBody = d;
		
		massOfBody += MassFe;
	}
	
	for(int i = NFe1 + NSi1 + NFe2; i < N; i++)
	{
		r.x = Pos[i].x - centerOfMass.x;
		r.y = Pos[i].y - centerOfMass.y;
		r.z = Pos[i].z - centerOfMass.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(d > radiusOfBody) radiusOfBody = d;
		
		massOfBody += MassSi;
	}
	
	fprintf(RunStatsFile, "\nMass =  %e Kilograms\n", massOfBody*massConvertion);
	fprintf(RunStatsFile, "\nRadius =  %f Kilometers\n", radiusOfBody*lengthConvertion);
	
	fprintf(RunStatsFile, "\nThe center of mass = (%f, %f, %f) Kilometers from (0, 0, 0)\n", centerOfMass.x*lengthConvertion, centerOfMass.y*lengthConvertion, centerOfMass.z*lengthConvertion);
	
	linearVelocity = getLinearVelocitySeperate(2);
	fprintf(RunStatsFile, "\nThe average linear velocity = (%f, %f, %f)", linearVelocity.x*velocityConvertion, linearVelocity.y*velocityConvertion, linearVelocity.z*velocityConvertion);
	mag = vectorMagnitude(linearVelocity);
	fprintf(RunStatsFile, "\nThe magitude of the avergae linear velocity = %f Kilometers/second\n", mag*velocityConvertion);
	
	angularMomentum = getAngularMomentumSeperate(2, getCenterOfMassSeperate(2), getLinearVelocitySeperate(2));
	fprintf(RunStatsFile, "\nThe angular momentum = (%e, %e, %e)", angularMomentum.x*AngularMomentumConvertion, angularMomentum.y*AngularMomentumConvertion, angularMomentum.z*AngularMomentumConvertion);
	mag = vectorMagnitude(angularMomentum);
	fprintf(RunStatsFile, "\nThe magitude of the angular momentum = %e Kilograms*kilometers*kilometers/second\n", mag*AngularMomentumConvertion);
}

void recordStartPosVelOfCreatedBodiesSeperate()
{
	hipMemcpy( Pos, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpy Pos1");
	hipMemcpy( Vel, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpy Vel");
	
	fwrite(Pos, sizeof(float4), N, StartPosAndVelFile);
	fwrite(Vel, sizeof(float4), N, StartPosAndVelFile);
}

int findEarthAndMoon()
{
	int groupId[N], used[N];
	float mag, dx, dy, dz;
	float touch = Diameter*1.5;
	int groupNumber, numberOfGroups;
	int k;
	
	for(int i = 0; i < N; i++)
	{
		groupId[i] = -1;
		used[i] = 0;
	}
	
	groupNumber = 0;
	for(int i = 0; i < N; i++)
	{
		if(groupId[i] == -1)
		{
			groupId[i] = groupNumber;
			//find all from this group
			k = i;
			while(k < N)
			{
				if(groupId[k] == groupNumber && used[k] == 0)
				{
					for(int j = i; j < N; j++)
					{
						dx = Pos[k].x - Pos[j].x;
						dy = Pos[k].y - Pos[j].y;
						dz = Pos[k].z - Pos[j].z;
						mag = sqrt(dx*dx + dy*dy + dz*dz);
						if(mag < touch)
						{
							groupId[j] = groupNumber;
						}
					}
					used[k] = 1;
					k = i;
				}
				else k++;	
			}
			
		}
		groupNumber++;
	}
	numberOfGroups = groupNumber;
	
	if(numberOfGroups == 1)
	{
		printf("\n No Moon found\n");
	}
	
	int count;
	int *groupSize = (int *)malloc(numberOfGroups*sizeof(int));
	for(int i = 0; i < numberOfGroups; i++)
	{
		count = 0;
		for(int j = 0; j < N; j++)
		{
			if(i == groupId[j]) count++;
		}
		groupSize[i] = count;
	}
	
	int earthGroupId = -1;
	NumberOfEarthElements = 0;
	for(int i = 0; i < numberOfGroups; i++)
	{
		if(groupSize[i] > NumberOfEarthElements)
		{
			NumberOfEarthElements = groupSize[i];
			earthGroupId = i;
		}
	}
	
	int moonGroupId = -1;
	NumberOfMoonElements = 0;
	for(int i = 0; i < numberOfGroups; i++)
	{
		if(groupSize[i] > NumberOfMoonElements && i != earthGroupId)
		{
			NumberOfMoonElements = groupSize[i];
			moonGroupId = i;
		}
	}
	
	free(groupSize);
	EarthIndex = (int *)malloc(NumberOfEarthElements*sizeof(int));
	MoonIndex = (int *)malloc(NumberOfMoonElements*sizeof(int));
	
	int earthCount = 0;
	int moonCount = 0;
	for(int j = 0; j < N; j++)
	{
		if(groupId[j] == earthGroupId) 
		{
			EarthIndex[earthCount] = j;
			earthCount++;
		}
		else if(groupId[j] == moonGroupId)  
		{
			MoonIndex[moonCount] = j;
			moonCount++;
		}
	}
	
	return(1);	
}

float getMassCollision(int scope)
{
	float mass = 0.0;
	
	if(scope == 0) // entire system
	{
		for(int i = 0; i < N; i++)
		{
			if(i < NFe) mass += MassFe;
			else mass += MassSi;
		}
	}
	else if(scope == 1) // earth-moon syatem
	{
		for(int i = 0; i < NumberOfEarthElements; i++)
		{
			if(EarthIndex[i] < NFe) mass += MassFe;
			else mass += MassSi;
		}
		for(int i = 0; i < NumberOfMoonElements; i++)
		{
			if(MoonIndex[i] < NFe) mass += MassFe;
			else mass += MassSi;
		}
	}
	else if(scope == 2) // earth
	{
		for(int i = 0; i < NumberOfEarthElements; i++)
		{
			if(EarthIndex[i] < NFe) mass += MassFe;
			else mass += MassSi;
		}
	}
	else if(scope == 3) // moon
	{
		for(int i = 0; i < NumberOfMoonElements; i++)
		{
			if(MoonIndex[i] < NFe) mass += MassFe;
			else mass += MassSi;
		}
	}
	else
	{
		printf("\nTSU Error: In getMassCollision function bodyId invalid\n");
		exit(0);
	}
	return(mass);
}

float3 getCenterOfMassCollision(int scope)
{
	float totalMass;
	float3 centerOfMass;
	centerOfMass.x = 0.0;
	centerOfMass.y = 0.0;
	centerOfMass.z = 0.0;
	
	if(scope == 0) // Entire System
	{
		for(int i = 0; i < N; i++)
		{
			if(i < NFe)
			{
		    	centerOfMass.x += Pos[i].x*MassFe;
				centerOfMass.y += Pos[i].y*MassFe;
				centerOfMass.z += Pos[i].z*MassFe;
			}
			else
			{
		    	centerOfMass.x += Pos[i].x*MassSi;
				centerOfMass.y += Pos[i].y*MassSi;
				centerOfMass.z += Pos[i].z*MassSi;
			}
		}
		totalMass = getMassCollision(0);
		centerOfMass.x /= totalMass;
		centerOfMass.y /= totalMass;
		centerOfMass.z /= totalMass;
	}
	else if(scope == 1) // Earth-Moon System
	{
		for(int i = 0; i < NumberOfEarthElements; i++)
		{
			if(EarthIndex[i] < NFe)
			{
		    		centerOfMass.x += Pos[EarthIndex[i]].x*MassFe;
				centerOfMass.y += Pos[EarthIndex[i]].y*MassFe;
				centerOfMass.z += Pos[EarthIndex[i]].z*MassFe;
			}
			else
			{
		    		centerOfMass.x += Pos[EarthIndex[i]].x*MassSi;
				centerOfMass.y += Pos[EarthIndex[i]].y*MassSi;
				centerOfMass.z += Pos[EarthIndex[i]].z*MassSi;
			}
		}
		for(int i = 0; i < NumberOfMoonElements; i++)
		{
			if(MoonIndex[i] < NFe)
			{
		    		centerOfMass.x += Pos[MoonIndex[i]].x*MassFe;
				centerOfMass.y += Pos[MoonIndex[i]].y*MassFe;
				centerOfMass.z += Pos[MoonIndex[i]].z*MassFe;
			}
			else
			{
		    		centerOfMass.x += Pos[MoonIndex[i]].x*MassSi;
				centerOfMass.y += Pos[MoonIndex[i]].y*MassSi;
				centerOfMass.z += Pos[MoonIndex[i]].z*MassSi;
			}
		}
		totalMass = getMassCollision(1);
		centerOfMass.x /= totalMass;
		centerOfMass.y /= totalMass;
		centerOfMass.z /= totalMass;
		
	}
	else if(scope == 2) // Earth
	{
		for(int i = 0; i < NumberOfEarthElements; i++)
		{
			if(EarthIndex[i] < NFe)
			{
		    		centerOfMass.x += Pos[EarthIndex[i]].x*MassFe;
				centerOfMass.y += Pos[EarthIndex[i]].y*MassFe;
				centerOfMass.z += Pos[EarthIndex[i]].z*MassFe;
			}
			else
			{
		    		centerOfMass.x += Pos[EarthIndex[i]].x*MassSi;
				centerOfMass.y += Pos[EarthIndex[i]].y*MassSi;
				centerOfMass.z += Pos[EarthIndex[i]].z*MassSi;
			}
		}
		totalMass = getMassCollision(2);
		centerOfMass.x /= totalMass;
		centerOfMass.y /= totalMass;
		centerOfMass.z /= totalMass;
	}
	else if(scope == 3) // Moon
	{
		for(int i = 0; i < NumberOfMoonElements; i++)
		{
			if(MoonIndex[i] < NFe)
			{
		    		centerOfMass.x += Pos[MoonIndex[i]].x*MassFe;
				centerOfMass.y += Pos[MoonIndex[i]].y*MassFe;
				centerOfMass.z += Pos[MoonIndex[i]].z*MassFe;
			}
			else
			{
		    		centerOfMass.x += Pos[MoonIndex[i]].x*MassSi;
				centerOfMass.y += Pos[MoonIndex[i]].y*MassSi;
				centerOfMass.z += Pos[MoonIndex[i]].z*MassSi;
			}
		}
		totalMass = getMassCollision(3);
		centerOfMass.x /= totalMass;
		centerOfMass.y /= totalMass;
		centerOfMass.z /= totalMass;
	}
	else
	{
		printf("\nTSU Error: In getCenterOfMassCollision function scope invalid\n");
		exit(0);
	}
	return(centerOfMass);
}

float3 getLinearVelocityCollision(int scope)
{
	float totalMass;
	float3 linearVelocity;
	linearVelocity.x = 0.0;
	linearVelocity.y = 0.0;
	linearVelocity.z = 0.0;
	
	if(scope == 0) // entire system
	{
		for(int i = 0; i < N; i++)
		{
			if(i < NFe)
			{
		    		linearVelocity.x += Vel[i].x*MassFe;
				linearVelocity.y += Vel[i].y*MassFe;
				linearVelocity.z += Vel[i].z*MassFe;
			}
			else
			{
		    		linearVelocity.x += Vel[i].x*MassSi;
				linearVelocity.y += Vel[i].y*MassSi;
				linearVelocity.z += Vel[i].z*MassSi;
			}
		}
		totalMass = getMassCollision(0);
		linearVelocity.x /= totalMass;
		linearVelocity.y /= totalMass;
		linearVelocity.z /= totalMass;
	}	
	else if(scope == 1) // earth-moon system
	{
		for(int i = 0; i < NumberOfEarthElements; i++)
		{
			if(EarthIndex[i] < NFe)
			{
		    	linearVelocity.x += Vel[EarthIndex[i]].x*MassFe;
				linearVelocity.y += Vel[EarthIndex[i]].y*MassFe;
				linearVelocity.z += Vel[EarthIndex[i]].z*MassFe;
			}
			else
			{
		    	linearVelocity.x += Vel[EarthIndex[i]].x*MassSi;
				linearVelocity.y += Vel[EarthIndex[i]].y*MassSi;
				linearVelocity.z += Vel[EarthIndex[i]].z*MassSi;
			}
		}
		for(int i = 0; i < NumberOfMoonElements; i++)
		{
			if(MoonIndex[i] < NFe)
			{
		    	linearVelocity.x += Vel[MoonIndex[i]].x*MassFe;
				linearVelocity.y += Vel[MoonIndex[i]].y*MassFe;
				linearVelocity.z += Vel[MoonIndex[i]].z*MassFe;
			}
			else
			{
		    	linearVelocity.x += Vel[MoonIndex[i]].x*MassSi;
				linearVelocity.y += Vel[MoonIndex[i]].y*MassSi;
				linearVelocity.z += Vel[MoonIndex[i]].z*MassSi;
			}
		}
		totalMass = getMassCollision(1);
		linearVelocity.x /= totalMass;
		linearVelocity.y /= totalMass;
		linearVelocity.z /= totalMass;
	}
	else if(scope == 2) //earth
	{
		for(int i = 0; i < NumberOfEarthElements; i++)
		{
			if(EarthIndex[i] < NFe)
			{
		    	linearVelocity.x += Vel[EarthIndex[i]].x*MassFe;
				linearVelocity.y += Vel[EarthIndex[i]].y*MassFe;
				linearVelocity.z += Vel[EarthIndex[i]].z*MassFe;
			}
			else
			{
		    	linearVelocity.x += Vel[EarthIndex[i]].x*MassSi;
				linearVelocity.y += Vel[EarthIndex[i]].y*MassSi;
				linearVelocity.z += Vel[EarthIndex[i]].z*MassSi;
			}
		}
		totalMass = getMassCollision(2);
		linearVelocity.x /= totalMass;
		linearVelocity.y /= totalMass;
		linearVelocity.z /= totalMass;
	}
	else if(scope == 3) //moon
	{
		for(int i = 0; i < NumberOfMoonElements; i++)
		{
			if(MoonIndex[i] < NFe)
			{
		    	linearVelocity.x += Vel[MoonIndex[i]].x*MassFe;
				linearVelocity.y += Vel[MoonIndex[i]].y*MassFe;
				linearVelocity.z += Vel[MoonIndex[i]].z*MassFe;
			}
			else
			{
		    	linearVelocity.x += Vel[MoonIndex[i]].x*MassSi;
				linearVelocity.y += Vel[MoonIndex[i]].y*MassSi;
				linearVelocity.z += Vel[MoonIndex[i]].z*MassSi;
			}
		}
		totalMass = getMassCollision(3);
		linearVelocity.x /= totalMass;
		linearVelocity.y /= totalMass;
		linearVelocity.z /= totalMass;
	}
	else
	{
		printf("\nTSU Error: in getlinearVelocityEarthMoonSystem function scope invalid\n");
		exit(0);
	}
	return(linearVelocity);
}

float3 getAngularMomentumCollision(int scope)
{
	float3 centerOfMass, linearVelocity, angularMomentum;
	float3 r;
	float3 v;
	angularMomentum.x = 0.0;
	angularMomentum.y = 0.0;
	angularMomentum.z = 0.0;
	
	if(scope == 0) //Entire system
	{
		centerOfMass = getCenterOfMassCollision(0);
		linearVelocity = getLinearVelocityCollision(0);
		for(int i = 0; i < N; i++)
		{
			r.x = Pos[i].x - centerOfMass.x;
			r.y = Pos[i].y - centerOfMass.y;
			r.z = Pos[i].z - centerOfMass.z;
		
			v.x = Vel[i].x - linearVelocity.x;
			v.y = Vel[i].y - linearVelocity.y;
			v.z = Vel[i].z - linearVelocity.z;
			if(i < NFe)
			{
		    		angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassFe;
				angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassFe;
				angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassFe;
			}
			else
			{
				angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassSi;
				angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassSi;
				angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassSi;
			}
		}
	}
	else if(scope == 1) //Earth-Moon system
	{
		centerOfMass = getCenterOfMassCollision(1);
		linearVelocity = getLinearVelocityCollision(1);
		for(int i = 0; i < NumberOfEarthElements; i++)
		{
			r.x = Pos[EarthIndex[i]].x - centerOfMass.x;
			r.y = Pos[EarthIndex[i]].y - centerOfMass.y;
			r.z = Pos[EarthIndex[i]].z - centerOfMass.z;
		
			v.x = Vel[EarthIndex[i]].x - linearVelocity.x;
			v.y = Vel[EarthIndex[i]].y - linearVelocity.y;
			v.z = Vel[EarthIndex[i]].z - linearVelocity.z;
			if(EarthIndex[i] < NFe)
			{
		    		angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassFe;
				angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassFe;
				angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassFe;
			}
			else
			{
				angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassSi;
				angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassSi;
				angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassSi;
			}
		}
		for(int i = 0; i < NumberOfMoonElements; i++)
		{
			r.x = Pos[MoonIndex[i]].x - centerOfMass.x;
			r.y = Pos[MoonIndex[i]].y - centerOfMass.y;
			r.z = Pos[MoonIndex[i]].z - centerOfMass.z;
		
			v.x = Vel[MoonIndex[i]].x - linearVelocity.x;
			v.y = Vel[MoonIndex[i]].y - linearVelocity.y;
			v.z = Vel[MoonIndex[i]].z - linearVelocity.z;
			if(MoonIndex[i] < NFe)
			{
		    		angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassFe;
				angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassFe;
				angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassFe;
			}
			else
			{
				angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassSi;
				angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassSi;
				angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassSi;
			}
		}
	}
	else if(scope == 2) //Earth
	{
		centerOfMass = getCenterOfMassCollision(2);
		linearVelocity = getLinearVelocityCollision(2);
		for(int i = 0; i < NumberOfEarthElements; i++)
		{
			r.x = Pos[EarthIndex[i]].x - centerOfMass.x;
			r.y = Pos[EarthIndex[i]].y - centerOfMass.y;
			r.z = Pos[EarthIndex[i]].z - centerOfMass.z;
		
			v.x = Vel[EarthIndex[i]].x - linearVelocity.x;
			v.y = Vel[EarthIndex[i]].y - linearVelocity.y;
			v.z = Vel[EarthIndex[i]].z - linearVelocity.z;
			if(EarthIndex[i] < NFe)
			{
		    		angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassFe;
				angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassFe;
				angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassFe;
			}
			else
			{
				angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassSi;
				angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassSi;
				angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassSi;
			}
		}
	}
	else if(scope == 3) //Moon
	{
		centerOfMass = getCenterOfMassCollision(3);
		linearVelocity = getLinearVelocityCollision(3);
		for(int i = 0; i < NumberOfMoonElements; i++)
		{
			r.x = Pos[MoonIndex[i]].x - centerOfMass.x;
			r.y = Pos[MoonIndex[i]].y - centerOfMass.y;
			r.z = Pos[MoonIndex[i]].z - centerOfMass.z;
		
			v.x = Vel[MoonIndex[i]].x - linearVelocity.x;
			v.y = Vel[MoonIndex[i]].y - linearVelocity.y;
			v.z = Vel[MoonIndex[i]].z - linearVelocity.z;
			if(MoonIndex[i] < NFe)
			{
		    		angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassFe;
				angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassFe;
				angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassFe;
			}
			else
			{
				angularMomentum.x +=  (r.y*v.z - r.z*v.y)*MassSi;
				angularMomentum.y += -(r.x*v.z - r.z*v.x)*MassSi;
				angularMomentum.z +=  (r.x*v.y - r.y*v.x)*MassSi;
			}
		}
	}
	else
	{
		printf("\nTSU Error: in getAngularMomentumCollision function scope invalid\n");
		exit(0);
	}
	return(angularMomentum);
}

void printContinueStatsToScreen(double time)
{	
	double timeConverter = UnitTime;
	double lengthConverter = UnitLength;
	double massConverter = UnitMass; 
	//double velocityConverter = UnitLength/UnitTime; 
	double momentumConverter = UnitMass*UnitLength*UnitLength/UnitTime;
	
	float3 r, v;
	double d, mass, mag, size, angle, x, y, z;
	
	float massEarth;
	float3 centerOfMassEarth;
	float3 linearVelocityEarth;
	
	float3 centerOfMassEarthMoonMaterial;
	float3 averageVelocityEarthMoonMaterial;
	
	int earthMaterialFeCountBody1 = 0;
	int earthMaterialFeCountBody2 = 0;
	int earthMaterialSiCountBody1 = 0;
	int earthMaterialSiCountBody2 = 0;
	float earthMaterialMass = 0.0;
	
	int moonMaterialFeCountBody1 = 0;
	int moonMaterialFeCountBody2 = 0;
	int moonMaterialSiCountBody1 = 0;
	int moonMaterialSiCountBody2 = 0;
	float moonMaterialMass = 0.0;
	
	int escapeMaterialFeCountBody1 = 0;
	int escapeMaterialFeCountBody2 = 0;
	int escapeMaterialSiCountBody1 = 0;
	int escapeMaterialSiCountBody2 = 0;
	float escapeMaterialMass = 0.0;
	
	int unusedMaterialFeCountBody1 = 0;
	int unusedMaterialFeCountBody2 = 0;
	int unusedMaterialSiCountBody1 = 0;
	int unusedMaterialSiCountBody2 = 0;
	float unusedMaterialMass = 0.0;
	
	float3 angularMomentumHolder;
	float3 angularMomentumEarthMoonMaterial;
	float3 angularMomentumEarthMaterial;
	float3 angularMomentumMoonMaterial;
	
	//Finding radius of what the current Earth is
	findEarthAndMoon();
	centerOfMassEarth = getCenterOfMassCollision(2);
	massEarth = getMassCollision(2);
	float radiusOfEarth = 0.0;
	for(int i = 0; i < NumberOfEarthElements; i++)
	{
		r.x = Pos[EarthIndex[i]].x - centerOfMassEarth.x;
		r.y = Pos[EarthIndex[i]].y - centerOfMassEarth.y;
		r.z = Pos[EarthIndex[i]].z - centerOfMassEarth.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(d > radiusOfEarth) radiusOfEarth = d;
	}
	
	// Finding Roche limit and setting sphere to create Earth and sphere to create the Moon 
	float densityEarth = massEarth/((Pi*4.0/3.0)*radiusOfEarth*radiusOfEarth*radiusOfEarth);
	float densitySi = MassSi/((Pi*4.0/3.0)*(Diameter/2.0)*(Diameter/2.0)*(Diameter/2.0));
	float rocheLimit = 2.44*radiusOfEarth*pow((densityEarth/densitySi),1.0/3.0);
	float radiusEarthMaterial = rocheLimit;
	float radiusMoonMaterial  = NUMBEROFEARTHRADIFORMOONMATERIAL*radiusOfEarth;
	
	// Finding mass of Earth material, Moon Material
	// Finding the center of mass and average velocity of the material we estimating will make the Earth-Moon system 
	// Finding Moon mix and Earth mix
	earthMaterialMass = 0.0;
	moonMaterialMass = 0.0;
	
	centerOfMassEarthMoonMaterial.x = 0.0;
	centerOfMassEarthMoonMaterial.y = 0.0;
	centerOfMassEarthMoonMaterial.z = 0.0;
	
	averageVelocityEarthMoonMaterial.x = 0.0;
	averageVelocityEarthMoonMaterial.y = 0.0;
	averageVelocityEarthMoonMaterial.z = 0.0;
	
	for(int i = 0; i < N; i++)
	{
		
		r.x = Pos[i].x - centerOfMassEarth.x;
		r.y = Pos[i].y - centerOfMassEarth.y;
		r.z = Pos[i].z - centerOfMassEarth.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(d < radiusEarthMaterial)
		{
			if(i < NFe) 	mass = MassFe;	
			else 		mass = MassSi;
			
			earthMaterialMass += mass;
			
			centerOfMassEarthMoonMaterial.x += mass*Pos->x;
			centerOfMassEarthMoonMaterial.y += mass*Pos->y;
			centerOfMassEarthMoonMaterial.z += mass*Pos->z;
			
			averageVelocityEarthMoonMaterial.x += mass*Vel->x;
			averageVelocityEarthMoonMaterial.y += mass*Vel->y;
			averageVelocityEarthMoonMaterial.z += mass*Vel->z;
			
			if(i < NFe1) 				earthMaterialFeCountBody1++;
			else if(i < NFe1 + NFe2) 		earthMaterialFeCountBody2++;
			else if(i < NFe1 + NFe2 + NSi1) 	earthMaterialSiCountBody1++;
			else					earthMaterialSiCountBody2++;
		}
		else if(d < radiusMoonMaterial)
		{
			if(i < NFe) 	mass = MassFe;
			else 		mass = MassSi;
			
			moonMaterialMass += mass;
			
			centerOfMassEarthMoonMaterial.x += mass*Pos->x;
			centerOfMassEarthMoonMaterial.y += mass*Pos->y;
			centerOfMassEarthMoonMaterial.z += mass*Pos->z;
			
			averageVelocityEarthMoonMaterial.x += mass*Vel->x;
			averageVelocityEarthMoonMaterial.y += mass*Vel->y;
			averageVelocityEarthMoonMaterial.z += mass*Vel->z;
			
			if(i < NFe1) 				moonMaterialFeCountBody1++;
			else if(i < NFe1 + NFe2) 		moonMaterialFeCountBody2++;
			else if(i < NFe1 + NFe2 + NSi1) 	moonMaterialSiCountBody1++;
			else					moonMaterialSiCountBody2++;
			
		}
	}
	centerOfMassEarthMoonMaterial.x /= (earthMaterialMass + moonMaterialMass);
	centerOfMassEarthMoonMaterial.y /= (earthMaterialMass + moonMaterialMass);
	centerOfMassEarthMoonMaterial.z /= (earthMaterialMass + moonMaterialMass);
	
	averageVelocityEarthMoonMaterial.x /= (earthMaterialMass + moonMaterialMass);
	averageVelocityEarthMoonMaterial.y /= (earthMaterialMass + moonMaterialMass);
	averageVelocityEarthMoonMaterial.z /= (earthMaterialMass + moonMaterialMass);
	
	// Getting a rough estimate of how much of the extra material has escape velocity from what we 
	// considering will make the Earth-Moon system
	float velocity;
	float escapeVelocity;
	escapeMaterialMass = 0.0;
	unusedMaterialMass = 0.0;
	for(int i = 0; i < N; i++)
	{
		r.x = Pos[i].x - centerOfMassEarth.x;
		r.y = Pos[i].y - centerOfMassEarth.y;
		r.z = Pos[i].z - centerOfMassEarth.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(radiusMoonMaterial <= d)
		{
			r.x = Pos[i].x - centerOfMassEarthMoonMaterial.x;
			r.y = Pos[i].y - centerOfMassEarthMoonMaterial.y;
			r.z = Pos[i].z - centerOfMassEarthMoonMaterial.z;
			d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
			
			v.x = Vel[i].x - averageVelocityEarthMoonMaterial.x;
			v.y = Vel[i].y - averageVelocityEarthMoonMaterial.y;
			v.z = Vel[i].z - averageVelocityEarthMoonMaterial.z;
			velocity = sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
		
			escapeVelocity = sqrt(2.0*Gravity*(earthMaterialMass + moonMaterialMass)/d);
			
			if(velocity >= escapeVelocity)
			{
				if(i < NFe) 	mass = MassFe;
				else 		mass = MassSi;
			
				escapeMaterialMass += mass;
			
			 	if(i < NFe1) 				escapeMaterialFeCountBody1++;
				else if(i < NFe1 + NFe2) 		escapeMaterialFeCountBody2++;
				else if(i < NFe1 + NFe2 + NSi1) 	escapeMaterialSiCountBody1++;
				else					escapeMaterialSiCountBody2++;
			}
			else
			{
				if(i < NFe) 	mass = MassFe;
				else 		mass = MassSi;
			
				unusedMaterialMass += mass;
				if(i < NFe1) 				unusedMaterialFeCountBody1++;
				else if(i < NFe1 + NFe2) 		unusedMaterialFeCountBody2++;
				else if(i < NFe1 + NFe2 + NSi1) 	unusedMaterialSiCountBody1++;
				else					unusedMaterialSiCountBody2++;
			}
		}
	}
	
	// Finding the angular momentum of the Earth-Moon material
	// Finding the angular momentum of the Earth material
	// Finding the angular momentum of the Moon material
	linearVelocityEarth = getLinearVelocityCollision(2);
	
	angularMomentumEarthMoonMaterial.x = 0.0;
	angularMomentumEarthMoonMaterial.y = 0.0;
	angularMomentumEarthMoonMaterial.z = 0.0;
	
	angularMomentumEarthMaterial.x = 0.0;
	angularMomentumEarthMaterial.y = 0.0;
	angularMomentumEarthMaterial.z = 0.0;
	
	angularMomentumMoonMaterial.x = 0.0;
	angularMomentumMoonMaterial.y = 0.0;
	angularMomentumMoonMaterial.z = 0.0;
	
	for(int i = 0; i < N; i++)
	{
		r.x = Pos[i].x - centerOfMassEarth.x;
		r.y = Pos[i].y - centerOfMassEarth.y;
		r.z = Pos[i].z - centerOfMassEarth.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(d < radiusMoonMaterial)
		{
			v.x = Vel[i].x - linearVelocityEarth.x;
			v.y = Vel[i].y - linearVelocityEarth.y;
			v.z = Vel[i].z - linearVelocityEarth.z;
			if(i < NFe)
			{
		    		angularMomentumHolder.x =  (r.y*v.z - r.z*v.y)*MassFe;
				angularMomentumHolder.y = -(r.x*v.z - r.z*v.x)*MassFe;
				angularMomentumHolder.z =  (r.x*v.y - r.y*v.x)*MassFe;
			}
			else
			{
				angularMomentumHolder.x =  (r.y*v.z - r.z*v.y)*MassSi;
				angularMomentumHolder.y = -(r.x*v.z - r.z*v.x)*MassSi;
				angularMomentumHolder.z =  (r.x*v.y - r.y*v.x)*MassSi;
			}
			
			angularMomentumEarthMoonMaterial.x += angularMomentumHolder.x;
			angularMomentumEarthMoonMaterial.y += angularMomentumHolder.y;
			angularMomentumEarthMoonMaterial.z += angularMomentumHolder.z;
			
			if(d < radiusEarthMaterial)
			{
				angularMomentumEarthMaterial.x +=  angularMomentumHolder.x;
				angularMomentumEarthMaterial.y +=  angularMomentumHolder.y;
				angularMomentumEarthMaterial.z +=  angularMomentumHolder.z;
			}
			else
			{
				angularMomentumMoonMaterial.x +=  angularMomentumHolder.x;
				angularMomentumMoonMaterial.y +=  angularMomentumHolder.y;
				angularMomentumMoonMaterial.z +=  angularMomentumHolder.z;
			}
		}
	}
	
	printf("\n\n\n*************************************************************************\n");
	printf("\nThe following are the three stats to feed to the search program\n");
	
	x = angularMomentumEarthMoonMaterial.x*momentumConverter;
	y = angularMomentumEarthMoonMaterial.y*momentumConverter;
	z = angularMomentumEarthMoonMaterial.z*momentumConverter;
	mag = sqrt(x*x + y*y + z*z);
	printf("\nAngular momentum of the Earth-Moon system = %e", mag);
	printf("\nRatio Earth mass to Moon mass = %f", earthMaterialMass/moonMaterialMass);
	printf("\nMoon compotition ratio  = %f", (float)(moonMaterialFeCountBody1 + moonMaterialSiCountBody1)/(float)(moonMaterialFeCountBody2 + moonMaterialSiCountBody2));
	
	printf("\n\n\n*************************************************************************\n");
	printf("\nThe following are all the continuation stats of the run when time = %f hours\n", time*timeConverter/3600.0);
	printf("\nDistance is measured in Kilometers");
	printf("\nMass is measured in Kilograms");
	printf("\nTime is measured in seconds");
	printf("\nVelocity is measured in Kilometers/second");
	printf("\nAngular momentun is measured in Kilograms*Kilometers*Kilometers/seconds\n");
	
	printf("\nThe radius of Earth 		= %f", radiusOfEarth*lengthConverter);
	printf("\nRoche limit 			= %f", rocheLimit*lengthConverter);
	printf("\nRoche limit/radius of Earth 	= %f \n", rocheLimit/radiusOfEarth);
	
	x = angularMomentumEarthMoonMaterial.x*momentumConverter;
	y = angularMomentumEarthMoonMaterial.y*momentumConverter;
	z = angularMomentumEarthMoonMaterial.z*momentumConverter;
	printf("\nAngular momentum of the Earth-Moon material			 = (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	printf("\nMagnitude of the angular momentum of the Earth-Moon material 	 = %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	printf("\nAngle off ecliptic plane of the Earth-Moon's material rotation	 = %f\n", 90.0 - angle*180.0/Pi);
	
	x = angularMomentumEarthMaterial.x*momentumConverter;
	y = angularMomentumEarthMaterial.y*momentumConverter;
	z = angularMomentumEarthMaterial.z*momentumConverter;
	printf("\nAngular momentum of the Earth material				= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	printf("\nMagnitude of the angular momentum of the Earth material 	= %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	printf("\nAngle off ecliptic plane of the Earth's material rotation 	= %f\n", 90.0 - angle*180.0/Pi);
	
	x = angularMomentumMoonMaterial.x*momentumConverter;
	y = angularMomentumMoonMaterial.y*momentumConverter;
	z = angularMomentumMoonMaterial.z*momentumConverter;
	printf("\nAngular momentum of the Moon material			   	= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	printf("\nMagnitude of the angular momentum of the Moon material   	= %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	printf("\nAngle off ecliptic plane of the Moon's material rotation 	= %f\n", 90.0 - angle*180.0/Pi);
	
	printf("\nThe mass of Earth material		= %e", earthMaterialMass*massConverter);
	printf("\nThe Earth material count Fe body 1	= %d", earthMaterialFeCountBody1);
	printf("\nThe Earth material count Fe body 2	= %d", earthMaterialFeCountBody2);
	printf("\nThe Earth material count Si body 1	= %d", earthMaterialSiCountBody1);
	printf("\nThe Earth material count Si body 2	= %d", earthMaterialSiCountBody2);
	printf("\nThe Earth material Body1/Body2 ratio	= %f\n", (float)(earthMaterialFeCountBody1 + earthMaterialSiCountBody1)/(float)(earthMaterialFeCountBody2 + earthMaterialSiCountBody2));
	
	printf("\nThe mass of Moon material		= %e", moonMaterialMass*massConverter);
	printf("\nThe Moon material count Fe body 1	= %d", moonMaterialFeCountBody1);
	printf("\nThe Moon material count Fe body 2	= %d", moonMaterialFeCountBody2);
	printf("\nThe Moon material count Si body 1	= %d", moonMaterialSiCountBody1);
	printf("\nThe Moon material count Si body 2	= %d", moonMaterialSiCountBody2);
	printf("\nThe Moon material Body1/Body2 ratio	= %f\n", (float)(moonMaterialFeCountBody1 + moonMaterialSiCountBody1)/(float)(moonMaterialFeCountBody2 + moonMaterialSiCountBody2));
	
	printf("\nThe mass of escape material		= %e", escapeMaterialMass*massConverter);
	printf("\nThe escape material count Fe body 1	= %d", escapeMaterialFeCountBody1);
	printf("\nThe escape material count Fe body 2	= %d", escapeMaterialFeCountBody2);
	printf("\nThe escape material count Si body 1	= %d", escapeMaterialSiCountBody1);
	printf("\nThe escape material count Si body 2	= %d", escapeMaterialSiCountBody2);
	printf("\nThe escape material Body1/Body2 ratio	= %f\n", (float)(escapeMaterialFeCountBody1 + escapeMaterialSiCountBody1)/(float)(escapeMaterialFeCountBody2 + escapeMaterialSiCountBody2));
	
	printf("\nThe mass of unused material		= %e", unusedMaterialMass*massConverter);
	printf("\nThe unused material count Fe body 1	= %d", unusedMaterialFeCountBody1);
	printf("\nThe unused material count Fe body 2	= %d", unusedMaterialFeCountBody2);
	printf("\nThe unused material count Si body 1	= %d", unusedMaterialSiCountBody1);
	printf("\nThe unused material count Si body 2	= %d", unusedMaterialSiCountBody2);
	printf("\nThe unused material Body1/Body2 ratio	= %f\n", (float)(unusedMaterialFeCountBody1 + unusedMaterialSiCountBody1)/(float)(unusedMaterialFeCountBody2 + unusedMaterialSiCountBody2));
	
	printf("\n*************************************************************************\n\n\n");
}

void printContinueStatsToFile(double time)
{	
	double timeConverter = UnitTime;
	double lengthConverter = UnitLength;
	double massConverter = UnitMass; 
	//double velocityConverter = UnitLength/UnitTime; 
	double momentumConverter = UnitMass*UnitLength*UnitLength/UnitTime;
	
	float3 r, v;
	double d, mass, mag, size, angle, x, y, z;
	
	float massEarth;
	float3 centerOfMassEarth;
	float3 linearVelocityEarth;
	
	float3 centerOfMassEarthMoonMaterial;
	float3 averageVelocityEarthMoonMaterial;
	
	int earthMaterialFeCountBody1 = 0;
	int earthMaterialFeCountBody2 = 0;
	int earthMaterialSiCountBody1 = 0;
	int earthMaterialSiCountBody2 = 0;
	float earthMaterialMass = 0.0;
	
	int moonMaterialFeCountBody1 = 0;
	int moonMaterialFeCountBody2 = 0;
	int moonMaterialSiCountBody1 = 0;
	int moonMaterialSiCountBody2 = 0;
	float moonMaterialMass = 0.0;
	
	int escapeMaterialFeCountBody1 = 0;
	int escapeMaterialFeCountBody2 = 0;
	int escapeMaterialSiCountBody1 = 0;
	int escapeMaterialSiCountBody2 = 0;
	float escapeMaterialMass = 0.0;
	
	int unusedMaterialFeCountBody1 = 0;
	int unusedMaterialFeCountBody2 = 0;
	int unusedMaterialSiCountBody1 = 0;
	int unusedMaterialSiCountBody2 = 0;
	float unusedMaterialMass = 0.0;
	
	float3 angularMomentumEarthMoonMaterial;
	float3 angularMomentumEarthMaterial;
	float3 angularMomentumMoonMaterial;
	
	//Finding radius of what the current Earth is
	findEarthAndMoon();
	centerOfMassEarth = getCenterOfMassCollision(2);
	massEarth = getMassCollision(2);
	float radiusOfEarth = 0.0;
	for(int i = 0; i < NumberOfEarthElements; i++)
	{
		r.x = Pos[EarthIndex[i]].x - centerOfMassEarth.x;
		r.y = Pos[EarthIndex[i]].y - centerOfMassEarth.y;
		r.z = Pos[EarthIndex[i]].z - centerOfMassEarth.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(d > radiusOfEarth) radiusOfEarth = d;
	}
	
	// Finding Roche limit and setting sphere to create Earth and sphere to create the Moon 
	float densityEarth = massEarth/((Pi*4.0/3.0)*radiusOfEarth*radiusOfEarth*radiusOfEarth);
	float densitySi = MassSi/((Pi*4.0/3.0)*(Diameter/2.0)*(Diameter/2.0)*(Diameter/2.0));
	float rocheLimit = 2.44*radiusOfEarth*pow((densityEarth/densitySi),1.0/3.0);
	float radiusEarthMaterial = rocheLimit;
	float radiusMoonMaterial  = NUMBEROFEARTHRADIFORMOONMATERIAL*radiusOfEarth;
	
	// Finding mass of Earth material, Moon Material
	// Finding the center of mass and average velocity of the material we estimating will make the Earth-Moon system 
	// Finding Moon mix and Earth mix
	earthMaterialMass = 0.0;
	moonMaterialMass = 0.0;
	
	centerOfMassEarthMoonMaterial.x = 0.0;
	centerOfMassEarthMoonMaterial.y = 0.0;
	centerOfMassEarthMoonMaterial.z = 0.0;
	
	averageVelocityEarthMoonMaterial.x = 0.0;
	averageVelocityEarthMoonMaterial.y = 0.0;
	averageVelocityEarthMoonMaterial.z = 0.0;
	
	for(int i = 0; i < N; i++)
	{
		
		r.x = Pos[i].x - centerOfMassEarth.x;
		r.y = Pos[i].y - centerOfMassEarth.y;
		r.z = Pos[i].z - centerOfMassEarth.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(d < radiusEarthMaterial)
		{
			if(i < NFe) 	mass = MassFe;	
			else 		mass = MassSi;
			
			earthMaterialMass += mass;
			
			centerOfMassEarthMoonMaterial.x += mass*Pos->x;
			centerOfMassEarthMoonMaterial.y += mass*Pos->y;
			centerOfMassEarthMoonMaterial.z += mass*Pos->z;
			
			averageVelocityEarthMoonMaterial.x += mass*Vel->x;
			averageVelocityEarthMoonMaterial.y += mass*Vel->y;
			averageVelocityEarthMoonMaterial.z += mass*Vel->z;
			
			if(i < NFe1) 				earthMaterialFeCountBody1++;
			else if(i < NFe1 + NFe2) 		earthMaterialFeCountBody2++;
			else if(i < NFe1 + NFe2 + NSi1) 	earthMaterialSiCountBody1++;
			else					earthMaterialSiCountBody2++;
		}
		else if(d < radiusMoonMaterial)
		{
			if(i < NFe) 	mass = MassFe;
			else 		mass = MassSi;
			
			moonMaterialMass += mass;
			
			centerOfMassEarthMoonMaterial.x += mass*Pos->x;
			centerOfMassEarthMoonMaterial.y += mass*Pos->y;
			centerOfMassEarthMoonMaterial.z += mass*Pos->z;
			
			averageVelocityEarthMoonMaterial.x += mass*Vel->x;
			averageVelocityEarthMoonMaterial.y += mass*Vel->y;
			averageVelocityEarthMoonMaterial.z += mass*Vel->z;
			
			if(i < NFe1) 				moonMaterialFeCountBody1++;
			else if(i < NFe1 + NFe2) 		moonMaterialFeCountBody2++;
			else if(i < NFe1 + NFe2 + NSi1) 	moonMaterialSiCountBody1++;
			else					moonMaterialSiCountBody2++;
			
		}
	}
	centerOfMassEarthMoonMaterial.x /= (earthMaterialMass + moonMaterialMass);
	centerOfMassEarthMoonMaterial.y /= (earthMaterialMass + moonMaterialMass);
	centerOfMassEarthMoonMaterial.z /= (earthMaterialMass + moonMaterialMass);
	
	averageVelocityEarthMoonMaterial.x /= (earthMaterialMass + moonMaterialMass);
	averageVelocityEarthMoonMaterial.y /= (earthMaterialMass + moonMaterialMass);
	averageVelocityEarthMoonMaterial.z /= (earthMaterialMass + moonMaterialMass);
	
	// Getting a rough estimate of how much of the extra material has escape velocity from what we 
	// considering will make the Earth-Moon system
	float velocity;
	float escapeVelocity;
	escapeMaterialMass = 0.0;
	unusedMaterialMass = 0.0;
	for(int i = 0; i < N; i++)
	{
		r.x = Pos[i].x - centerOfMassEarth.x;
		r.y = Pos[i].y - centerOfMassEarth.y;
		r.z = Pos[i].z - centerOfMassEarth.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(radiusMoonMaterial <= d)
		{
			r.x = Pos[i].x - centerOfMassEarthMoonMaterial.x;
			r.y = Pos[i].y - centerOfMassEarthMoonMaterial.y;
			r.z = Pos[i].z - centerOfMassEarthMoonMaterial.z;
			d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
			
			v.x = Vel[i].x - averageVelocityEarthMoonMaterial.x;
			v.y = Vel[i].y - averageVelocityEarthMoonMaterial.y;
			v.z = Vel[i].z - averageVelocityEarthMoonMaterial.z;
			velocity = sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
		
			escapeVelocity = sqrt(2.0*Gravity*(earthMaterialMass + moonMaterialMass)/d);
			
			if(velocity >= escapeVelocity)
			{
				if(i < NFe) 	mass = MassFe;
				else 		mass = MassSi;
			
				escapeMaterialMass += mass;
			
			 	if(i < NFe1) 				escapeMaterialFeCountBody1++;
				else if(i < NFe1 + NFe2) 		escapeMaterialFeCountBody2++;
				else if(i < NFe1 + NFe2 + NSi1) 	escapeMaterialSiCountBody1++;
				else					escapeMaterialSiCountBody2++;
			}
			else
			{
				if(i < NFe) 	mass = MassFe;
				else 		mass = MassSi;
			
				unusedMaterialMass += mass;
				if(i < NFe1) 				unusedMaterialFeCountBody1++;
				else if(i < NFe1 + NFe2) 		unusedMaterialFeCountBody2++;
				else if(i < NFe1 + NFe2 + NSi1) 	unusedMaterialSiCountBody1++;
				else					unusedMaterialSiCountBody2++;
			}
		}
	}
	
	// Finding the angular momentum of the Earth-Moon material
	// Finding the angular momentum of the Earth material
	// Finding the angular momentum of the Moon material
	linearVelocityEarth = getLinearVelocityCollision(2);
	
	angularMomentumEarthMoonMaterial.x = 0.0;
	angularMomentumEarthMoonMaterial.y = 0.0;
	angularMomentumEarthMoonMaterial.z = 0.0;
	
	angularMomentumEarthMaterial.x = 0.0;
	angularMomentumEarthMaterial.y = 0.0;
	angularMomentumEarthMaterial.z = 0.0;
	
	angularMomentumMoonMaterial.x = 0.0;
	angularMomentumMoonMaterial.y = 0.0;
	angularMomentumMoonMaterial.z = 0.0;
	
	for(int i = 0; i < N; i++)
	{
		r.x = Pos[i].x - centerOfMassEarth.x;
		r.y = Pos[i].y - centerOfMassEarth.y;
		r.z = Pos[i].z - centerOfMassEarth.z;
		
		d = sqrt(r.x*r.x + r.y*r.y + r.z*r.z);
		
		if(d < radiusMoonMaterial)
		{
			v.x = Vel[i].x - linearVelocityEarth.x;
			v.y = Vel[i].y - linearVelocityEarth.y;
			v.z = Vel[i].z - linearVelocityEarth.z;
			if(i < NFe)
			{
		    		angularMomentumEarthMoonMaterial.x +=  (r.y*v.z - r.z*v.y)*MassFe;
				angularMomentumEarthMoonMaterial.y += -(r.x*v.z - r.z*v.x)*MassFe;
				angularMomentumEarthMoonMaterial.z +=  (r.x*v.y - r.y*v.x)*MassFe;
			}
			else
			{
				angularMomentumEarthMoonMaterial.x +=  (r.y*v.z - r.z*v.y)*MassSi;
				angularMomentumEarthMoonMaterial.y += -(r.x*v.z - r.z*v.x)*MassSi;
				angularMomentumEarthMoonMaterial.z +=  (r.x*v.y - r.y*v.x)*MassSi;
			}
			
			if(radiusEarthMaterial < d)
			{
				angularMomentumEarthMaterial.x +=  angularMomentumEarthMoonMaterial.x;
				angularMomentumEarthMaterial.y +=  angularMomentumEarthMoonMaterial.y;
				angularMomentumEarthMaterial.z +=  angularMomentumEarthMoonMaterial.z;
			}
			else
			{
				angularMomentumMoonMaterial.x +=  angularMomentumEarthMoonMaterial.x;
				angularMomentumMoonMaterial.y +=  angularMomentumEarthMoonMaterial.y;
				angularMomentumMoonMaterial.z +=  angularMomentumEarthMoonMaterial.z;
			}
		}
	}
	
	fprintf(ContinueRunStatsFile, "\n\n\n*************************************************************************\n");
	fprintf(ContinueRunStatsFile, "\nThe following are the three stats to feed to the search program\n");
	
	x = angularMomentumEarthMoonMaterial.x*momentumConverter;
	y = angularMomentumEarthMoonMaterial.y*momentumConverter;
	z = angularMomentumEarthMoonMaterial.z*momentumConverter;
	mag = sqrt(x*x + y*y + z*z);
	fprintf(ContinueRunStatsFile, "\nAngular momentum of the Earth-Moon system = %e", mag);
	fprintf(ContinueRunStatsFile, "\nRatio Earth mass to Moon mass = %f", earthMaterialMass/moonMaterialMass);
	fprintf(ContinueRunStatsFile, "\nMoon compotition ratio  = %f", (float)(moonMaterialFeCountBody1 + moonMaterialSiCountBody1)/(float)(moonMaterialFeCountBody2 + moonMaterialSiCountBody2));
	
	fprintf(ContinueRunStatsFile, "\n\n\n*************************************************************************\n");
	fprintf(ContinueRunStatsFile, "\nThe following are all the continuation stats of the run when time = %f hours\n", time*timeConverter/3600.0);
	fprintf(ContinueRunStatsFile, "\nDistance is measured in Kilometers");
	fprintf(ContinueRunStatsFile, "\nMass is measured in Kilograms");
	fprintf(ContinueRunStatsFile, "\nTime is measured in seconds");
	fprintf(ContinueRunStatsFile, "\nVelocity is measured in Kilometers/second");
	fprintf(ContinueRunStatsFile, "\nAngular momentun is measured in Kilograms*Kilometers*Kilometers/seconds\n");
	
	fprintf(ContinueRunStatsFile, "\nThe radius of Earth 		= %f", radiusOfEarth*lengthConverter);
	fprintf(ContinueRunStatsFile, "\nRoche limit 			= %f", rocheLimit*lengthConverter);
	fprintf(ContinueRunStatsFile, "\nRoche limit/radius of Earth 	= %f \n", rocheLimit/radiusOfEarth);
	
	x = angularMomentumEarthMoonMaterial.x*momentumConverter;
	y = angularMomentumEarthMoonMaterial.y*momentumConverter;
	z = angularMomentumEarthMoonMaterial.z*momentumConverter;
	fprintf(ContinueRunStatsFile, "\nAngular momentum of the Earth-Moon material			 = (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	fprintf(ContinueRunStatsFile, "\nMagnitude of the angular momentum of the Earth-Moon material 	 = %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	fprintf(ContinueRunStatsFile, "\nAngle off ecliptic plane of the Earth-Moon's material rotation	 = %f\n", 90.0 - angle*180.0/Pi);
	
	x = angularMomentumEarthMaterial.x*momentumConverter;
	y = angularMomentumEarthMaterial.y*momentumConverter;
	z = angularMomentumEarthMaterial.z*momentumConverter;
	fprintf(ContinueRunStatsFile, "\nAngular momentum of the Earth material				= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	fprintf(ContinueRunStatsFile, "\nMagnitude of the angular momentum of the Earth material 	= %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	fprintf(ContinueRunStatsFile, "\nAngle off ecliptic plane of the Earth's material rotation 	= %f\n", 90.0 - angle*180.0/Pi);
	
	x = angularMomentumMoonMaterial.x*momentumConverter;
	y = angularMomentumMoonMaterial.y*momentumConverter;
	z = angularMomentumMoonMaterial.z*momentumConverter;
	fprintf(ContinueRunStatsFile, "\nAngular momentum of the Moon material			   	= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	fprintf(ContinueRunStatsFile, "\nMagnitude of the angular momentum of the Moon material   	= %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	fprintf(ContinueRunStatsFile, "\nAngle off ecliptic plane of the Moon's material rotation 	= %f\n", 90.0 - angle*180.0/Pi);
	
	fprintf(ContinueRunStatsFile, "\nThe mass of Earth material		= %e", earthMaterialMass*massConverter);
	fprintf(ContinueRunStatsFile, "\nThe Earth material count Fe body 1	= %d", earthMaterialFeCountBody1);
	fprintf(ContinueRunStatsFile, "\nThe Earth material count Fe body 2	= %d", earthMaterialFeCountBody2);
	fprintf(ContinueRunStatsFile, "\nThe Earth material count Si body 1	= %d", earthMaterialSiCountBody1);
	fprintf(ContinueRunStatsFile, "\nThe Earth material count Si body 2	= %d", earthMaterialSiCountBody2);
	fprintf(ContinueRunStatsFile, "\nThe Earth material Body1/Body2 ratio	= %f\n", (float)(earthMaterialFeCountBody1 + earthMaterialSiCountBody1)/(float)(earthMaterialFeCountBody2 + earthMaterialSiCountBody2));
	
	fprintf(ContinueRunStatsFile, "\nThe mass of Moon material		= %e", moonMaterialMass*massConverter);
	fprintf(ContinueRunStatsFile, "\nThe Moon material count Fe body 1	= %d", moonMaterialFeCountBody1);
	fprintf(ContinueRunStatsFile, "\nThe Moon material count Fe body 2	= %d", moonMaterialFeCountBody2);
	fprintf(ContinueRunStatsFile, "\nThe Moon material count Si body 1	= %d", moonMaterialSiCountBody1);
	fprintf(ContinueRunStatsFile, "\nThe Moon material count Si body 2	= %d", moonMaterialSiCountBody2);
	fprintf(ContinueRunStatsFile, "\nThe Moon material Body1/Body2 ratio	= %f\n", (float)(moonMaterialFeCountBody1 + moonMaterialSiCountBody1)/(float)(moonMaterialFeCountBody2 + moonMaterialSiCountBody2));
	
	fprintf(ContinueRunStatsFile, "\nThe mass of escape material		= %e", escapeMaterialMass*massConverter);
	fprintf(ContinueRunStatsFile, "\nThe escape material count Fe body 1	= %d", escapeMaterialFeCountBody1);
	fprintf(ContinueRunStatsFile, "\nThe escape material count Fe body 2	= %d", escapeMaterialFeCountBody2);
	fprintf(ContinueRunStatsFile, "\nThe escape material count Si body 1	= %d", escapeMaterialSiCountBody1);
	fprintf(ContinueRunStatsFile, "\nThe escape material count Si body 2	= %d", escapeMaterialSiCountBody2);
	fprintf(ContinueRunStatsFile, "\nThe escape material Body1/Body2 ratio	= %f\n", (float)(escapeMaterialFeCountBody1 + escapeMaterialSiCountBody1)/(float)(escapeMaterialFeCountBody2 + escapeMaterialSiCountBody2));
	
	fprintf(ContinueRunStatsFile, "\nThe mass of unused material		= %e", unusedMaterialMass*massConverter);
	fprintf(ContinueRunStatsFile, "\nThe unused material count Fe body 1	= %d", unusedMaterialFeCountBody1);
	fprintf(ContinueRunStatsFile, "\nThe unused material count Fe body 2	= %d", unusedMaterialFeCountBody2);
	fprintf(ContinueRunStatsFile, "\nThe unused material count Si body 1	= %d", unusedMaterialSiCountBody1);
	fprintf(ContinueRunStatsFile, "\nThe unused material count Si body 2	= %d", unusedMaterialSiCountBody2);
	fprintf(ContinueRunStatsFile, "\nThe unused material Body1/Body2 ratio	= %f\n", (float)(unusedMaterialFeCountBody1 + unusedMaterialSiCountBody1)/(float)(unusedMaterialFeCountBody2 + unusedMaterialSiCountBody2));
	
	fprintf(ContinueRunStatsFile, "\n*************************************************************************\n\n\n");
}

void printCollisionStatsToScreen(double time)
{	
	double mag, size, angle, x, y, z;
	
	double timeConverter = UnitTime;
	double lengthConverter = UnitLength;
	double massConverter = UnitMass; 
	double velocityConverter = UnitLength/UnitTime; 
	double momentumConverter = UnitMass*UnitLength*UnitLength/UnitTime;
	
	findEarthAndMoon();
	int earthFeCountBody1 = 0;
	int earthFeCountBody2 = 0;
	int earthSiCountBody1 = 0;
	int earthSiCountBody2 = 0;
	int moonFeCountBody1 = 0;
	int moonFeCountBody2 = 0;
	int moonSiCountBody1 = 0;
	int moonSiCountBody2 = 0;
	
	float massUniversalSystem = getMassCollision(0);
	float massEarthMoonSystem = getMassCollision(1);
	float massEarth = getMassCollision(2);
	float massMoon = getMassCollision(3);
	
	float3 centerOfMassUniversalSystem = getCenterOfMassCollision(0);
	float3 centerOfMassEarthMoonSystem = getCenterOfMassCollision(1);
	float3 centerOfMassEarth = getCenterOfMassCollision(2);
	float3 centerOfMassMoon = getCenterOfMassCollision(3);
	
	float3 linearVelocityUniversalSystem = getLinearVelocityCollision(0);
	float3 linearVelocityEarthMoonSystem = getLinearVelocityCollision(1);
	float3 linearVelocityEarth = getLinearVelocityCollision(2);
	float3 linearVelocityMoon = getLinearVelocityCollision(3);
	
	float3 angularMomentumUniversalSystem = getAngularMomentumCollision(0);
	float3 angularMomentumEarthMoonSystem = getAngularMomentumCollision(1);
	float3 angularMomentumEarth = getAngularMomentumCollision(2);
	float3 angularMomentumMoon = getAngularMomentumCollision(3);
	
	for(int i = 0; i < NumberOfEarthElements; i++)
	{
		if(EarthIndex[i] < NFe1) 			earthFeCountBody1++;
		else if(EarthIndex[i] < NFe1 + NFe2) 		earthFeCountBody2++;
		else if(EarthIndex[i] < NFe1 + NFe2 + NSi1) 	earthSiCountBody1++;
		else 						earthSiCountBody2++;
	}
	
	for(int i = 0; i < NumberOfMoonElements; i++)
	{
		if(MoonIndex[i] < NFe1) 			moonFeCountBody1++;
		else if(MoonIndex[i] < NFe1 + NFe2) 		moonFeCountBody2++;
		else if(MoonIndex[i] < NFe1 + NFe2 + NSi1) 	moonSiCountBody1++;
		else 						moonSiCountBody2++;
	}
	
	printf("\n\n\n*************************************************************************\n\n\n");
	printf("\nThe following are the stats of the run when time = %f hours\n", time*timeConverter/3600.0);
	printf("\nDistance is measured in Kilometers");
	printf("\nMass is measured in Kilograms");
	printf("\nTime is measured in seconds");
	printf("\nVelocity is measured in Kilometers/second");
	printf("\nAngular momentun is measured in Kilograms*Kilometers*Kilometers/seconds\n");
	
	printf("\nThe mass of Earth 		= %e", massEarth*massConverter);
	printf("\nThe mass of Moon 		= %e", massMoon*massConverter);
	if(massMoon != 0.0) printf("\nThe mass ratio Earth/Moon 	= %f\n", massEarth/massMoon);
	
	printf("\nMoon iron from body 1 		= %d", moonFeCountBody1);
	printf("\nMoon silicate from body 1 	= %d", moonSiCountBody1);
	printf("\nMoon iron from body 2 		= %d", moonFeCountBody2);
	printf("\nMoon silicate from body 2 	= %d", moonSiCountBody2);
	if((moonFeCountBody2 + moonSiCountBody2) == 0)
	{
		printf("\nThe Moon is only composed of elements from body 1\n");
	}
	else if((moonFeCountBody1 + moonSiCountBody1) == 0)
	{
		printf("\nThe Moon is only composed of elements from body 2\n");
	}
	else
	{
		printf("\nMoon ratio body1/body2 		= %f\n", (float)(moonFeCountBody1 + moonSiCountBody1)/(float)(moonFeCountBody2 + moonSiCountBody2));
	}
	
	printf("\nEarth iron from body 1 		= %d", earthFeCountBody1);
	printf("\nEarth silicate from body 1 	= %d", earthSiCountBody1);
	printf("\nEarth iron from body 2 		= %d", earthFeCountBody2);
	printf("\nEarth silicate from body 2 	= %d", earthSiCountBody2);
	if((earthFeCountBody2 + earthSiCountBody2) == 0)
	{
		printf("\nThe Earth is only composed of elements from body 1\n");
	}
	else if((earthFeCountBody1 + earthSiCountBody1) == 0)
	{
		printf("\nThe Earth is only composed of elements from body 2\n");
	}
	else
	{
		printf("\nEarth ratio body1/body2 		= %f\n", (float)(earthFeCountBody1 + earthSiCountBody1)/(float)(earthFeCountBody2 + earthSiCountBody2));
	}
	
	//It is always assumed that the ecliptic plane is the xz-plane.
	x = angularMomentumEarthMoonSystem.x*momentumConverter;
	y = angularMomentumEarthMoonSystem.y*momentumConverter;
	z = angularMomentumEarthMoonSystem.z*momentumConverter;
	printf("\nAngular momentum of the Earth Moon system 		= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	printf("\nMagnitude of the angular momentum of the system 	= %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	printf("\nAngle off ecliptic plane of the system's rotation 	= %f\n", 90.0 - angle*180.0/Pi);
	
	x = angularMomentumEarth.x*momentumConverter;
	y = angularMomentumEarth.y*momentumConverter;
	z = angularMomentumEarth.z*momentumConverter;
	printf("\nAngular momentum of the Earth 				= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	printf("\nMagnitude of the angular momentum of the Earth 		= %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	printf("\nAngle off ecliptic plane of the Earth's rotation 	= %f\n", 90.0 - angle*180.0/Pi);
	
	x = angularMomentumMoon.x*momentumConverter;
	y = angularMomentumMoon.y*momentumConverter;
	z = angularMomentumMoon.z*momentumConverter;
	printf("\nAngular momentum of the Moon 				= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	printf("\nMagnitude of the angular momentum of the Moon 		= %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	printf("\nAngle off ecliptic plane of the Moon's rotation 	= %f\n", 90.0 - angle*180.0/Pi);
	
	x = centerOfMassEarthMoonSystem.x*lengthConverter;
	y = centerOfMassEarthMoonSystem.y*lengthConverter;
	z = centerOfMassEarthMoonSystem.z*lengthConverter;
	printf("\nCenter of mass of the Earth-Moon system 		= (%f, %f, %f)", x, y, z);
	
	x = centerOfMassEarth.x*lengthConverter;
	y = centerOfMassEarth.y*lengthConverter;
	z = centerOfMassEarth.z*lengthConverter;
	printf("\nCenter of mass of the Earth system 			= (%f, %f, %f)", x, y, z);
	
	x = centerOfMassMoon.x*lengthConverter;
	y = centerOfMassMoon.y*lengthConverter;
	z = centerOfMassMoon.z*lengthConverter;
	printf("\nCenter of mass of the Moon system 			= (%f, %f, %f)\n", x, y, z);
	
	x = linearVelocityEarthMoonSystem.x*velocityConverter;
	y = linearVelocityEarthMoonSystem.y*velocityConverter;
	z = linearVelocityEarthMoonSystem.z*velocityConverter;
	printf("\nLinear Velocity of the Earth-Moon system 		= (%f, %f, %f)", x, y, z);
	
	x = linearVelocityEarth.x*velocityConverter;
	y = linearVelocityEarth.y*velocityConverter;
	z = linearVelocityEarth.z*velocityConverter;
	printf("\nLinear Velocity of the Earth system 			= (%f, %f, %f)", x, y, z);
	
	x = linearVelocityMoon.x*velocityConverter;
	y = linearVelocityMoon.y*velocityConverter;
	z = linearVelocityMoon.z*velocityConverter;
	printf("\nLinear Velocity of the Moon system 			= (%f, %f, %f)\n", x, y, z);
	
	printf("\n*****Stats of the entire system to check the numerical scheme's validity*****\n");
	
	x = centerOfMassUniversalSystem.x*lengthConverter;
	y = centerOfMassUniversalSystem.y*lengthConverter;
	z = centerOfMassUniversalSystem.z*lengthConverter;
	printf("\nCenter of mass of the entire system 		        = (%f, %f, %f)\n", x, y, z);
	
	x = linearVelocityUniversalSystem.x*velocityConverter;
	y = linearVelocityUniversalSystem.y*velocityConverter;
	z = linearVelocityUniversalSystem.z*velocityConverter;
	printf("\nLinear velocity of the entire system system 		= (%f, %f, %f)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	printf("\nMagnitude of the linear velocity of the entire system 	= %f\n", mag);
	
	x = angularMomentumUniversalSystem.x*momentumConverter;
	y = angularMomentumUniversalSystem.y*momentumConverter;
	z = angularMomentumUniversalSystem.z*momentumConverter;
	printf("\nAngular momentum of the entire system system 		= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	printf("\nMagnitude of the angular momentum of the entire system 	= %e\n", mag);
	
	printf("\n*************************************************************************\n");
	
	printf("\n******************* Just the good stuff *********************************\n");
	
	printf("\n percent off correct Earth mass = %f ", 100.0*(massEarth*massConverter/(MassOfEarth)));
	printf("\n percent off correct Moon mass  = %f ", 100.0*(massMoon*massConverter/(MassOfMoon)));
	printf("\n\n Earth mass percent iron = %f mass percent silicate = %f", float(earthFeCountBody1*MassFe + earthFeCountBody2*MassFe)/massEarth, float(earthSiCountBody1*MassSi + earthSiCountBody2*MassSi)/massEarth);
	printf("\n Moon mass percent iron = %f mass percent silicate = %f", float(moonFeCountBody1*MassFe + moonFeCountBody2*MassFe)/massMoon, float(moonSiCountBody1*MassSi + moonSiCountBody2*MassSi)/massMoon);
	if((moonFeCountBody2 + moonSiCountBody2) != 0)
	{
		printf("\n\n Moon body1/body2 ratio     = %f ", float(moonFeCountBody1*MassFe + moonSiCountBody1*MassSi)/float(moonFeCountBody2*MassFe + moonSiCountBody2*MassSi));
	}
	
	x = angularMomentumEarthMoonSystem.x*momentumConverter;
	y = angularMomentumEarthMoonSystem.y*momentumConverter;
	z = angularMomentumEarthMoonSystem.z*momentumConverter;
	mag = sqrt(x*x + y*y + z*z);
	printf("\n Percent off correct angular momentum of the Earth-Moon System = %f ", 100.0*(1.0 - mag/AngularMomentumEarthMoonSystem));
	
	x = angularMomentumEarth.x*momentumConverter;
	y = angularMomentumEarth.y*momentumConverter;
	z = angularMomentumEarth.z*momentumConverter;
	mag = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/mag);
	printf("\n Percent off correct axial tilt of the Earth = %f ", 100.0*(1.0 - angle/EarthAxialTilt));
	
	
	printf("\n\n*************************************************************************\n\n\n");
}

void recordFinalCollisionStat(double time)
{
	double mag, size, angle, x, y, z;
	
	double timeConverter = UnitTime;
	double lengthConverter = UnitLength;
	double massConverter = UnitMass; 
	double velocityConverter = UnitLength/UnitTime; 
	double momentumConverter = UnitMass*UnitLength*UnitLength/UnitTime;
	
	findEarthAndMoon();
	int earthFeCountBody1 = 0;
	int earthFeCountBody2 = 0;
	int earthSiCountBody1 = 0;
	int earthSiCountBody2 = 0;
	int moonFeCountBody1 = 0;
	int moonFeCountBody2 = 0;
	int moonSiCountBody1 = 0;
	int moonSiCountBody2 = 0;
	
	float massUniversalSystem = getMassCollision(0);
	float massEarthMoonSystem = getMassCollision(1);
	float massEarth = getMassCollision(2);
	float massMoon = getMassCollision(3);
	
	float3 centerOfMassUniversalSystem = getCenterOfMassCollision(0);
	float3 centerOfMassEarthMoonSystem = getCenterOfMassCollision(1);
	float3 centerOfMassEarth = getCenterOfMassCollision(2);
	float3 centerOfMassMoon = getCenterOfMassCollision(3);
	
	float3 linearVelocityUniversalSystem = getLinearVelocityCollision(0);
	float3 linearVelocityEarthMoonSystem = getLinearVelocityCollision(1);
	float3 linearVelocityEarth = getLinearVelocityCollision(2);
	float3 linearVelocityMoon = getLinearVelocityCollision(3);
	
	float3 angularMomentumUniversalSystem = getAngularMomentumCollision(0);
	float3 angularMomentumEarthMoonSystem = getAngularMomentumCollision(1);
	float3 angularMomentumEarth = getAngularMomentumCollision(2);
	float3 angularMomentumMoon = getAngularMomentumCollision(3);
	
	for(int i = 0; i < NumberOfEarthElements; i++)
	{
		if(EarthIndex[i] < NFe1) 			earthFeCountBody1++;
		else if(EarthIndex[i] < NFe1 + NFe2) 		earthFeCountBody2++;
		else if(EarthIndex[i] < NFe1 + NFe2 + NSi1) 	earthSiCountBody1++;
		else 						earthSiCountBody2++;
	}
	
	for(int i = 0; i < NumberOfMoonElements; i++)
	{
		if(MoonIndex[i] < NFe1) 			moonFeCountBody1++;
		else if(MoonIndex[i] < NFe1 + NFe2) 		moonFeCountBody2++;
		else if(MoonIndex[i] < NFe1 + NFe2 + NSi1) 	moonSiCountBody1++;
		else 						moonSiCountBody2++;
	}
	
	fprintf(RunStatsFile,"\n\n\n*************************************************************************\n\n");
	fprintf(RunStatsFile,"\nThe following are the final stats of the run when time = %f hours\n", time*timeConverter/3600.0);
	fprintf(RunStatsFile,"\nDistance is measured in Kilometers");
	fprintf(RunStatsFile,"\nMass is measured in Kilograms");
	fprintf(RunStatsFile,"\nTime is measured in seconds");
	fprintf(RunStatsFile,"\nVelocity is measured in Kilometers/second");
	fprintf(RunStatsFile,"\nAngular momentun is measured in Kilograms*Kilometers*Kilometers/seconds\n");
	
	fprintf(RunStatsFile,"\nThe mass of Earth 		= %e", massEarth*massConverter);
	fprintf(RunStatsFile,"\nThe mass of Moon 		= %e", massMoon*massConverter);
	if(massMoon != 0.0) fprintf(RunStatsFile,"\nThe mass ratio Earth/Moon 	= %f\n", massEarth/massMoon);
	
	fprintf(RunStatsFile,"\nMoon iron from body 1 		= %d", moonFeCountBody1);
	fprintf(RunStatsFile,"\nMoon silicate from body 1 	= %d", moonSiCountBody1);
	fprintf(RunStatsFile,"\nMoon iron from body 2 		= %d", moonFeCountBody2);
	fprintf(RunStatsFile,"\nMoon silicate from body 2 	= %d", moonSiCountBody2);
	if((moonFeCountBody2 + moonSiCountBody2) == 0)
	{
		fprintf(RunStatsFile,"\nThe Moon is only composed of elements from body 1\n");
	}
	else if((moonFeCountBody1 + moonSiCountBody1) == 0)
	{
		fprintf(RunStatsFile,"\nThe Moon is only composed of elements from body 2\n");
	}
	else
	{
		fprintf(RunStatsFile,"\nMoon ratio body1/body2 		= %f\n", (float)(moonFeCountBody1 + moonSiCountBody1)/(float)(moonFeCountBody2 + moonSiCountBody2));
	}
	
	fprintf(RunStatsFile,"\nEarth iron from body 1 		= %d", earthFeCountBody1);
	fprintf(RunStatsFile,"\nEarth silicate from body 1 	= %d", earthSiCountBody1);
	fprintf(RunStatsFile,"\nEarth iron from body 2 		= %d", earthFeCountBody2);
	fprintf(RunStatsFile,"\nEarth silicate from body 2 	= %d", earthSiCountBody2);
	if((earthFeCountBody2 + earthSiCountBody2) == 0)
	{
		fprintf(RunStatsFile,"\nThe Earth is only composed of elements from body 1\n");
	}
	else if((earthFeCountBody1 + earthSiCountBody1) == 0)
	{
		fprintf(RunStatsFile,"\nThe Earth is only composed of elements from body 2\n");
	}
	else
	{

		fprintf(RunStatsFile,"\nEarth ratio body1/body2 		= %f\n", (float)(earthFeCountBody1 + earthSiCountBody1)/(float)(earthFeCountBody2 + earthSiCountBody2));
	}
	
	//It is always assumed that the ecliptic plane is the xz-plane.
	x = angularMomentumEarthMoonSystem.x*momentumConverter;
	y = angularMomentumEarthMoonSystem.y*momentumConverter;
	z = angularMomentumEarthMoonSystem.z*momentumConverter;
	fprintf(RunStatsFile,"\nAngular momentum of the Earth Moon system 		= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	fprintf(RunStatsFile,"\nMagnitude of the angular momentum of the system 	= %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	fprintf(RunStatsFile,"\nAngle off ecliptic plane of the system's rotation 	= %f\n", 90.0 - angle*180.0/Pi);
	
	x = angularMomentumEarth.x*momentumConverter;
	y = angularMomentumEarth.y*momentumConverter;
	z = angularMomentumEarth.z*momentumConverter;
	fprintf(RunStatsFile,"\nAngular momentum of the Earth 				= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	fprintf(RunStatsFile,"\nMagnitude of the angular momentum of the Earth 		= %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	fprintf(RunStatsFile,"\nAngle off ecliptic plane of the Earth's rotation 	= %f\n", 90.0 - angle*180.0/Pi);
	
	x = angularMomentumMoon.x*momentumConverter;
	y = angularMomentumMoon.y*momentumConverter;
	z = angularMomentumMoon.z*momentumConverter;
	fprintf(RunStatsFile,"\nAngular momentum of the Moon 				= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	fprintf(RunStatsFile,"\nMagnitude of the angular momentum of the Moon 		= %e", mag);
	size = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/size);
	fprintf(RunStatsFile,"\nAngle off ecliptic plane of the Moon's rotation 	= %f\n", 90.0 - angle*180.0/Pi);
	
	x = centerOfMassEarthMoonSystem.x*lengthConverter;
	y = centerOfMassEarthMoonSystem.y*lengthConverter;
	z = centerOfMassEarthMoonSystem.z*lengthConverter;
	fprintf(RunStatsFile,"\nCenter of mass of the Earth-Moon system 		= (%f, %f, %f)", x, y, z);
	
	x = centerOfMassEarth.x*lengthConverter;
	y = centerOfMassEarth.y*lengthConverter;
	z = centerOfMassEarth.z*lengthConverter;
	fprintf(RunStatsFile,"\nCenter of mass of the Earth system 			= (%f, %f, %f)", x, y, z);
	
	x = centerOfMassMoon.x*lengthConverter;
	y = centerOfMassMoon.y*lengthConverter;
	z = centerOfMassMoon.z*lengthConverter;
	fprintf(RunStatsFile,"\nCenter of mass of the Moon system 			= (%f, %f, %f)\n", x, y, z);
	
	x = linearVelocityEarthMoonSystem.x*velocityConverter;
	y = linearVelocityEarthMoonSystem.y*velocityConverter;
	z = linearVelocityEarthMoonSystem.z*velocityConverter;
	fprintf(RunStatsFile,"\nLinear Velocity of the Earth-Moon system 		= (%f, %f, %f)", x, y, z);
	
	x = linearVelocityEarth.x*velocityConverter;
	y = linearVelocityEarth.y*velocityConverter;
	z = linearVelocityEarth.z*velocityConverter;
	fprintf(RunStatsFile,"\nLinear Velocity of the Earth system 			= (%f, %f, %f)", x, y, z);
	
	x = linearVelocityMoon.x*velocityConverter;
	y = linearVelocityMoon.y*velocityConverter;
	z = linearVelocityMoon.z*velocityConverter;
	fprintf(RunStatsFile,"\nLinear Velocity of the Moon system 			= (%f, %f, %f)\n", x, y, z);
	
	fprintf(RunStatsFile,"\n*****Stats of the entire system to check the numerical scheme's validity*****\n");
	
	x = centerOfMassUniversalSystem.x*lengthConverter;
	y = centerOfMassUniversalSystem.y*lengthConverter;
	z = centerOfMassUniversalSystem.z*lengthConverter;
	fprintf(RunStatsFile,"\nCenter of mass of the entire system 		        = (%f, %f, %f)\n", x, y, z);
	
	x = linearVelocityUniversalSystem.x*velocityConverter;
	y = linearVelocityUniversalSystem.y*velocityConverter;
	z = linearVelocityUniversalSystem.z*velocityConverter;
	fprintf(RunStatsFile,"\nLinear velocity of the entire system system 		= (%f, %f, %f)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	fprintf(RunStatsFile,"\nMagnitude of the linear velocity of the entire system 	= %f\n", mag);
	
	x = angularMomentumUniversalSystem.x*momentumConverter;
	y = angularMomentumUniversalSystem.y*momentumConverter;
	z = angularMomentumUniversalSystem.z*momentumConverter;
	fprintf(RunStatsFile,"\nAngular momentum of the entire system system 		= (%e, %e, %e)", x, y, z);
	mag = sqrt(x*x + y*y + z*z);
	fprintf(RunStatsFile,"\nMagnitude of the angular momentum of the entire system 	= %e\n", mag);
	
	fprintf(RunStatsFile,"\n*************************************************************************\n");
	
	fprintf(RunStatsFile,"\n******************* Just the good stuff *********************************\n");
	
	fprintf(RunStatsFile,"\n percent off correct Earth mass = %f ", 100.0*(massEarth*massConverter/(MassOfEarth)));
	fprintf(RunStatsFile,"\n percent off correct Moon mass  = %f ", 100.0*(massMoon*massConverter/(MassOfMoon)));
	fprintf(RunStatsFile,"\n\n Earth mass percent iron = %f mass percent silicate = %f", float(earthFeCountBody1*MassFe + earthFeCountBody2*MassFe)/massEarth, float(earthSiCountBody1*MassSi + earthSiCountBody2*MassSi)/massEarth);
	fprintf(RunStatsFile,"\n Moon mass percent iron = %f mass percent silicate = %f", float(moonFeCountBody1*MassFe + moonFeCountBody2*MassFe)/massMoon, float(moonSiCountBody1*MassSi + moonSiCountBody2*MassSi)/massMoon);
	if((moonFeCountBody2 + moonSiCountBody2) != 0)
	{
		fprintf(RunStatsFile,"\n\n Moon body1/body2 ratio     = %f ", float(moonFeCountBody1*MassFe + moonSiCountBody1*MassSi)/float(moonFeCountBody2*MassFe + moonSiCountBody2*MassSi));
	}
	
	x = angularMomentumEarthMoonSystem.x*momentumConverter;
	y = angularMomentumEarthMoonSystem.y*momentumConverter;
	z = angularMomentumEarthMoonSystem.z*momentumConverter;
	mag = sqrt(x*x + y*y + z*z);
	fprintf(RunStatsFile,"\n Percent off correct angular momentum of the Earth-Moon System = %f ", 100.0*(1.0 - mag/AngularMomentumEarthMoonSystem));
	
	x = angularMomentumEarth.x*momentumConverter;
	y = angularMomentumEarth.y*momentumConverter;
	z = angularMomentumEarth.z*momentumConverter;
	mag = sqrt(x*x + y*y + z*z) * sqrt(x*x + z*z);
	angle = acos((x*x + z*z)/mag);
	fprintf(RunStatsFile,"\n Percent off correct axial tilt of the Earth = %f ", 100.0*(1.0 - angle/EarthAxialTilt));
	
	fprintf(RunStatsFile,"\n\n*************************************************************************\n\n\n");
}

void recordPosAndVel()
{
	fwrite(Pos, sizeof(float4), N, PosAndVelFile);
	fwrite(Vel, sizeof(float4), N, PosAndVelFile);
}

void recordContinuePosAndVel(double time)
{
	fwrite(&time, sizeof(double), 1, ContinueRunPosAndVelFile);
	fwrite(Pos, sizeof(float4), N, ContinueRunPosAndVelFile);
	fwrite(Vel, sizeof(float4), N, ContinueRunPosAndVelFile);
}

void drawSimplePictureSeperate()
{
	float3 centerOfMass1 = getCenterOfMassSeperate(1);
	float3 centerOfMass2 = getCenterOfMassSeperate(2);
	float3 linearVelocity1 = getLinearVelocitySeperate(1);
	float3 linearVelocity2 = getLinearVelocitySeperate(2);
	float3 angularMomentum1 = getAngularMomentumSeperate(1, centerOfMass1, linearVelocity1);
	float3 angularMomentum2 = getAngularMomentumSeperate(2, centerOfMass2, linearVelocity2);
	float Stretch;
	
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	
	//Coloring all the elements 
	glBegin(GL_POINTS);
     		for(int i=0; i<N; i++)
		{
			if(i < NFe1) 
			{
		    		glColor3d(1.0,0.0,0.0);
			}
			else if(i < NFe1 + NSi1)
			{
				glColor3d(1.0,1.0,0.5);
			}
			else if(i < NFe1 + NSi1 + NFe2) 
			{
		    		glColor3d(1.0,0.0,1.0);
			}
			else
			{
				glColor3d(0.0,0.5,0.0);
			}
			
			glVertex3f(Pos[i].x, Pos[i].y, Pos[i].z);
		}
	glEnd();

	glLineWidth(1.0);
	//Placing a green vector in the direction of the disired linear motion of each body
	glColor3f(0.0,1.0,0.0);
	Stretch = 1.0;
	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMass1.x, centerOfMass1.y, centerOfMass1.z);
		glVertex3f(centerOfMass1.x + InitialVelocity1.x*Stretch, centerOfMass1.y + InitialVelocity1.y*Stretch, centerOfMass1.z + InitialVelocity1.z*Stretch);
	glEnd();

	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMass2.x, centerOfMass2.y, centerOfMass2.z);
		glVertex3f(centerOfMass2.x + InitialVelocity2.x*Stretch, centerOfMass2.y + InitialVelocity2.y*Stretch, centerOfMass2.z + InitialVelocity2.z*Stretch);
	glEnd();
	
	//Placing a yellow vector in the direction of the actual linear motion of each body
	glColor3f(1.0,1.0,0.0);
	Stretch = 30.0;
	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMass1.x, centerOfMass1.y, centerOfMass1.z);
		glVertex3f(centerOfMass1.x + linearVelocity1.x*Stretch, centerOfMass1.y + linearVelocity1.y*Stretch, centerOfMass1.z + linearVelocity1.z*Stretch);
	glEnd();

	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMass2.x, centerOfMass2.y, centerOfMass2.z);
		glVertex3f(centerOfMass2.x + linearVelocity2.x*Stretch, centerOfMass2.y + linearVelocity2.y*Stretch, centerOfMass2.z + linearVelocity2.z*Stretch);
	glEnd();
	
	//Placing a blue vector in the direction of the disired angular momentum 
	glColor3f(0.0,0.0,1.0);	
	Stretch = 50.0;
	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMass1.x, centerOfMass1.y, centerOfMass1.z);
		glVertex3f(centerOfMass1.x + InitialSpin1.x*Stretch, centerOfMass1.y + InitialSpin1.y*Stretch, centerOfMass1.z + InitialSpin1.z*Stretch);
	glEnd();
	
	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMass2.x, centerOfMass2.y, centerOfMass2.z);
		glVertex3f(centerOfMass2.x + InitialSpin2.x*Stretch, centerOfMass2.y + InitialSpin2.y*Stretch, centerOfMass2.z + InitialSpin2.z*Stretch);
	glEnd();
	
	//Placing a red vector in the direction of the actual angular momentum 
	glColor3f(1.0,0.0,0.0);	
	Stretch = 50.0;
	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMass1.x, centerOfMass1.y, centerOfMass1.z);
		glVertex3f(centerOfMass1.x + angularMomentum1.x*Stretch, centerOfMass1.y + angularMomentum1.y*Stretch, centerOfMass1.z + angularMomentum1.z*Stretch);
	glEnd();
	
	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMass2.x, centerOfMass2.y, centerOfMass2.z);
		glVertex3f(centerOfMass2.x + angularMomentum2.x*Stretch, centerOfMass2.y + angularMomentum2.y*Stretch, centerOfMass2.z + angularMomentum2.z*Stretch);
	glEnd();
	
	glutSwapBuffers();
}

void drawPictureCollision()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glBegin(GL_POINTS);
     		for(int i=0; i<N; i++)
		{
			if(i < NFe1) 
			{
		    		glColor3d(1.0,0.0,0.0);
			}
			else if(i < NFe1 + NFe2)
			{
				glColor3d(1.0,0.0,1.0);
			}
			else if(i < NFe1 + NFe2 + NSi1) 
			{
				glColor3d(1.0,1.0,0.5);
			}
			else
			{
				glColor3d(0.0,0.5,0.0);
			}
			
			glVertex3f(Pos[i].x, Pos[i].y, Pos[i].z);
		}
	glEnd();
	
	glutSwapBuffers();
}

void drawAnalysisPictureCollision()
{
	int i;
	
	findEarthAndMoon();
	float massSystem = getMassCollision(0);
	float massEarth = getMassCollision(1);
	float massMoon = getMassCollision(2);
	float3 centerOfMassSystem = getCenterOfMassCollision(0);
	float3 centerOfMassEarth = getCenterOfMassCollision(1);
	float3 centerOfMassMoon = getCenterOfMassCollision(2);
	float3 linearVelocitySystem = getLinearVelocityCollision(0);
	float3 linearVelocityEarth = getLinearVelocityCollision(1);
	float3 linearVelocityMoon = getLinearVelocityCollision(2);
	float3 angularMomentumSystem = getAngularMomentumCollision(0);
	float3 angularMomentumEarth = getAngularMomentumCollision(1);
	float3 angularMomentumMoon = getAngularMomentumCollision(2);
	float Stretch;
	
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	
	//Coloring all the elements
	glPointSize(1.0);
	glBegin(GL_POINTS);
     		for(i=0; i<N; i++)
		{
			if(i < NFe1) 
			{
		    		glColor3d(1.0,0.0,0.0);
			}
			else if(i < NFe1 + NFe2)
			{
				glColor3d(1.0,0.0,1.0);
			}
			else if(i < NFe1 + NFe2 + NSi1) 
			{
				glColor3d(1.0,1.0,0.5);
			}
			else
			{
				glColor3d(0.0,0.5,0.0);
			}
			
			glVertex3f(Pos[i].x, Pos[i].y, Pos[i].z);
		}
	glEnd();
	
	glPointSize(1.0);
	//Recoloring the Earth elements blue
	glColor3d(0.0,0.0,1.0);
	glBegin(GL_POINTS);
		for(i = 0; i < NumberOfEarthElements; i++)
		{	
				glVertex3f(Pos[EarthIndex[i]].x, Pos[EarthIndex[i]].y, Pos[EarthIndex[i]].z);
		}
	glEnd();
	
	//Recoloring the Moon elements red
	glColor3d(1.0,0.0,0.0);
	glBegin(GL_POINTS);
		for(i = 0; i < NumberOfMoonElements; i++)
		{	
			glVertex3f(Pos[MoonIndex[i]].x, Pos[MoonIndex[i]].y, Pos[MoonIndex[i]].z);
		}
	glEnd();

	glLineWidth(1.0);
	//Placing green vectors in the direction of linear velocity of the Moon
	Stretch = 1.0;
	glColor3f(0.0,1.0,0.0);
	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMassMoon.x, centerOfMassMoon.y, centerOfMassMoon.z);
		glVertex3f(	centerOfMassMoon.x + linearVelocityMoon.x*Stretch, 
				centerOfMassMoon.y + linearVelocityMoon.y*Stretch, 
				centerOfMassMoon.z + linearVelocityMoon.z*Stretch);
	glEnd();
	
	//Place a white point at the center of mass of the Earth-Moon system
	glColor3d(1.0,1.0,1.0);
	glPointSize(10.0);
	glBegin(GL_POINTS);
		glVertex3f(centerOfMassSystem.x, centerOfMassSystem.y, centerOfMassSystem.z);
	glEnd();
	
	//Place a yellow point at the center of mass of the Earth
	glColor3d(1.0,1.0,0.0);
	glPointSize(5.0);
	glBegin(GL_POINTS);
		glVertex3f(centerOfMassEarth.x, centerOfMassEarth.y, centerOfMassEarth.z);
	glEnd();
	
	//Place a yellow point at the center of mass of the Moon
	glColor3d(1.0,1.0,0.0);
	glPointSize(5.0);
	glBegin(GL_POINTS);
		glVertex3f(centerOfMassMoon.x, centerOfMassMoon.y, centerOfMassMoon.z);
	glEnd();
	
	//Placing white vectors in the direction of the angular momentum of the Earth-Moon system
	glColor3f(1.0,1.0,1.0);
	Stretch = 1.0;
	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMassSystem.x, centerOfMassSystem.y, centerOfMassSystem.z);
		glVertex3f(	centerOfMassSystem.x + angularMomentumSystem.x*Stretch/massSystem, 
				centerOfMassSystem.y + angularMomentumSystem.y*Stretch/massSystem, 
				centerOfMassSystem.z + angularMomentumSystem.z*Stretch/massSystem);
	glEnd();
	
	//Placing blue vectors in the direction of the angular momentum of the Earth
	Stretch = 1.0;
	glBegin(GL_LINE_LOOP);
	glColor3f(0.0,0.0,1.0);
		glVertex3f(centerOfMassEarth.x, centerOfMassEarth.y, centerOfMassEarth.z);
		glVertex3f(	centerOfMassEarth.x + angularMomentumEarth.x*Stretch/massEarth, 
				centerOfMassEarth.y + angularMomentumEarth.y*Stretch/massEarth, 
				centerOfMassEarth.z + angularMomentumEarth.z*Stretch/massEarth);
	glEnd();
	
	//Placing red vectors in the direction of the angular momentum of the Moon
	Stretch = 1.0;
	glColor3f(1.0,0.0,0.0);
	glBegin(GL_LINE_LOOP);
		glVertex3f(centerOfMassMoon.x, centerOfMassMoon.y, centerOfMassMoon.z);
		glVertex3f(	centerOfMassMoon.x + angularMomentumMoon.x*Stretch/massMoon, 
				centerOfMassMoon.y + angularMomentumMoon.y*Stretch/massMoon, 
				centerOfMassMoon.z + angularMomentumMoon.z*Stretch/massMoon);
	glEnd();
	
	glutSwapBuffers();
	
	free(EarthIndex);
	free(MoonIndex);
}

void transformInitialConditionsFromSeperateToCollision()
{
	int k;
	hipMemcpy( PlaceHolder, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpy Pos2");
	k = 0;
	for(int i = 0; i < NFe1; i++)
	{
		Pos[k] = PlaceHolder[i];
		k++;
	}
	for(int i = NFe1 + NSi1; i < NFe1 + NSi1 + NFe2; i++)
	{
		Pos[k] = PlaceHolder[i];
		k++;
	}
	for(int i = NFe1; i < NFe1 + NSi1; i++)
	{
		Pos[k] = PlaceHolder[i];
		k++;
	}
	for(int i = NFe1 + NSi1 + NFe2; i < N; i++)
	{
		Pos[k] = PlaceHolder[i];
		k++;
	}
	
	hipMemcpy( PlaceHolder, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpy Vel");
	k = 0;
	for(int i = 0; i < NFe1; i++)
	{
		Vel[k] = PlaceHolder[i];
		k++;
	}
	for(int i = NFe1 + NSi1; i < NFe1 + NSi1 + NFe2; i++)
	{
		Vel[k] = PlaceHolder[i];
		k++;
	}
	for(int i = NFe1; i < NFe1 + NSi1; i++)
	{
		Vel[k] = PlaceHolder[i];
		k++;
	}
	for(int i = NFe1 + NSi1 + NFe2; i < N; i++)
	{
		Vel[k] = PlaceHolder[i];
		k++;
	}
}

void nBodySeperate()
{ 
	float time = 0.0;
	int   tdraw = 1;
	
	int dampCheck = 0;
	int rest1Check = 0;
	int spinCheck = 0;
	
    	hipMemcpy( Pos_DEV0, Pos, N *sizeof(float4), hipMemcpyHostToDevice );
    	errorCheck("hipMemcpy Pos3");
    	hipMemcpy( Vel_DEV0, Vel, N *sizeof(float4), hipMemcpyHostToDevice );
    	errorCheck("hipMemcpy Vel");
   
	while(time < SetupTime)
	{	
		getForcesSeperate<<<GridConfig, BlockConfig>>>(Pos_DEV0, Vel_DEV0, Force_DEV0, ForceSeperateConstant);
		if(time < DampTime) 
		{
			if(dampCheck == 0)
			{
				printf("\n************************************************** Damping is on\n");
				dampCheck = 1;
				tdraw = 0;
			}
			moveBodiesDampedSeperate<<<GridConfig, BlockConfig>>>(Pos_DEV0, Vel_DEV0, Force_DEV0, MoveSeperateConstant, DampRateBody1, DampRateBody2);
		}
		else if(time < DampTime + DampRestTime)
		{
			if(rest1Check == 0)
			{
				printf("\n************************************************** Damp rest stage is on\n");
				rest1Check = 1;
				tdraw = 0;
			}
			moveBodiesSeperate<<<GridConfig, BlockConfig>>>(Pos_DEV0, Vel_DEV0, Force_DEV0, MoveSeperateConstant);
		}
		else
		{
			if(spinCheck == 0)
			{
				hipMemcpy( Pos, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
				errorCheck("hipMemcpy Pos4");
				hipMemcpy( Vel, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
				errorCheck("hipMemcpy Vel");
				spinBodySeperate(1, InitialSpin1);
				spinBodySeperate(2, InitialSpin2);
				hipMemcpy( Pos_DEV0, Pos, N *sizeof(float4), hipMemcpyHostToDevice );
				errorCheck("hipMemcpy Pos5");
	    			hipMemcpy( Vel_DEV0, Vel, N *sizeof(float4), hipMemcpyHostToDevice );
	    			errorCheck("hipMemcpy Vel");
				printf("\n************************************************** bodies have been spun\n");
				printf("\n************************************************** spin rest stage is on\n");
				spinCheck = 1;
			}
			moveBodiesSeperate<<<GridConfig, BlockConfig>>>(Pos_DEV0, Vel_DEV0, Force_DEV0, MoveSeperateConstant);
		}
    
		if(tdraw == DrawRate) 
		{
		    	hipMemcpy( Pos, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
		    	errorCheck("hipMemcpy Pos6");
		    	hipMemcpy( Vel, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
		    	errorCheck("hipMemcpy Vel");
		    	drawSimplePictureSeperate();
			//drawPictureSeperate();
			printf("\nSetup time in hours = %f\n", time*UnitTime/3600.0);
			tdraw = 0;
		}
		tdraw++;
		
		time += Dt;
	}
}

void resetInitialConditions()
{
	hipMemcpy( Pos, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpy Pos7");
	hipMemcpy( Vel, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpy Vel");
	setBodyPositionSeperate(1, InitialPosition1.x, InitialPosition1.y, InitialPosition1.z);
	setBodyVelocitySeperate(1, InitialVelocity1.x, InitialVelocity1.y, InitialVelocity1.z);
	setBodyPositionSeperate(2, InitialPosition2.x, InitialPosition2.y, InitialPosition2.z);
	setBodyVelocitySeperate(2, InitialVelocity2.x, InitialVelocity2.y, InitialVelocity2.z);
	printf("\n************************************************** Initial velocities have been given\n");
	hipMemcpy( Pos_DEV0, Pos, N *sizeof(float4), hipMemcpyHostToDevice );
	errorCheck("hipMemcpy Pos8");
    	hipMemcpy( Vel_DEV0, Vel, N *sizeof(float4), hipMemcpyHostToDevice );
    	errorCheck("hipMemcpy Vel");
	printf("\n************************************************** The bodies have been created and intialized\n");
}	

void copyCreatedBodiesUpToDevice()
{	
	if(NumberOfGpus == 1 || UseMultipleGPU == 0)
	{
		hipMemcpy( Pos_DEV0, Pos, N *sizeof(float4), hipMemcpyHostToDevice );
		errorCheck("hipMemcpy Pos9");
		hipMemcpy( Vel_DEV0, Vel, N *sizeof(float4), hipMemcpyHostToDevice );
		errorCheck("hipMemcpy Vel");
	}
	else
	{
		hipSetDevice(0);
		errorCheck("hipSetDevice 0");
		hipMemcpyAsync( PosFstHalf_0, Pos, (N/2)*sizeof(float4), hipMemcpyHostToDevice );
		errorCheck("hipMemcpyAsync PosFstHalf 0");
		hipMemcpyAsync( PosSndHalf_0, Pos+(N/2), (N/2)*sizeof(float4), hipMemcpyHostToDevice );
		errorCheck("hipMemcpyAsync PosSndHalf 0");
		hipMemcpyAsync( VelFstHalf_0, Vel, (N/2)*sizeof(float4), hipMemcpyHostToDevice );
		errorCheck("hipMemcpyAsync VelFstHalf 0");
		hipMemcpyAsync( VelSndHalf_0, Vel+(N/2), (N/2)*sizeof(float4), hipMemcpyHostToDevice );
		errorCheck("hipMemcpyAsync VelSndHalf 0");
		
		hipSetDevice(1);
		errorCheck("hipSetDevice 0");
		hipMemcpyAsync( PosFstHalf_1, Pos, (N/2)*sizeof(float4), hipMemcpyHostToDevice );
		errorCheck("hipMemcpyAsync PosFstHalf 0");
		hipMemcpyAsync( PosSndHalf_1, Pos+(N/2), (N/2)*sizeof(float4), hipMemcpyHostToDevice );
		errorCheck("hipMemcpyAsync PosSndHalf 0");
		hipMemcpyAsync( VelFstHalf_1, Vel, (N/2)*sizeof(float4), hipMemcpyHostToDevice );
		errorCheck("hipMemcpyAsync VelFstHalf 0");
		hipMemcpyAsync( VelSndHalf_1, Vel+(N/2), (N/2)*sizeof(float4), hipMemcpyHostToDevice );
		errorCheck("hipMemcpyAsync VelSndHalf 0");
	}
}

double nBodyCollisionSingleGPU()
{ 
	int   tDraw = 1; 
	int   tRecord = 1;
		
	while(RunTime <= TotalRunTime)
	{
		getForcesCollisionSingleGPU<<<GridConfig, BlockConfig>>>(Pos_DEV0, Vel_DEV0, Force_DEV0, ForceCollisionConstant);
		moveBodiesCollisionSingleGPU<<<GridConfig, BlockConfig>>>(Pos_DEV0, Vel_DEV0, Force_DEV0, MoveCollisionConstant);
		
		if(tDraw == DrawRate) 
		{
			hipMemcpy( Pos, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			hipMemcpy( Vel, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
			if	(DrawQuality == 1) drawAnalysisPictureCollision(); 
			else if	(DrawQuality == 2) drawPictureCollision();
			else 
			{
				printf("\nTSU Error: Invalid draw quality\n");
				exit(0);
			}
			tDraw = 0;
			printf("\nCollision run time = %f hours\n", RunTime*UnitTime/3600.0);
		}
		tDraw++;
		
		if(PrintCollisionStats == 1) 
		{
			hipMemcpy( Pos, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			hipMemcpy( Vel, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");	
			printCollisionStatsToScreen(RunTime);
			PrintCollisionStats = 0;
		}
		
		if(PrintContinueStats == 1) 
		{
			hipMemcpy( Pos, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			hipMemcpy( Vel, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
			printContinueStatsToScreen(RunTime);	
			PrintContinueStats = 0;
		}
				
		if(WriteToFile == 1 && tRecord == RecordRate) 
		{
			hipMemcpy( Pos, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			hipMemcpy( Vel, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
			recordPosAndVel();	
			tRecord = 0;
		}
		tRecord++;
		
		RunTime += Dt;
	}
	RunTime = RunTime - Dt;
	
	hipMemcpy( Pos, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpyAsync Pos");
	hipMemcpy( Vel, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpyAsync Vel");
	
	return(RunTime);
}

double nBodyCollisionDoubleGPU()
{ 
	int   tDraw = 1; 
	int   tRecord = 1;
	cout << "\nCollision run time start = " << RunTime*UnitTime/3600.0 << " hours." << endl;
	
	while(RunTime <= TotalRunTime)
	{
		hipSetDevice(0);
		errorCheck("hipSetDevice 0");
		getForcesCollisionDoubleGPU0<<<GridConfig, BlockConfig>>>(PosFstHalf_0, PosSndHalf_0, VelFstHalf_0, VelSndHalf_0,   ForceFstHalf_0, N, ForceCollisionConstant);
		errorCheck("getForcesCollisionDoubleGPU 0");
		moveBodiesCollisionDoubleGPU0<<<GridConfig, BlockConfig>>>(PosFstHalf_0,  VelFstHalf_0,   ForceFstHalf_0,  N, MoveCollisionConstant);
		errorCheck("moveBodiesCollisionDoubleGPU 0");
		
		hipSetDevice(1);
		errorCheck("hipSetDevice 1");
		getForcesCollisionDoubleGPU1<<<GridConfig, BlockConfig>>>(PosFstHalf_1, PosSndHalf_1, VelFstHalf_1, VelSndHalf_1,   ForceSndHalf_1,  N, ForceCollisionConstant);
		errorCheck("getForcesCollisionDoubleGPU 1");
		moveBodiesCollisionDoubleGPU1<<<GridConfig, BlockConfig>>>(PosSndHalf_1,  VelSndHalf_1,   ForceSndHalf_1,  N, MoveCollisionConstant);
		errorCheck("moveBodiesCollisionDoubleGPU 1");
		
		hipDeviceSynchronize();
		errorCheck("hipDeviceSynchronize 1");

		hipSetDevice(0);
		errorCheck("hipSetDevice 0");
		hipMemcpyPeerAsync(PosFstHalf_1,1,PosFstHalf_0,0,(N/2)*sizeof(float4));
		errorCheck("hipMemcpyPeerAsync 0 - Pos");
		hipMemcpyPeerAsync(VelFstHalf_1,1,VelFstHalf_0,0,(N/2)*sizeof(float4));
		errorCheck("hipMemcpyPeerAsync 0 - Vel");
		
		hipDeviceSynchronize();
		errorCheck("hipDeviceSynchronize 2");
		
		hipSetDevice(1);
		errorCheck("hipSetDevice 1");
		hipMemcpyPeerAsync(PosSndHalf_0,0,PosSndHalf_1,1,(N/2)*sizeof(float4));
		errorCheck("hipMemcpyPeerAsync 1 - Pos");
		hipMemcpyPeerAsync(VelSndHalf_0,0,VelSndHalf_1,1,(N/2)*sizeof(float4));
		errorCheck("hipMemcpyPeerAsync 1 - Vel");
		
		hipDeviceSynchronize();
		errorCheck("hipDeviceSynchronize 3");
		
		if(tDraw == DrawRate) 
		{
			hipSetDevice(0);
			errorCheck("hipSetDevice 0");
			hipMemcpyAsync(Pos, PosFstHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			hipMemcpyAsync(Pos+(N/2), PosSndHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			
			hipSetDevice(1);
			errorCheck("hipSetDevice 1");
			hipMemcpyAsync(Vel, VelFstHalf_1, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
			hipMemcpyAsync(Vel+(N/2), VelSndHalf_1,  (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
			
			if	(DrawQuality == 1) drawAnalysisPictureCollision(); 
			else if	(DrawQuality == 2) drawPictureCollision();
			else 
			{
				printf("\nTSU Error: Invalid draw quality\n");
				exit(0);
			}
			tDraw = 0;
			cout << "\nCollision run time = " << RunTime*UnitTime/3600.0 << " hours." << endl;
		}
		tDraw++;
		
		if(PrintCollisionStats == 1) 
		{
			hipSetDevice(0);
			errorCheck("hipSetDevice 0");
			hipMemcpyAsync(Pos, PosFstHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			hipMemcpyAsync(Pos+(N/2), PosSndHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			
			hipSetDevice(1);
			errorCheck("hipSetDevice 1");
			hipMemcpyAsync(Vel, VelFstHalf_1, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
			hipMemcpyAsync(Vel+(N/2), VelSndHalf_1,  (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
			
			printCollisionStatsToScreen(RunTime);
			PrintCollisionStats = 0;
		}
		
		if(PrintContinueStats == 1) 
		{
			hipSetDevice(0);
			errorCheck("hipSetDevice 0");
			hipMemcpyAsync(Pos, PosFstHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			hipMemcpyAsync(Pos+(N/2), PosSndHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			
			hipSetDevice(1);
			errorCheck("hipSetDevice 1");
			hipMemcpyAsync(Vel, VelFstHalf_1, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
			hipMemcpyAsync(Vel+(N/2), VelSndHalf_1,  (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
			
			printContinueStatsToScreen(RunTime);
			PrintContinueStats = 0;
		}
		
		if(WriteToFile == 1 && tRecord == RecordRate) 
		{
			hipSetDevice(0);
			errorCheck("hipSetDevice 0");
			hipMemcpyAsync(Pos, PosFstHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			hipMemcpyAsync(Pos+(N/2), PosSndHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Pos");
			
			hipSetDevice(1);
			errorCheck("hipSetDevice 1");
			hipMemcpyAsync(Vel, VelFstHalf_1, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
			hipMemcpyAsync(Vel+(N/2), VelSndHalf_1,  (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
			errorCheck("hipMemcpyAsync Vel");
		
			recordPosAndVel();	
			tRecord = 0;
		}
		tRecord++;
	
		RunTime += Dt;
	}
	RunTime = RunTime -Dt;
	cout << "\nCollision run time end = " << RunTime*UnitTime/3600.0 << " hours." << endl;
	
	hipSetDevice(0);
	errorCheck("hipSetDevice 0");
	hipMemcpyAsync(Pos, PosFstHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpyAsync Pos");
	hipMemcpyAsync(Pos+(N/2), PosSndHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpyAsync Pos");
	
	hipSetDevice(1);
	errorCheck("hipSetDevice 1");
	hipMemcpyAsync(Vel, VelFstHalf_1, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpyAsync Vel");
	hipMemcpyAsync(Vel+(N/2), VelSndHalf_1,  (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
	errorCheck("hipMemcpyAsync Vel");
	
	return(RunTime);
}

void cleanKill(double time)
{
	if(NumberOfGpus == 1 || UseMultipleGPU == 0) 
	{
		hipMemcpy( Pos, Pos_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
		errorCheck("hipMemcpyAsync Pos");
		hipMemcpy( Vel, Vel_DEV0, N *sizeof(float4), hipMemcpyDeviceToHost );
		errorCheck("hipMemcpyAsync Vel");
	}
	else
	{
		hipSetDevice(0);
		errorCheck("hipSetDevice 0");
		hipMemcpyAsync(Pos, PosFstHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
		errorCheck("hipMemcpyAsync Pos");
		hipMemcpyAsync(Pos+(N/2), PosSndHalf_0, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
		errorCheck("hipMemcpyAsync Pos");
	
		hipSetDevice(1);
		errorCheck("hipSetDevice 1");
		hipMemcpyAsync(Vel, VelFstHalf_1, (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
		errorCheck("hipMemcpyAsync Vel");
		hipMemcpyAsync(Vel+(N/2), VelSndHalf_1,  (N/2)*sizeof(float4), hipMemcpyDeviceToHost );
		errorCheck("hipMemcpyAsync Vel");
	}
	
	recordFinalCollisionStat(time);
	
	recordContinuePosAndVel(time);
	
	printContinueStatsToFile(time);
	
	cleanUpCollision();
	exit(0);
}

static void signalHandler(int signum)
{
	int command;
    
	cout << "\n\n******************************************************" << endl;
	cout << "Enter:666 to kill the run." << endl;
	cout << "Enter:1 to cleanly terminate the run.\t(not valid in the setup stage)." << endl;
	cout << "Enter:2 to change the draw rate." << endl;
	cout << "Enter:3 to change the draw quality.\t(not valid in the setup stage)." << endl;
	cout << "Enter:4 to set your eye location." << endl;
	cout << "Enter:5 to set the Center of Mass as your center." << endl;
	cout << "Enter:6 to print the run stats.\t(not valid in the setup stage)." << endl;
	cout << "Enter:7 to print the continue stats.\t(not valid in the setup stage)." << endl;
	cout << "Enter:8 to change the total run time." << endl;
	cout << "Enter:9 to continue the run." << endl;
	cout << "******************************************************\n\nCommand: ";
    
	cin >> command;
    
	if(command == 666)
	{
	
		cout << "\n\n******************************************************" << endl;
		cout << "Are you sure you want to terminate the run?" << endl;
		cout << "Enter:666 again if you are sure. Enter anything else to continue the run." << endl;
		cout << "******************************************************\n\nCommand: ";
		cin >> command;
		
		if(command == 666)
		{
			cleanUpCollision();
			exit(0);
		}
	}
	else if(command == 1)
	{
		cleanKill(RunTime);
	}
	else if(command == 2)
	{
		cout << "\nEnter the desired draw rate: ";
		cin >> DrawRate;
		cout << "\nDrawRate: " << DrawRate << endl;
	}
	else if(command == 3)
	{
		cout << "\nEnter the desired draw quality.\n1 for analysis.\n2 for standard." << endl;
		cin >> DrawQuality;
		cout << "\nDrawQuality: " << DrawQuality << endl;
	}
	else if (command == 4)
	{
    	cout << "******************************************************" << endl;
		cout << "Here is where your current Eye is at: " << endl;
		cout << "EyeX: " << EyeX << endl;
		cout << "EyeY: " << EyeY << endl;
		cout << "EyeZ: " << EyeZ << endl;
		cout << "Changing this will determine how close/far you are." << endl;
    	cout << "******************************************************" << endl;
		cout << "\nEnter the desired x location of your eye (double): ";
		cin >> EyeX;
		cout << "Enter the desired y location of your eye (double): ";
		cin >> EyeY;
		cout << "Enter the desired z location of your eye (double): ";
		cin >> EyeZ;
	    	
	    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glLoadIdentity();
		glMatrixMode(GL_PROJECTION);
		glLoadIdentity();
		glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
		glMatrixMode(GL_MODELVIEW);
		gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	    	//glutPostRedisplay();
	    	//Display();
	}
	else if (command == 5)
	{
		float3 temp = getCenterOfMassCollision(0);
		cout << "******************************************************" << endl;
		cout << "Center of Mass in the X-direction: " << temp.x << endl;
		cout << "Center of Mass in the Y-direction: " << temp.y << endl;
		cout << "Center of Mass in the Z-direction: " << temp.z << endl;
		cout << "This is the Center of Mass of the System" << endl;
    	cout << "******************************************************" << endl;
		
		CenterX = temp.x;
		CenterY = temp.y;
		CenterZ = temp.z;

		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glLoadIdentity();
		glMatrixMode(GL_PROJECTION);
		glLoadIdentity();
		glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
		glMatrixMode(GL_MODELVIEW);
		gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	    	//glutPostRedisplay();
	    	//Display();
	}
	else if (command == 6)
	{
		PrintCollisionStats = 1;
	}
	else if (command == 7)
	{
		PrintContinueStats = 1;
	}
	else if (command == 8)
	{
		cout << "\nEnter the desired TotalRunTime (float): ";
		cin >> TotalRunTime;

		TotalRunTime *= 3600.0/UnitTime;
	}
	else if (command == 9)
	{
		cout << "\nRun continued." << endl;
	}
	else
	{
		cout <<"\n\n Invalid Command\n" << endl;
	}
}

void typeOfRunCheck() 
{
	cout << "\nEnter 0 to create a new Run.\nEnter 1 to create a branch Run.\nEnter 2 to continue an existing Run.\n\n";
	cin >> TypeOfRun;
}

void readRootStartPosAndVelFile()
{
	FILE *temp = fopen("RootStartPosAndVel","rb");
	fread(Pos, sizeof(float4), N, temp);
	fread(Vel, sizeof(float4), N, temp);
	fclose(temp);
	
	fseek(PosAndVelFile,0,SEEK_END);
}

void readContinuePosAndVel()
{
	ContinueRunPosAndVelFile = fopen("ContinueRunPosAndVel","rb");
	fread(&RunTime, sizeof(double), 1, ContinueRunPosAndVelFile);
	fread(Pos, sizeof(float4), N, ContinueRunPosAndVelFile);
	fread(Vel, sizeof(float4), N, ContinueRunPosAndVelFile);
	//ContinueRunPosAndVelFile.clear();
	fclose(ContinueRunPosAndVelFile);
}

void control()
{	
	double time;
	struct sigaction sa;
	
	sa.sa_handler = signalHandler;
	sigemptyset(&sa.sa_mask);
	sa.sa_flags = SA_RESTART; // Restart functions if interrupted by handler
	if (sigaction(SIGINT, &sa, NULL) == -1)
	{
		printf("\nTSU Error: sigaction error\n");
	}

	//Setup run
	if (TypeOfRun == 0) {
		createFolderForNewRun();
		readRunParameters();
		setRunParameters();
		openNewRunFiles();
		recordSetupStats();
		loadKernalConstantStructures();
		allocateCPUMemory();
		checkSetupForErrors();
	
		//Create and initialize bodies
		deviceSetupSeperate();	
		createBodies();	
		nBodySeperate();
		resetInitialConditions();    	
		recordStatsOfCreatedBodies(); 
		recordStartPosVelOfCreatedBodiesSeperate();  	
		transformInitialConditionsFromSeperateToCollision();    	
		cleanUpSeperate();
	
		//Collide bodies
		deviceSetupCollision();
		copyCreatedBodiesUpToDevice();
	
		if(NumberOfGpus == 1 || UseMultipleGPU == 0) time = nBodyCollisionSingleGPU();
		else time = nBodyCollisionDoubleGPU();
	
		recordFinalCollisionStat(time);
		recordContinuePosAndVel(time);
		printContinueStatsToFile(time);
		cleanUpCollision();
		printf("\n DONE \n");
		exit(0);
	}
	else if (TypeOfRun == 1) 
	{
		createFolderForBranchRun(RootFolderName);
		readRunParameters();
		setRunParameters();
		readBranchParameters();
		setBranchParameters();
		
		openBranchRunFiles();
		
		allocateCPUMemory();
		
		readRootStartPosAndVelFile();
		
		InitialPosition1.x += BranchPosition1.x;
		InitialPosition1.y += BranchPosition1.y;
		InitialPosition1.z += BranchPosition1.z;
		
		InitialPosition2.x += BranchPosition2.x;
		InitialPosition2.y += BranchPosition2.y;
		InitialPosition2.z += BranchPosition2.z;
		
		InitialVelocity1.x += BranchVelocity1.x;
		InitialVelocity1.y += BranchVelocity1.y;
		InitialVelocity1.z += BranchVelocity1.z;
		
		InitialVelocity2.x += BranchVelocity2.x;
		InitialVelocity2.y += BranchVelocity2.y;
		InitialVelocity2.z += BranchVelocity2.z;
		
		InitialSpin1.x += BranchSpin1.x;
		InitialSpin1.y += BranchSpin1.y;
		InitialSpin1.z += BranchSpin1.z;
		InitialSpin1.w += BranchSpin1.w;
		
		InitialSpin2.x += BranchSpin2.x;
		InitialSpin2.y += BranchSpin2.y;
		InitialSpin2.z += BranchSpin2.z;
		InitialSpin2.w += BranchSpin2.w;
		
		recordSetupStats();
		
		loadKernalConstantStructures();
		checkSetupForErrors();
		deviceSetupSeperate();
		
		//From here down to nBodySeperate is like the create bodies above but all that needs to be done is move and spin 
		setBodyPositionSeperate(1, InitialPosition1.x, InitialPosition1.y, InitialPosition1.z);
		//setBodyVelocitySeperate(1, InitialVelocity1.x, InitialVelocity1.y, InitialVelocity1.z);
		setBodyPositionSeperate(2, InitialPosition2.x, InitialPosition2.y, InitialPosition2.z);
		//setBodyVelocitySeperate(2, InitialVelocity2.x, InitialVelocity2.y, InitialVelocity2.z);
		
		//This is really the added spin but must be put in initail to fool nBodySeperate because the original spin is already done
		InitialSpin1 = BranchSpin1;
		InitialSpin2 = BranchSpin2;
		
		DampTime = -1.0;
		DampRestTime = -1.0;
		SetupTime = BranchSpinRestTime;
		
		nBodySeperate();
		
		resetInitialConditions();    	
		recordStatsOfCreatedBodies(); 
		recordStartPosVelOfCreatedBodiesSeperate();  	
		transformInitialConditionsFromSeperateToCollision();    	
		cleanUpSeperate();
	
		//Collide bodies
		TotalRunTime = BranchRunTime;
		deviceSetupCollision();
		copyCreatedBodiesUpToDevice();
	
		if(NumberOfGpus == 1 || UseMultipleGPU == 0) time = nBodyCollisionSingleGPU();
		else time = nBodyCollisionDoubleGPU();
	
		recordFinalCollisionStat(time);
		recordContinuePosAndVel(time);
		printContinueStatsToFile(time);
		cleanUpCollision();
		printf("\n DONE \n");
		exit(0);
	}
	else if (TypeOfRun == 2)
	{
		chdir(RootFolderName);
		
		readRunParameters();
		setRunParameters();
		
		loadKernalConstantStructures();
		allocateCPUMemory();
		checkSetupForErrors();
		
		readContinuePosAndVel();
		
		openContinueRunFiles();

		TotalRunTime = AddedRunTime*3600.0/UnitTime + RunTime;
		
		//Collide bodies
		deviceSetupCollision();
		copyCreatedBodiesUpToDevice();
		
		if(NumberOfGpus == 1 || UseMultipleGPU == 0) time = nBodyCollisionSingleGPU();
		else time = nBodyCollisionDoubleGPU();
	
		recordFinalCollisionStat(time);
		recordContinuePosAndVel(time);
		printContinueStatsToFile(time);
		cleanUpCollision();
		printf("\n DONE \n");
		exit(0);
	}
	else
	{
		printf("\n Bad TypeOfRun value \n");
		exit(0);
	}
}

//https://www.opengl.org/archives/resources/faq/technical/viewing.htm
void Display(void)
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glLoadIdentity();
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(Left, Right, Bottom, Top, Front, Back);
	glMatrixMode(GL_MODELVIEW);
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
}

void reshape(GLint w, GLint h) 
{
	glViewport(0, 0, w, h);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(Left, Right, Bottom, Top, Front, Back);
	glMatrixMode(GL_MODELVIEW);
}

void init()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glLoadIdentity();
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(Left, Right, Bottom, Top, Front, Back);
	glMatrixMode(GL_MODELVIEW);
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
}

int main(int argc, char** argv)
{
	if( argc < 1)
	{
		printf("\n You need to intire the run type (int 0 new run, 1 branch run, or 2 continue run) on the comand line\n"); 
		exit(0);
	}
	else
	{
		TypeOfRun = atoi(argv[1]);
	}
	
	if( TypeOfRun == 1)
	{
		if(argc < 2)
		{
			printf("\n You need to intire a root folder to work from on the comand line\n");
			exit(0);
		}
		else
		{
			strcat(RootFolderName, argv[2]);
		}
	}
	
	if( TypeOfRun == 2)
	{
		if(argc < 2)
		{
			printf("\n You need to intire a root folder to work from on the comand line\n");
			exit(0);
		}
		else 
		{
			strcat(RootFolderName, argv[2]);
		}
		
		if(argc < 3)
		{
			printf("\n You need to intire the extra run time for the continuation\n");
			exit(0);
		}
		else 
		{
			AddedRunTime = atof(argv[3]);
		}
	}
	
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Giant Impact Hypothesis Simulation");
	
	glutReshapeFunc(reshape);
	
	init();
	
	glShadeModel(GL_SMOOTH);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutIdleFunc(control);
	glutMainLoop();
	return 0;
}






